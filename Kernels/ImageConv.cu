
#include <hip/hip_runtime.h>
__device__ void imageconvolution(void *param)
{

float *input = (float *) param;
int IW = (int)input[0];   //Image Width
int MW = (int)input[1]; //MASK_WIDTH;
float* image = input+2; 
float* mask = image + IW;
float* imageout = image + MW + IW;
int warp_size=32;
int threadId = threadIdx.x % warp_size;
float value =0;
int start;
int index;
printf("%d - %d \n", MW, IW);
//this function includes 2 floating point operations
while(threadId < IW)
{
start = threadId - (MW/2);
for(int i=0; i<MW;i++){
        index= start + i;
        if(index >=0 && index < IW)
                value = value + image[index] * mask[i];
}
threadId = threadId + warp_size;
imageout[threadId] = value;
printf("%d - %f \n", threadId, imageout[threadId]);
}
}

