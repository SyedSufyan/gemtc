#include "hip/hip_runtime.h"
#include <math.h>
#include "../debunk/bilat/helper/uint_util.hcu"
#include "../debunk/bilat/helper/float_util.hcu"
#define PI 3.14159265

__device__
float gaussian1d(float x, float sigma)
{
        float variance = pow(sigma,2);
        float exponent = -pow(x,2)/(2*variance);
        return expf(exponent) / sqrt(2 * PI * variance);
}

__device__
float gaussian2d(float x, float y, float sigma)
{
        float variance = pow(sigma,2);
        float exponent = -(pow(x,2) + pow(y,2))/(2*variance);
        return expf(exponent) / (2 * PI * variance);
}
__device__
void bilateralFilter(void *inp)
{
        unsigned int idx = threadIdx.x % 32;
	float3* input;
	float3* output;
	uint2 dims;
        float3 *in = (float3 *)inp;
        float3 first = in[0];
	float3 second =in[1];
        float3 third = in[2];

        int size = first.x;
	int width = first.y;
	int height = first.z;
	dims = make_uint2(width,height);
	//unsigned int channel = second.x;
	unsigned int radius = second.y;
	float sigma_spatial = second.z;
	float sigma_range = third.x;

	input = in + 3;
	output = input + size;
	while(idx < size) {
//	printf("Thread : %d\n",idx);
        uint2 pos = idx_to_co(idx,dims);
        int img_x = pos.x;
        int img_y = pos.y;

        if(img_x >= dims.x || img_y >= dims.y) return;

        float3 currentColor = input[idx];

        float3 res = make_float3(0.0f,0.0f,0.0f);
        float3 normalization = make_float3(0.0f,0.0f,0.0f);;


        for(int i = -radius; i <= radius; i++) {
                for(int j = -radius; j <= radius; j++) {
                        int x_sample = img_x+i;
                        int y_sample = img_y+j;

                        //mirror edges
                        if( x_sample < 0) x_sample = -x_sample;
                        if( y_sample < 0) y_sample = -y_sample;
                        if( x_sample > dims.x - 1) x_sample = dims.x - 1 - i;
                        if( y_sample > dims.y - 1) y_sample = dims.y - 1 - j;


                        float3 tmpColor = input[co_to_idx(make_uint2(x_sample,y_sample),dims)];

                        float gauss_spatial = gaussian2d(i,j,sigma_spatial);
                        float3 gauss_range;
                        gauss_range.x = gaussian1d(currentColor.x - tmpColor.x, sigma_range);
                        gauss_range.y = gaussian1d(currentColor.y - tmpColor.y, sigma_range);
                        gauss_range.z = gaussian1d(currentColor.z - tmpColor.z, sigma_range);

                        float3 weight = gauss_spatial * gauss_range;
                        normalization = normalization + weight;
                        res = res + (tmpColor * weight);

                }
        }

        res.x /= normalization.x;
        res.y /= normalization.y;
        res.z /= normalization.z;
        output[idx] = res;
	idx +=32;
	}
}
