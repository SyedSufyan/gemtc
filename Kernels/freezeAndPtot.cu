#include "hip/hip_runtime.h"
/*
 * Application:- Imogen's ported cukern_FreezeSpeed_hydro() kernel from "gpuImogen/gpuclass/freezeAndPtot.cu"
 * Purpose:-
 *     This function is used to derive pressure and freeze parameters to enforce minimum pressure.
 */
#include <stdio.h>
#include <string.h>
#include <stdarg.h>
#ifdef UNIX
#include <stdint.h>
#include <unistd.h>
#endif

/*
 * These macros are different from Imogen's macros as we can only have 32 size 
 */
#define BLOCKDIM 32
#define MAXPOW   5

#define ALFVEN_FACTOR 1
/*
 * Shader frequency of GTX 480 
 * Better will be to deriver this in case we are simulation in a different GPU
 * But calling function to derive frequncy so many times will be still expensive
 * Tip:- Get this from the caller as input parameter.
 */
#define SHADER_CLOCK 1401000

/*
 * Kernel for FreezeSpeed_hydro
 */
__device__ void cukern_FreezeSpeed_hydro(void* params)
{
    /*
     * Kernel Benchmarking parameters
     * Uncomment to benchmark inside CUDA kernel.
     * Don't uncomment otherwise else it will lead to unnecessary console logs.
     */
    //uint start, stop;
    //start = clock();
    double* locBloc = (double*)gemtcSharedMemory();

    double gammafunc[6];
    int tid = threadIdx.x % 32;

    double* paramsIn = (double*)params;

    /*
     * Get X dimension
     */
    int nx = (int)paramsIn[0];

    /*
     * Get Y dimension
     */
    paramsIn = paramsIn + 1;
    int ny = (int)paramsIn[0];

    /*
     * Get Z dimension
     */
    paramsIn = paramsIn + 1;
    int nz = (int)paramsIn[0];

    /*
     * Get rho
     */
    paramsIn = paramsIn + 1;
    double* rho = paramsIn;

    /*
     * Get E
     */
    paramsIn = paramsIn + nx*ny*nz;
    double* E = paramsIn; 

    /*
     * Get px
     */
    paramsIn = paramsIn + nx*ny*nz;
    double* px = paramsIn; 

    /*
     * Get py
     */
    paramsIn = paramsIn + nx*ny*nz;
    double* py = paramsIn; 

    /*
     * Get pz
     */
    paramsIn = paramsIn + nx*ny*nz;
    double* pz = paramsIn; 

    /*
     * Purehydro simulation hence magnetic variables are not needed.
     * We want to avoid unnecessary data transfer b/w CPU <-> GPU
     * Remember the application must also not provide these parameters
     * else offset computation will break
     */
#if 0 
    /*
     * Get bx
     */
    paramsIn = paramsIn + nx*ny*nz;
    double* bx = paramsIn; 

    /*
     * Get by
     */
    paramsIn = paramsIn + nx*ny*nz;
    double* by = paramsIn; 

    /*
     * Get bz
     */
    paramsIn = paramsIn + nx*ny*nz;
    double* bz = paramsIn; 
#endif 
    
    /*
     * Get gamma
     */
    paramsIn = paramsIn + 1;
    double gamma = paramsIn[0]; 

    /*
     * Get cs0
     */
    paramsIn = paramsIn + 1;
    double cs0 = paramsIn[0]; 

    /*
     * Output variable
     */

    /*
     * Get pressa
     */
    paramsIn = paramsIn + 1;
    double* ptot = paramsIn;

    /*
     * Get freezea
     */
    paramsIn = paramsIn + nx*ny*nz;
    double* freeze = paramsIn;

    gammafunc[0] = gamma;
    gammafunc[1] = gamma - 1.0;
    gammafunc[2] = gamma*(gamma-1.0);
    gammafunc[3] = (1.0 - .5*gamma);
    gammafunc[4] = cs0*cs0; // min c_s squared ;
    gammafunc[5] = (ALFVEN_FACTOR - .5*gamma*(gamma-1.0));
  
#define gam gammafunc[0]
#define gm1 gammafunc[1]
#define gg1 gammafunc[2]
#define cs0sq gammafunc[4]

#define PRESSURE Cs
// cs0sq = gamma rho^(gamma-1))

    /*
     * Traverse 3-D data and solve the equations
     */
    for(int i = 0; i < ny; i++) {
        for(int j = 0; j < nz; j++) { 
            int x = tid + nx*(i + ny*j);
            int addrMax = nx + nx*(i + ny*j);

            double Cs, CsMax;
            double psqhf, rhoinv;
            //double gg1 = gam*(gam-1.0);
            //double gm1 = gam - 1.0;


            CsMax = 0.0;
            locBloc[tid] = 0.0;

            while(x < addrMax) {
                rhoinv   = 1.0/rho[x];
                psqhf    = .5*(px[x]*px[x]+py[x]*py[x]+pz[x]*pz[x]);

                PRESSURE = gm1*(E[x] - psqhf*rhoinv);
                if(gam*PRESSURE*rhoinv < cs0sq) {
                    PRESSURE = cs0sq/(gam*rhoinv);
                    E[x] = psqhf*rhoinv + PRESSURE/gm1;
                } /* Constrain temperature to a minimum value */
                ptot[x] = PRESSURE;

                Cs      = sqrt(gamma * PRESSURE *rhoinv) + abs(px[x]*rhoinv);
                if(Cs > CsMax) CsMax = Cs;

                x += BLOCKDIM;
            }

            locBloc[tid] = CsMax;

            //__syncthreads();

            if (tid % 8 == 0) { // keep threads  [0 8 16 ...]

                // Each searches the max of the nearest 8 points
                for(x = 1; x < 8; x++) {
                    if(locBloc[tid+x] > locBloc[tid]) locBloc[tid] = locBloc[tid+x];
                    //__syncthreads();
                }

                // The last thread takes the max of these maxes
                if(tid == 0) {
                    for(x = 8; x < BLOCKDIM; x+= 8) {
                        if(locBloc[x] > locBloc[0]) locBloc[0] = locBloc[x];
                    }

                    // NOTE: This is the dead-stupid backup if all else fails.
                    //if(threadIdx.x > 0) return;
                    //for(x = 1; x < GLOBAL_BLOCKDIM; x++)  if(locBloc[x] > locBloc[0]) locBloc[0] = locBloc[x];

                    freeze[i + ny*j] = locBloc[0];
                }
            }
        }
    }
    /*
     * Kernel Benchmarking parameters
     * Uncomment to benchmark inside CUDA kernel.
     * Don't uncomment otherwise else it will lead to unnecessary console logs.
     */
    //stop = clock();
    //float time;
    //if (stop > start) {
    //    time = (float)(stop - start)/(float)SHADER_CLOCK;
    //} else {
    //    time = (float)(stop + (0xffffffff - start))/(float)SHADER_CLOCK;
    //}
    //printf("Time taken %f ms\n", time);    
}
