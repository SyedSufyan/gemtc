#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdarg.h>
#ifdef UNIX
#include <stdint.h>
#include <unistd.h>
#endif

/*
 * THIS FUNCTION 
 * This is the Cuda Fluid TVD function; It takes a single forward-time step, CFD or MHD, of the
 * conserved-transport part of the fluid equations using a total variation diminishing scheme to
 * perform a non-oscillatory update.
 * requires predicted half-step values from a 1st order upwind scheme.
*/

#define BLOCKLEN 28
#define BLOCKLENP2 30
#define BLOCKLENP4 32

__device__ __inline__ double fluxLimiter_Vanleer(double derivL, double derivR);

#define RHOMIN fluidParams[0]
#define MIN_ETHERM fluidParams[1]

__device__ void cukern_TVDStep_hydro_uniform(void* params)
{
    //uint start, stop;
    //start = clock();
    double fluidParams[2];
    //printf("GPU: GEBUG LOG 1\n");
    int tid = threadIdx.x % 32;
    double* paramsIn = (double*)params;

    /*
     * Get dimensions
     */
    int nx = (int)paramsIn[0];

    paramsIn = paramsIn + 1;
    int ny = (int)paramsIn[0];

    paramsIn = paramsIn + 1;
    int nz = (int)paramsIn[0];

    /*
     * Get rho
     */
    paramsIn = paramsIn + 1;
    double* rho = paramsIn;

    /*
     * Get E
     */
    paramsIn = paramsIn + nx*ny*nz;
    double* E = paramsIn; 

    /*
     * Get px
     */
    paramsIn = paramsIn + nx*ny*nz;
    double* px = paramsIn; 

    /*
     * Get py
     */
    paramsIn = paramsIn + nx*ny*nz;
    double* py = paramsIn; 

    /*
     * Get pz
     */
    paramsIn = paramsIn + nx*ny*nz;
    double* pz = paramsIn; 

    /*
     * Purehydro simulation hence magnetic variables are not needed.
     * We want to avoid unnecessary data transfer b/w CPU <-> GPU
     * Remember the application must also not provide these parameters
     * else offset computation will break
     */
#if 0 
    /*
     * Get bx
     */
    paramsIn = paramsIn + nx*ny*nz;
    double* bx = paramsIn; 

    /*
     * Get by
     */
    paramsIn = paramsIn + nx*ny*nz;
    double* by = paramsIn; 

    /*
     * Get bz
     */
    paramsIn = paramsIn + nx*ny*nz;
    double* bz = paramsIn; 
#endif 
    /*
     * Get P
     */
    paramsIn = paramsIn + nx*ny*nz;
    double* P = paramsIn; 

    /*
     * Output variable
     */

    /*
     * Get rho_out
     */
    paramsIn = paramsIn + nx*ny*nz;
    double* rho_out = paramsIn;

    /*
     * Get E_out
     */
    paramsIn = paramsIn + nx*ny*nz;
    double* E_out = paramsIn;

    /*
     * Get px_out
     */
    paramsIn = paramsIn + nx*ny*nz;
    double* px_out = paramsIn;

    /*
     * Get py_out
     */
    paramsIn = paramsIn + nx*ny*nz;
    double* py_out = paramsIn;

    /*
     * Get pz_out
     */
    paramsIn = paramsIn + nx*ny*nz;
    double* pz_out = paramsIn;

    /*
     * Get Cfreeze
     */
    paramsIn = paramsIn + nx*ny*nz;
    double* Cfreeze = paramsIn; 

    /*
     * Get lambda
     */
    paramsIn = paramsIn + ny*nz;
    double lambda = paramsIn[0]; 

    /*
     * Get rhomin
     */
    paramsIn = paramsIn + 1;
    double rhomin = paramsIn[0]; 

    /*
     * Get gamma
     */
    paramsIn = paramsIn + 1;
    double gamma = paramsIn[0]; 

    fluidParams[0] = rhomin;
// assert     cs > cs_min
//     g P / rho > g rho_min^(g-1)
// (g-1) e / rho > rho_min^(g-1)
//             e > rho rho_min^(g-1)/(g-1)
    //printf("GPU :: rhomin %lf, gamma %lf\n", rhomin, gamma);
    fluidParams[1] = pow(rhomin, gamma-1.0)/(gamma-1.0);
    double halfLambda = 0.5 * lambda;

    //cukern_TVDStep_hydro_uniform                         (*rho,    *E,     *px,      *py,     *pz,     *P,      *Cfreeze, *rhoW,  *enerW,     *pxW,     *pyW,     *pzW,     lambda, nx);
    double C_f, velocity;
    double q_i[5];
    double w_i;
    double* fluxLR = (double *) gemtcSharedMemory();
    double* fluxDerivA = fluxLR + BLOCKLENP4*2;
    double* fluxDerivB = fluxDerivA + BLOCKLENP4 + 1;
    //printf("ThreadIdx %d\n", threadIdx.x);
    //return;
    for(int i = 0; i < ny; i++) {
        for(int j = 0; j < nz; j++) {

            /* 
             * Step 0 - obligatory setup 
             */
            int I0 = nx*(i + j*ny);
            int Xindex = (tid-2);
            int Xtrack = Xindex;
            Xindex += nx*(tid < 2);

            int x; /* = Xindex % nx; */
            int k;
            bool doIflux = (tid > 1) && (tid < BLOCKLENP2);
            double prop_i[5];

            unsigned int threadIndexL = (tid-1+BLOCKLENP4)%BLOCKLENP4;

            /* 
             * Step 1 - calculate W values 
             */
            C_f = Cfreeze[i + j*ny];

            while(Xtrack < nx+2) {
                //printf("tid = %d, i = %d\n", tid, i);
                x = I0 + (Xindex % nx);

                q_i[0] = rho[x]; /* Preload these out here */
                q_i[1] = E[x]; /* So we avoid multiple loops */
                q_i[2] = px[x]; /* over them inside the flux loop */
                q_i[3] = py[x];
                q_i[4] = pz[x];
                velocity = q_i[2] / q_i[0];

                /* rho, E, px, py, pz going down */
                /* Iterate over variables to flux */
                for(k = 0; k < 5; k++) {
                    /* Step 1 - Calculate raw fluxes */
                    switch(k) {
                        case 0: w_i = q_i[2]; break;
                        case 1: w_i = (velocity * (q_i[1] + P[x]) ) ; break;
                        case 2: w_i = (velocity * q_i[2] + P[x]); break;
                        case 3: w_i = (velocity * q_i[3]); break;
                        case 4: w_i = (velocity * q_i[4]); break;
                    }

                    /* Step 2 - Decouple to L/R flux */
                    /* NOTE there is a missing .5 here, accounted for in the h(al)f of lambdahf */
                    fluxLR[tid] = (C_f*q_i[k] - w_i); /* Left  going flux */
                    fluxLR[BLOCKLENP4 + tid] = (C_f*q_i[k] + w_i); /* Right going flux */
                    //__syncthreads();

                    /* Step 3 - Differentiate fluxes & call limiter */
                    /* left flux */
                    fluxDerivA[tid] = fluxLR[threadIndexL] - fluxLR[tid];
                    fluxDerivB[tid] = fluxLR[BLOCKLENP4 + tid] - fluxLR[BLOCKLENP4 + threadIndexL];
                    //__syncthreads();
        
                    /* right flux */
                    fluxLR[tid] += fluxLimiter_Vanleer(fluxDerivA[tid], fluxDerivA[tid+1]);
                    fluxLR[BLOCKLENP4 + tid] += fluxLimiter_Vanleer(fluxDerivB[tid+1], fluxDerivB[tid]);
                    //__syncthreads();

                    /* Step 4 - Perform flux and write to output array */
                    if( doIflux && (Xindex < nx) ) {
                        switch(k) {
                            case 0:
                                prop_i[0] = rho_out[x] - halfLambda * ( fluxLR[tid] - fluxLR[tid+1] + \
                                                   fluxLR[BLOCKLENP4 + tid] - fluxLR[BLOCKLENP4 + threadIndexL]  );
                                break;
                            case 1:
                                prop_i[1] = E_out[x] - halfLambda * ( fluxLR[tid] - fluxLR[tid+1] + \
                                                   fluxLR[BLOCKLENP4 + tid] - fluxLR[BLOCKLENP4 + threadIndexL]  );
                                break;
                            case 2:
                                prop_i[2] = px_out[x] - halfLambda * ( fluxLR[tid] - fluxLR[tid+1] + \
                                                   fluxLR[BLOCKLENP4 + tid] - fluxLR[BLOCKLENP4 + threadIndexL]  );
                                break;
                            case 3:
                                prop_i[3] = py_out[x] - halfLambda * ( fluxLR[tid] - fluxLR[tid+1] + \
                                                   fluxLR[BLOCKLENP4 + tid] - fluxLR[BLOCKLENP4 + threadIndexL]  );
                                break;
                            case 4:
                                prop_i[4] = pz_out[x] - halfLambda * ( fluxLR[tid] - fluxLR[tid+1] + \
                                                   fluxLR[BLOCKLENP4 + tid] - fluxLR[BLOCKLENP4 + threadIndexL]  );
                                break;
                        }
                    }
                    //__syncthreads();
                }

                if( doIflux && (Xindex < nx) ) {
                    prop_i[0] = (prop_i[0] < RHOMIN) ? RHOMIN : prop_i[0];
                    w_i = .5*(prop_i[2]*prop_i[2] + prop_i[3]*prop_i[3] + prop_i[4]*prop_i[4])/prop_i[0];

                    if((prop_i[1] - w_i) < prop_i[0]*MIN_ETHERM) {
                        prop_i[1] = w_i + prop_i[0]*MIN_ETHERM;
                    }

                    rho_out[x] = prop_i[0];
                    E_out[x] = prop_i[1];
                    px_out[x] = prop_i[2];
                    py_out[x] = prop_i[3];
                    pz_out[x] = prop_i[4];
                }

                //__syncthreads();

                Xindex += BLOCKLEN;
                Xtrack += BLOCKLEN;
            }
        }
    }
    //stop = clock();
    //float time;
    //if (stop > start) {
    //    time = (float)(stop - start)/(float)SHADER_CLOCK;
    //} else {
    //    time = (float)(stop + (0xffffffff - start))/(float)SHADER_CLOCK;
    //}
    //printf("Time taken %f ms\n", time);    
}

__device__ double fluxLimiter_Vanleer(double derivL, double derivR)
{
    double r;

    r = derivL * derivR;
    if(r < 0.0) { r = 0.0; }

    r = r / ( derivL + derivR);
    if (isnan(r)) { r = 0.0; }

    return r;
}


