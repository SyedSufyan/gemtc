#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>

#define MIN(a,b) (a<b?a:b)
__device__ size_t string_len(const char *str){
  const char *s;
  for(s=str; *s; ++s);
  return(s-str);
}

__device__ void LCSubstring(void *params){
  //char* output = (char*)params;
  //char *string = "New message!!";
  //int i=0;
  //while(i<14){
  //  output[i]=string[i];
  //  i++;
  //}
  char** strs = (char**)(params); 
  
  char* s1 = *strs;
  char* s2 = *++strs; 
  int s1_size = string_len(s1); 
  int s2_size = string_len(s2);

  if(s1_size == 0 || s2_size == 0){
    return;
  } 

  int* table = (int*)malloc(sizeof(int)*s1_size*s2_size)
  int i,j;

  //Make sure all the values in the table are set to 0. 
  for(i=0; i<s1_size; i++){
    for(j=0; j<s2_size; j++){
      table[i][j] = 0;
    }
  }

  int max = 0; 
  char* ret = (char*)malloc(MIN(s1_size, s2_size)); 

  for(i=0; i<s1_size;i++){
    for(j=0; j<s2_size;j++){
      if(s1[i] == s2[j]){
        if(i==0 || j==0){
          table[i][j] = 1;
        }
        else{
          table[i][j] = table[i-1][j-1] + 1;
        }
        if(table[i][j] > max){
          max = table[i][j];
          strncpy(ret, &s1[i-max+1], max);
        }
      }
    }
  }

}
