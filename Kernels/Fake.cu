
#include <hip/hip_runtime.h>
__device__ void Fake(void *params){
  int *np = (int*)params;
  int *nb = np + 1;

  int size = (*np) * (*nb); 

  double *pos = (double *)(np + 2);
  *np = 15;
  *nb = 68;
 
  double *vel = pos + size;
  int i;
  for(i=0; i<size; i++){
    pos[i] = 3*i; 
    vel[i] = pos[i] * 2;
  }
}
