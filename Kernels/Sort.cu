
#include <hip/hip_runtime.h>
//This micro-kernel currently does not use shared memory
//It could be improved by adding this caching.

//This micro-kernel currently uses a selection sort.
//This is done for simplicity of testing and should be replaced
//  before using it for serious testing with a better sort.

__device__ void Sort( void* param)
{
    float* paramIn = (float*)param;
    int N = (int)(*paramIn);
    float* a  = paramIn+1;  //input data, will currently be trashed
    float* b  = a + N*sizeof(float); //location for result array

    int warp_size = 32;
    int tid = threadIdx.x%warp_size;

    float *sub = a + tid*N/warp_size;  //Sub list that each warp will sort
    
    //Selection Sort, eventually do Merge or something faster
    int i, j;
    for(i=0; i<N/warp_size-1; i++){
      //Find min is remaining list
      int min = i;
      for(j=i+1; j<N/warp_size; j++){
        if(sub[j]<sub[min])min = j;
      }
      //Swap ith place with min
      float temp = sub[i];
      sub[i]=sub[min];
      sub[min]=temp;
    }

    //Merge loops
    int subs;  //number of sub lists I currently have
    for(subs=warp_size; subs!=1; subs=subs/2){
      if(tid<subs/2){
        //Merge our two lists
        int sub_size = N/subs;
        float *sub1 = a + 2*tid*sub_size;
        float *sub2 = sub1 + sub_size;
        
        float *ret = b + 2*tid*sub_size;
        
        //Merge our two lists into their location in ret
        int p1 = 0;  //place in first list
        int p2 = 0;  //place in second list
        int cur= 0;
        while(p1<sub_size && p2<sub_size){
          if(sub1[p1]<sub[p2]){
            ret[cur++] = sub1[p1++];
          }else{
            ret[cur++] = sub2[p2++];
          }
        }
        //Copy any elements left in our lists after the first list runs out
        while(p1<sub_size) ret[cur++]=sub1[p1++];
        while(p2<sub_size) ret[cur++]=sub2[p2++];

        int k;
        for(k=0;k<2*sub_size;k++)sub1[k]=ret[k];
      }
      /*      //Copy the now sorted sub arrays back into sub to be merged again
      int k;
      for(k=tid; k<N; k+=warp_size) a[k]=b[k];
      */
    }
}
