/*
 * Application:- SciColSim's expensive function update_probabilities_all_visible() from "scicolsim-2013-03-07/src/optimizer.cpp"
 * Purpose:-
 *    It implements the GeMTC kernel for gemtc_update_probabilities_all_visible() of scicolsim application.
 * Difference from SciColSim.cu, this does the whole processing by 1 thread in the warp.
 * Hence has been developed for efficiency comparison.
 *
 * How to use:-
 *     Replace SciColSim.cu with this file.
 * mv SciColSim.cu SciColSim_Warp.cu
 * mv SciColSim_Single.c SciColSim.cu
 * build scicolsim application.
 */

#include <hip/hip_runtime.h>
#include <cfloat>

/*
 * Type used to represent state of edge (final and tried)
 * Use bitfield so compact in memory
typedef struct {
    bool final : 1;
    bool tried : 1;
} edge_state;
 */

/*
 * Shader frequency of GTX 480 
 * Better will be to deriver this in case we are simulation in a different GPU
 * But calling function to derive frequncy so many times will be still expensive
 * Tip:- Get this from the caller as input parameter.
 */
#define SHADER_CLOCK 1401000
typedef struct {
    int final;
    int tried;
} edge_state;

__device__ void gemtc_update_probabilities_all_visible(void* params)
{
    /*
     * Kernel Benchmarking parameters
     * Uncomment to benchmark inside CUDA kernel.
     * Don't uncomment otherwise else it will lead to unnecessary console logs.
     */
    //uint start, stop;
    //start = clock();
    if (threadIdx.x % 32 != 0) {
        //printf("GPU Returning thread %d\n", threadIdx.x);
        return;
    }
    int N_nodes, warp_size = 32;

    //double *Summa = (double *) gemtcSharedMemory();
    double Summa = 0.;
    /*
     * Get input parameters
     */
    double* paramsIn = (double*)params;

    /*
     * First argument is basis of getting into this function
     */
    paramsIn = paramsIn + 1;
    /*
     * Pre-compute logs
     */
    N_nodes = (int)paramsIn[0];
    //printf(" thread %d::N_nodes %d\n", threadIdx.x, N_nodes);

    /*
     * Alpha i
     */
    paramsIn = paramsIn + 1;
    double alpha_i = (double)paramsIn[0];
    //printf(" thread %d::Alpha_i %f\n", threadIdx.x, alpha_i);

    /*
     * Alpha m
     */
    paramsIn = paramsIn + 1;
    double alpha_m = (double)paramsIn[0];
    //printf(" thread %d::Alpha_m %f\n", threadIdx.x, alpha_m);

    /*
     * k_max     
     */
    paramsIn = paramsIn + 1;
    double k_max = (double)paramsIn[0];
    printf(" thread %d::k_max %f\n", threadIdx.x, k_max);

    /*
     * beta     
     */
    paramsIn = paramsIn + 1;
    double beta = (double)paramsIn[0];
    //printf(" thread %d::Beta %f\n", threadIdx.x, beta);

    /*
     * gamma 
     */
    paramsIn = paramsIn + 1;
    double gamma = (double)paramsIn[0];
    //printf(" thread %d::gamma %f\n", threadIdx.x, gamma);

    /*
     * delta 
     */
    paramsIn = paramsIn + 1;
    double delta = (double)paramsIn[0];
    //printf(" thread %d::Delta%f\n", threadIdx.x, delta);

    /*
     * Total probability 
     */
    paramsIn = paramsIn + 1;
    //double TotalProb = (double)paramsIn[0];
    //printf(" thread %d::Total_probability %f\n", threadIdx.x, TotalProb);

    /*
     * LogRanks
     */
    paramsIn = paramsIn + 1;
    double *LogRanks = (double*)paramsIn;

    /*
     * Rank
     */
    paramsIn = paramsIn + N_nodes;
    double *Rank = (double*)paramsIn;
    //for (int i = threadIdx.x; i < N_nodes; i+=warp_size) {
    for (int i = 0; i < N_nodes; i++) {
        LogRanks[i] = log(Rank[i]+1.);
    }

    /*
     * ProbSums
     */
    paramsIn = paramsIn + N_nodes;
    double *ProbSums = (double*)paramsIn;

    /*
     * Prob
     */
    paramsIn = paramsIn + N_nodes;
    double* Prob = (double*)paramsIn;

    /*
     * Dist
     */
    paramsIn = paramsIn + N_nodes*N_nodes;
    double* Dist = (double*)paramsIn;

    /*
     * State
     */
    paramsIn = paramsIn + N_nodes*N_nodes;
    edge_state *State = (edge_state*)paramsIn;
    paramsIn = paramsIn + N_nodes*N_nodes;
/*
        printf(" GPU STATE\n");
    for(int i=0; i<5;i++) {
        for(int j=0;j<5;j++) {
            printf("    State[%d][%d].final = %d:: State[%d][%d].tried = %d", i, j, State[i*N_nodes + j].final, i, j, State[i*N_nodes + j].tried);
        }
        printf("\n");
    }
        printf("\n");
*/
    printf("GPU: DEBUG LOG 1 size %d\n", paramsIn - (double*)params);
    /*
     * Compute sampling probabilities 
     * first pass: \xi_i,j
     */

    //for(int i = threadIdx.x; i < N_nodes-1; i+=warp_size){
    for(int i = 0; i < N_nodes-1; i++){
        double probSum = 0.0;
        for( int j=i+1; j<N_nodes; j++){
    //printf("GPU: i=%d j=%d\n", i, j);
                
            if(!State[i*N_nodes + j].tried){
                double bg = 0.;
                Prob[i*N_nodes + j] = alpha_i*min(LogRanks[i], LogRanks[j]) +
                  alpha_m*max(LogRanks[i], LogRanks[j]);  
                
                if (Dist[i*N_nodes + j] > 0.){
                    
                    double k = Dist[i*N_nodes + j];
                    if (k >= k_max){
                        k = k_max-1;
                    }
                    
                    bg = beta * log(k/k_max) + gamma * log(1. - k/k_max);
                    
                } else {
                    bg = delta;
                }
                   
                Prob[i*N_nodes + j] = exp(Prob[i*N_nodes + j] + bg);
                /*
                 * Probability can tip over into infinite in rare cases.
                 * Return to representable number to avoid strange behavior
                 */
                if (!isfinite(Prob[i*N_nodes + j])) {
                  //printf("DEBUG: Invalid Prob[%i][%i]: %.17g\n",
                  //        i, j, Prob[i][j]);
                  if (isnan(Prob[i*N_nodes + j])) {
                    Prob[i*N_nodes + j] = DBL_MIN;
                  } else {
                    Prob[i*N_nodes + j] = DBL_MAX;
                  }
                }
                if (Prob[i*N_nodes + j] == 0.0) {
                  /*
                   * Don't want zero prob: otherwise will never be sample
                   */
                  //printf("DEBUG: Prob[%i][%i] == 0.0\n", i, j);
                  Prob[i*N_nodes + j] = DBL_MIN;
                }
                probSum += Prob[i*N_nodes + j];
            }
        }
        ProbSums[i] = probSum;
        Summa += probSum;
    }
    printf("GPU: DEBUG LOG 4\n");

    /*
     * Reduction to get summation
    int local_sum = 0.;
    for(int i = threadIdx.x; i < N_nodes-1; i+=warp_size){
        local_sum += ProbSums[i];
    }
    Summa[threadIdx.x] = local_sum;
    __syncthreads();

    while((warp_size >> 1) > 0) {
        if(threadIdx.x < warp_size) {
            Summa[threadIdx.x] += Summa[threadIdx.x + warp_size];
        }
        __syncthreads();
    }
     */

    printf("Summa = %f\n", Summa);
    /*
     * second pass: normalize
     */
    //if (isfinite(Summa[0])) {
    if (!isfinite(Summa)) {
      /*
       * Need to avoid dividing by infinity
       */
      //printf("DEBUG: infinite prob summ: %.17g\n", Summa);
      //Summa[0] = DBL_MAX;
      Summa = DBL_MAX;
    }

    //for(int i=threadIdx.x; i<N_nodes-1; i+=warp_size){
    for(int i=0; i<N_nodes-1; i++){
        for( int j=i+1; j<N_nodes; j++){
                
            if(!State[i*N_nodes + j].tried){
                //Prob[i*N_nodes + j] /= Summa[0];
                Prob[i*N_nodes + j] /= Summa;
            }
        }
        //ProbSums[i] /= Summa[0];
        ProbSums[i] /= Summa;
    }
    /*
     * Sum should be 1, since normalized
     */
    //TotalProb = 1.;
    /*
     * Kernel Benchmarking parameters
     * Uncomment to benchmark inside CUDA kernel.
     * Don't uncomment otherwise else it will lead to unnecessary console logs.
     */
    //stop = clock();
    //float time;
    //if (stop > start) {
    //    time = (float)(stop - start)/(float)SHADER_CLOCK;
    //} else {
    //    time = (float)(stop + (0xffffffff - start))/(float)SHADER_CLOCK;
    //}
    //printf("GPU: Time taken %f ms\n", time);    
    return;
}

__device__ void gemtc_sample_all_visible(void *param)
{
    return;
}

__device__ void gemtc_scicolsim(void *params)
{
    double *operation = (double*)params;

    switch((int)*operation) {
    case 1:
        gemtc_update_probabilities_all_visible(params);
        break;
    case 2:
        gemtc_sample_all_visible(params);
        break;
    };
}
