
#include <hip/hip_runtime.h>
// Not a real AppKernel
// Dummy Kernel for HPDC Paper

__device__ void MatrixMultiply(void *input)
{ 
  // calibrate for warp size
  int warp_size = 32;
  int thread = threadIdx.x % warp_size;
  
  // unbox the host parameters
  float* inputParams = (float*)input;
  int matrixWidth = inputParams[0];
  float *matrixA = inputParams+1;
  float *matrixB = matrixA + matrixWidth*matrixWidth;
  float *matrixOut = matrixA + 2*matrixWidth*matrixWidth;
  
  for (unsigned int i = thread; i < matrixWidth; i=i+32){
    for (unsigned int j = 0; j < matrixWidth; j++) {
      float sum = 0;
      for (unsigned int k = 0; k < matrixWidth; k++) {
	float a = matrixA[i * matrixWidth + k];
	float b = matrixB[k * matrixWidth + j];
	sum += a * b;
      }
      matrixOut[i * matrixWidth + j ] = sum;
    }
  }
}

