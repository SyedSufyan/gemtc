
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ void MatrixSquare(void *param)
{ 
    float *input = (float *) param;
    int warp_size=32;
    int matrixWidth = (int)input[0];
    float* matrix = input+1;
    float* matrixOut = matrix + matrixWidth*matrixWidth;
    //printf("%d\n", matrixWidth);
#if 1 
    int thread = threadIdx.x % warp_size;
        
    for (unsigned int i = thread; i < matrixWidth; i=i+32)
    {
      for (unsigned int j = 0; j < matrixWidth; j++) {
         float sum = 0;
         for (unsigned int k = 0; k < matrixWidth; k++) {
           float a = matrix[i * matrixWidth + k];
           float b = matrix[k * matrixWidth + j];
           sum += a * b;
         }
         //matrixOut[i * matrixWidth + j + (matrixWidth * matrixWidth)] = sum;
         matrixOut[i * matrixWidth + j ] = sum;
      }
   }
#endif
}
