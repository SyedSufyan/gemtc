
#include <hip/hip_runtime.h>
__device__ void ArrayMax( void* param)
{
    float* paramIn = (float*)param;
    int N = (int)(*paramIn);
    paramIn = paramIn + 1;
    float* a  = paramIn;
    float* b = a + N;
    int tid = threadIdx.x;
    int cacheIndex = threadIdx.x;
    float   temp = a[tid];
    while (tid < N) 
    {
        //temp += a[tid] * b[tid];
        //temp = a[tid] ;//>= a[tid+1] ? a[tid] : a[tid+1];
        if (temp < a[tid])
            temp = a[tid];
        tid += 32;
    }
    // set the cache values
    b[cacheIndex] = temp;
    //printf("Cache[%d]=%g\n", cacheIndex, temp);
#if 1
    // synchronize threads in this block
    //__syncthreads();

    // for reductions, threadsPerBlock must be a power of 2
    // because of the following code
    int i = 32/2;
    //if (cacheIndex < 0)
        //printf("i=%d,blockDim.x=%d,tid=%d\n",i, blockDim.x, cacheIndex);
    while (i != 0) 
    {
        if (cacheIndex < i )
        {
            //cache[cacheIndex] += cache[cacheIndex + i];
            if (b[cacheIndex] >=  b[cacheIndex + i])
                b[cacheIndex] = b[cacheIndex];
            else
                b[cacheIndex] = b[cacheIndex+i];
            //printf("i=%d,tid1=%d,blockIdx.x=%d,tid=%d,%g,%g\n",
            //       i, tid1,blockIdx.x, cacheIndex, cache[cacheIndex],cache[cacheIndex+1]);
        }
        //__syncthreads();
        i /= 2;
    }

    //if (cacheIndex == 0)
    //{
        //printf("c[blockIdx.x]:%g, :%d\n", cache[0],blockIdx.x);
    //    c[cacheIndex] = cache[0];
    //}
#endif
}
#if 0 
__device__ void VecDot( void* param)
{
    int N = 32;
    float* paramIn = (float*)param;
    float* paramInOrig = (float*)param;
    int size = (int)(*paramIn);
    paramIn = paramIn + 1;
    float* a  = paramIn;
    paramIn = paramIn + size;
    float* b = paramIn;
    paramIn = paramIn + size;
    float* c = paramIn;
    //int tid = threadIdx.x%N + blockIdx.x * blockDim.x;
    int tid = threadIdx.x%N;
    int cacheIndex = threadIdx.x%N;
    float   temp = 0;
    while (tid < size)
    {
        temp += a[tid] * b[tid];
        tid = tid + N;
        //tid += blockDim.x * gridDim.x;
    }
    // set the cache values
    c[cacheIndex] = temp;
    
#if 1     
    // synchronize threads in this block
    //__syncthreads();

    // for reductions, threadsPerBlock must be a power of 2
    // because of the following code
    //int i = blockDim.x/2;
    int i = N/2;
    while (i != 0) {
        if (cacheIndex < i)
            c[cacheIndex] += c[cacheIndex + i];
        //__syncthreads();
        i /= 2;
    }
    //__syncthreads();
    if (cacheIndex == 0)
    {
       //paramInOrig[0] = 44;
       //printf("Val: %f\n", c[0]);
    }
#endif
     //paramInOrig[0] = 44;
}
#endif
#if 0
__global__ void dot( float *a, float *b, float *c ) {
    __shared__ float cache[threadsPerBlock];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float   temp = 0;
    while (tid < N) {
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }
    
    // set the cache values
    cache[cacheIndex] = temp;
    
    // synchronize threads in this block
    __syncthreads();

    // for reductions, threadsPerBlock must be a power of 2
    // because of the following code
    int i = blockDim.x/2;
    while (i != 0) {
        if (cacheIndex < i)
            cache[cacheIndex] += cache[cacheIndex + i];
        __syncthreads();
        i /= 2;
    }

    if (cacheIndex == 0)
        c[blockIdx.x] = cache[0];
}
#endif

