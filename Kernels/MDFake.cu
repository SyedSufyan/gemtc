
#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>

/* The purpose of these microkernels is to 
offer the user a sanity check. These microkernels 
take the exact same parameters as their "real" 
implementations and perform simple modifications 
so the user can be sure the kernel is unpacking 
and modifying the parameters the correct way. */ 

__device__ void ComputeTest(void* params){
  
  //Params| &table | offset | 
  //Bytes |   8    |   4    | 
  
 void *table = *((void**)params);
 int offset = *((int*)(((void**)params)+1));
  
  //Extract all the values. 
  int np = *((int*) table);
  int nd = *(((int*) table)+1);

  int size = np * nd;

  double *mass = (double*)(((int*)table)+2);
  double *pos = mass + 1;
  double *vel = pos + size; 
  double *acc = vel + size;
  double *f = acc + size;

  double *pe = f + size;
  double *ke = pe + size;

  int j;
  int tid = threadIdx.x % 32; 
  int k = offset + tid; 
  //Compute all the potential energy and forces.     
   for(j=0; j<np; j++){
      if(k == j){ continue; }

      int index = j + k *nd;
      f[index] += 1;
      pe[index] += 1;
      ke[index] += 1;
    }
}
