
#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>

/* The purpose of these microkernels is to 
offer the user a sanity check. These microkernels 
take the exact same parameters as their "real" 
implementations and perform simple modifications 
so the user can be sure the kernel is unpacking 
and modifying the parameters the correct way. */ 

__device__ void FakeCompute(void* params){
 
  //Params | np | nd | mass |   pos  |   vel  |    f   |   pe   |   ke   |
  //Bytes  | 4  | 4  |  8   | 8*size | 8*size | 8*size | 8*size | 8*size | 
 
  int np = *((int*) params);
  int nd = *(((int*) params) + 1);

  int size = np * nd;

  double *mass = (double*)(((int*)params) + 2);
  double *pos = mass + 1; 
  double *vel = pos + size; 
  double *f = vel + size;

  double *pe = f + size;
  double *ke = pe + size;

  int i;
  for(i=0; i<size; i++){
    pos[i] = i;
    vel[i] = i * 2;
    f[i] = i * 3;
    pe[i] = i * 4;
    ke[i] = i * 5;
  }
}


__device__ void FakeInit(void *params){ 
  //Params| np | nd |  *acc  |  *vel  |  *pos  | *box | seed | 
  //Bytes | 4  |  4 | size*8 | size*8 | size*8 | nd*8 |   4  | 
  
  int *np = (int*)(params);
  int *nd = np + 1;

  int size = (*np) * (*nd);

  double *acc = ((double*)(params)) + 1;
  double *vel = acc + size; 
  double *pos = vel + size;
  double *box = pos + size; 

  int *seed = (int*)(box + *nd); //box size is ND.  
  
  int i;
  for(i=0; i<size; i++){
    acc[i] = i;
    vel[i] = i * 2;
    pos[i] = i * 3;
  }

  box[0] = 107;
  box[1] = 107;

  *np = 1;
  *nd = 2;
  *seed = 3; 
}

__device__ void FakeUpdate(void* params){
   //Params: | np | nd |  *pos  |  *vel  |   *f   |  *acc  | mass | dt | 
  //Bytes:  | 4  | 4  | 8*size | 8*size | 8*size | 8*size |  8   |  8 | 
 
  int np = *((int*)params);
  int nd = *(((int*)params) + 1);

  int size = np * nd; 

  double *pos = ((double*)(params) + 1); 
  double *vel = pos + size;
  double *f = vel + size;
  double *acc = f + size; 

  // double mass = *(acc + size);
  //double dt = *(acc + size + 1); 

  int i; 
  for(i=0; i<size; i++){
    pos[i] = i;
    vel[i] = i*2; 
    f[i] = i*3;
    acc[i] = i*4; 
  }
}
