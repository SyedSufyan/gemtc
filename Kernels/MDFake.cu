
#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>

/* The purpose of these microkernels is to 
offer the user a sanity check. These microkernels 
take the exact same parameters as their "real" 
implementations and perform simple modifications 
so the user can be sure the kernel is unpacking 
and modifying the parameters the correct way. */ 

__device__ void UnpackTable(void* p){
 
  //Params | np | nd |  mass  |   pos  |   vel  |  acc   |   f    |  pe    |    ke  |
  //Bytes  | 4  | 4  |    8   | 8*size | 8*size | 8*size | 8*size | 8*size | 8*size |

  void *params = *((void**)p);

  int *np = (int*) params;
  int *nd = ((int*) params) + 1;

  int size = (*np) * (*nd);

  double *mass = ((double*)params) + 1;
  double *pos = mass + 1; 
  double *vel = pos + size; 
  double *acc = vel + size;
  double *f = acc + size;
  double *pe = f + size;
  double *ke = pe + size; 

  int i;

  *np = 107;
  *nd = 69;
  *mass = 3.1415;

  for(i=0; i<size; i++){
    pos[i] = i;
    vel[i] = i * 2;
    acc[i] = i * 3;
    f[i] = i * 4;
    pe[i] = i * 5;
    ke[i] = i * 6;
  }
}


__device__ void FakeInit(void *params){ 
  //Params| table | box[] | seed | offset |
  
  void *table = *((void**)params); 

  //Unpack Table; 
  int *np = (int*)(table);
  int *nd = np + 1;

  int size = (*np) * (*nd);

  double *pos = ((double*)(table)) + 1;
  double *vel = pos + size; 
  double *acc = vel + size; 
  
  //Unpack Params; 
  double *box = (double*)(((void**)params)+1);
  int *seed   = (int*)(box + *nd);
  int *offset = seed + 1; 

  int i;
  for(i=0; i<size; i++){
    acc[i] = i;
    vel[i] = i * 2;
    pos[i] = i * 3;
  }

  box[0] = 107;
  box[1] = 107;

  *np = 1;
  *nd = 2;
  *seed = 3;
  *offset = 4;
}

__device__ void FakeUpdate(void* params){
   //Params: | np | nd |  *pos  |  *vel  |   *f   |  *acc  | mass | dt | 
   //Bytes:  | 4  | 4  | 8*size | 8*size | 8*size | 8*size |  8   |  8 | 
 
  int np = *((int*)params);
  int nd = *(((int*)params) + 1);

  int size = np * nd; 

  double *pos = ((double*)(params) + 1); 
  double *vel = pos + size;
  double *f = vel + size;
  double *acc = f + size; 

  // double mass = *(acc + size);
  //double dt = *(acc + size + 1); 

  int i; 
  for(i=0; i<size; i++){
    pos[i] = i;
    vel[i] = i*2; 
    f[i] = i*3;
    acc[i] = i*4; 
  }
}
