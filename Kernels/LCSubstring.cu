
#include <hip/hip_runtime.h>
#include<stdio.h>

#define MIN(a,b) (a<b?a:b)
#define MAX(a,b) (a>b?a:b)

__device__ size_t string_len(const char *str){
  const char *s;
  for(s=str; *s; ++s);
  return (s-str);
}

__device__ char* string_copy(char *dest, const char *src, size_t n){
  size_t k;
  for(k=0; k < n && src[k] != '\0'; k++){
    dest[k] = src[k];
  }
  for(; k < n; k++){
    dest[k] = '\0';
  }

  return dest; 
}

__device__ void LCSubstring(void *params){
  int *s1_size = (int*)params;
  int *s2_size = s1_size + 1;

  char *s1 = (char*)(s2_size+1);
  char *s2 = s1 + *s1_size + 1;

  char *ret = s2 + *s2_size + 1;
  *ret = NULL;

  if(*s1_size > 32 || *s2_size > 32 || *s1_size <= 0 || *s2_size <= 0){
      return;
  } 

  int max = MAX(*s1_size, *s2_size);
  int min = MIN(*s1_size, *s2_size);

  int i = threadIdx.x % max;

  int table[32][32], j;

  for(j=0; j<min; j++){
    table[i][j] = 0;
  }

  int longest = 0; 

  //TODO: This section can be parallelized. 
  for(i=0; i<max;i++){
    for(j=0; j<min;j++){
      if(s1[i] == s2[j]){
        if(i==0 || j==0){
          table[i][j] = 1;
        }
        else{
          table[i][j] = table[i-1][j-1] + 1;
        }
        if(table[i][j] > longest){
          longest = table[i][j];
          string_copy(ret, &s1[i-longest+1], longest);
        }
      }
    }
  }

}
