
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ void MatrixMultiply(void *input)
{ 
  int warp_size=32;
  int thread = threadIdx.x % warp_size;
  float* inputIn = (float*)input;
  int matrixWidth = inputIn[0];
  float *matrixA = inputIn+1;
    
  float *matrixB = matrixA + matrixWidth*matrixWidth;
  float *matrixOut = matrixA + 2*matrixWidth*matrixWidth;
    
  // Inlcude the oommented for printing the input and output
  /* 
    int i;
     
    // If master thread, print details
    printf("My thread id is: %d\n", thread);
    if(thread == 0){
      printf("Matrix Width is: %d\n", matrixWidth);
      printf("Printing Matrix A:\n");
      for(i=0; i<(matrixWidth*matrixWidth); i++){
      if (i%matrixWidth == 0 && i!=0)
        printf("\n");
	printf("%f ", matrixA[i]);
      }
    }

    // Print B

    if(thread == 0){
      printf("Matrix Width is: %d\n", matrixWidth);
      printf("Printing Matrix B:\n");
      for(i=0; i<(matrixWidth*matrixWidth); i++){
      if (i%matrixWidth == 0 && i!=0)
        printf("\n");
	printf("%f ", matrixB[i]);
      }
    }

    // Print C, i.e., The out Matrix
    if(thread == 0){
      printf("Matrix Width is: %d\n", matrixWidth);
      printf("Printing Matrix C:\n");
      for(i=0; i<(matrixWidth*matrixWidth); i++){
      if (i%matrixWidth == 0 && i!=0)
        printf("\n");
	printf("%f ", matrixOut[i]);
      }
    }
  */        
  for (unsigned int i = thread; i < matrixWidth; i=i+32)
    {
      for (unsigned int j = 0; j < matrixWidth; j++) {
	float sum = 0;
	for (unsigned int k = 0; k < matrixWidth; k++) {
	  float a = matrixA[i * matrixWidth + k];
	  float b = matrixB[k * matrixWidth + j];
	  sum += a * b;
	}
	matrixOut[i * matrixWidth + j ] = sum;
      }
    }
}
