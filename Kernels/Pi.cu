#include "hip/hip_runtime.h"
/*
 * Program to compute Pi using Monte Carlo methods 
 * TODO:-
 *     Don't pass the randomly computed values from application program
 * Instead generate then here.
 * 
 * Notes:-
 *    The program was created for presentation demo. 
 *
 * Theory:-
 * Any application using Monte Carlo simulation can be migrated to GeMTC
 * and can provide highly efficent results. This is because the amount
 * of data being passed via GeMTC will be less. 
 */
__device__ void gemtc_pi(void *params) 
{
    /*
     * Computed tid
     */
    int tid = threadIdx.x % 32;
    double* paramsIn = (double*)params;

    /*
     * Number of iteration, or number of sample
     */
    int niter = (int)paramsIn[0];

    /*
     * Output variable
     */
    paramsIn = paramsIn + 1;
    double *pi = paramsIn;

    /*
     * Get randomly computed values
     */
    paramsIn = paramsIn + 1;
    double *rand = paramsIn;

    /*
     * Fine to use shared memory here, 32 threads ony
     */
    int* count = (int *) gemtcSharedMemory();;
    count[tid] = 0;
    double x,y;
    int i; /* # of points in the 1st quadrant of unit circle */
    double z;

    for (i=tid; i<niter; i+=32) {
        x = rand[i];
        y = rand[i+niter];
        z = x*x+y*y;
        if (z<=1) {count[tid]++;}
    }
        //printf("count = %d\n", count[tid]);

    /*
     * Apply CUDA reduction to get count summation
     */
    int rounds = 32;
    while((rounds = rounds >> 1) > 0) {
        if(tid < rounds) {
            count[tid] += count[tid + rounds];
        }
        //__syncthreads();
    }

    /*
     * Compute Pi
     */
    if (tid == 0) {
        *pi=(double)count[0]/niter*4;
    }
}
