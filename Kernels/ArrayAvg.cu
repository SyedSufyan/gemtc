
#include <hip/hip_runtime.h>
__device__ void ArrayAvg( void* param)
{
    float* paramIn = (float*)param;
    int N = (int)(*paramIn);
    paramIn = paramIn + 1;
    float* a  = paramIn;
    float* b = a + N;
    int tid = threadIdx.x;
    int cacheIndex = threadIdx.x;

    float   temp = 0;
#if 1
    while (tid < N) 
    {
        temp = temp + a[tid];
        tid += 32;
    }
#endif
    // set the cache values
    b[cacheIndex] = temp;
    //printf("Cache[%d]=%g\n", cacheIndex, temp);
#if 1
    // synchronize threads in this block
    //__syncthreads();

    // for reductions, threadsPerBlock must be a power of 2
    // because of the following code
    int i = 32/2;
    //if (cacheIndex < 0)
        //printf("i=%d,blockDim.x=%d,tid=%d\n",i, blockDim.x, cacheIndex);
    while (i != 0) 
    {
        if (cacheIndex < i )
        {
            b[cacheIndex] += b[cacheIndex+i];
        }
        i /= 2;
    }

    if (cacheIndex == 0)
    {
        //printf("c[blockIdx.x]:%g, :%d\n", cache[0],blockIdx.x);
        b[0] =  b[0]/N;
    }
#endif
}
