
#include <hip/hip_runtime.h>
#include <stdio.h>
__device__ void VecAdd ( void* param1)
{
   int* mem = (int*)param1;
   int size = mem[0];
   int* A = mem+1;
   int* B = A+size;
   int* C = B+size;
   int warp_size = 32;
   int tid = threadIdx.x;
   //printf("%d,%d\n",size, threadIdx.x);
   //C[tid] = A[tid] + B[tid];
   while (tid < size)
   {
      C[tid] = A[tid] + B[tid];
      //printf("%d=%d\n", tid, C[tid]);
      tid = tid + warp_size;
   }

#if 0 
   int* A = (int*)param1;
   int* B = (int*)param1;
   int* C = (int*)param1;

   int tid = threadIdx.x + blockIdx.x*blockDim.x;

   while (tid < 32) {
      C[tid] = A[tid] + B[tid];
      tid += blockDim.x*gridDim.x;
   }
#endif
}
