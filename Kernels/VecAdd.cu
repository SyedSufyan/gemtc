
#include <hip/hip_runtime.h>
#include <stdio.h>
__device__ void VecAdd ( void* param1)
{
   float* mem = (float*)param1;
   int size = (int)mem[0];

   int As   = (int)mem[1];
   float *A = mem+2;

   float* C = A + As*size;

   int warp_size = 32;
   //C[tid] = A1[tid] + A2[tid] + A3[tid] + ...;

   int i;
   for(i=0; i<As; i++){
     float * cur = A + i*size;
     int tid = threadIdx.x%warp_size;
     while(tid<size){
       C[tid] += cur[tid];
       tid += warp_size;
     }
   }

   /*   while (tid < size)
   {
     int i, temp;
     temp=0;
     for(i=0; i<As; i++) temp += [tid]);
     C[tid]=temp;
     tid = tid + warp_size;
     }*/
}
