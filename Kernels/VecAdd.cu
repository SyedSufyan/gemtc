
#include <hip/hip_runtime.h>
#include <stdio.h>
__device__ void VecAdd ( void* param1)
{
   float* mem = (float*)param1;
   int size = (int)mem[0];

   int As   = (int)mem[1];
   float *A = mem+2;

   float* C = A + As*size;

   int warp_size = 32;
   int tid = threadIdx.x%warp_size;
   //C[tid] = A[tid] + B[tid];
   while (tid < size)
   {
     int i, temp=0;
     for(i=0;i<As;i++)temp += A[tid+i*size];
     C[tid]=temp;
     tid = tid + warp_size;
   }
}
