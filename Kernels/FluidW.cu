#include "hip/hip_runtime.h"
/*
 * Application:- Imogen's ported cukern_Wstep_hydro_uniform() kernel from "gpuImogen/gpuclass/cudaFluidW.cu"
 * Purpose:-
 *    This function calculates a first order accurate half-step of the conserved transport part of the fluid equations (CFD)
 * which is used as the predictor input to the matching TVD function. The kernel implemented is purehydro implying magnetic
 * parameters are zero.
 * 
 * The 1D segment of the fluid equations solved is
 *         | rho |         | px |
 *         | px  |         | vx px + P - bx^2 |
 *    d/dt | py  | = -d/dx | vx py - bx by |
 *         | pz  |         | vx pz - bx bz |
 *         | E   |         | vx (E+P) - bx (B dot v) |
 *
 * with auxiliary equations
 * vx = px / rho
 * P = (gamma-1)e + .5*B^2 = thermal pressure + magnetic pressure
 * e = E - .5*(p^2)/rho - .5*(B^2)
 * (The relation between internal energy e and thermal pressure is theoretically allowed to be far more complex than the ideal 
 * gas law being used). The hydro functions solve the same equations with B set to <0,0,0>.
 */

#include <stdio.h>
#include <string.h>
#include <stdarg.h>
#ifdef UNIX
#include <stdint.h>
#include <unistd.h>
#endif

/*
 * These macros are different from Imogen's macros as we can only have 32 size 
 */
#define BLOCKLEN 28
#define BLOCKLENP2 30
#define BLOCKLENP4 32

#define ALFVEN_FACTOR 1

/*
 * Shader frequency of GTX 480 
 * Better will be to deriver this in case we are simulation in a different GPU
 * But calling function to derive frequncy so many times will be still expensive
 * Tip:- Get this from the caller as input parameter.
 */
#define SHADER_CLOCK 1401000

/*
 * This function calculates a single half-step of the conserved transport part of the fluid equations
 * (CFD or MHD) which is used as the predictor input to the matching TVD function.
 */
__device__ void cukern_Wstep_hydro_uniform(void* params)
{
    /*
     * Kernel Benchmarking parameters
     * Uncomment to benchmark inside CUDA kernel.
     * Don't uncomment otherwise else it will lead to unnecessary console logs.
     */
    //uint start, stop;
    //start = clock();
    //printf("Entering thread %d\n", threadIdx.x);
    double fluidQtys[7];
#define FLUID_GAMMA   fluidQtys[0]
#define FLUID_GM1     fluidQtys[1]
#define FLUID_GG1     fluidQtys[2]
#define FLUID_MINMASS fluidQtys[3]
#define FLUID_MINEINT fluidQtys[4]

#define MHD_PRESS_B   fluidQtys[5]
#define MHD_CS_B      fluidQtys[6]

    //printf("GPU: GEBUG LOG 1\n");
    int tid = threadIdx.x % 32;
    double* paramsIn = (double*)params;

    /*
     * Get X dimension
     */
    int nx = (int)paramsIn[0];

    /*
     * Get Y dimension
     */
    paramsIn = paramsIn + 1;
    int ny = (int)paramsIn[0];

    /*
     * Get Z dimension
     */
    paramsIn = paramsIn + 1;
    int nz = (int)paramsIn[0];

    /*
     * Get rho
     */
    paramsIn = paramsIn + 1;
    double* rho = paramsIn;

    /*
     * Get E
     */
    paramsIn = paramsIn + nx*ny*nz;
    double* E = paramsIn; 

    /*
     * Get px
     */
    paramsIn = paramsIn + nx*ny*nz;
    double* px = paramsIn; 

    /*
     * Get py
     */
    paramsIn = paramsIn + nx*ny*nz;
    double* py = paramsIn; 

    /*
     * Get pz
     */
    paramsIn = paramsIn + nx*ny*nz;
    double* pz = paramsIn; 

    /*
     * Purehydro simulation hence magnetic variables are not needed.
     * We want to avoid unnecessary data transfer b/w CPU <-> GPU
     * Remember the application must also not provide these parameters
     * else offset computation will break
     */
#if 0 
    /*
     * Get bx
     */
    paramsIn = paramsIn + nx*ny*nz;
    double* bx = paramsIn; 

    /*
     * Get by
     */
    paramsIn = paramsIn + nx*ny*nz;
    double* by = paramsIn; 

    /*
     * Get bz
     */
    paramsIn = paramsIn + nx*ny*nz;
    double* bz = paramsIn; 
#endif 
    /*
     * Get Ptot
     */
    paramsIn = paramsIn + nx*ny*nz;
    double* Ptot = paramsIn; 

    /*
     * Get c_f
     */
    paramsIn = paramsIn + nx*ny*nz;
    double* Cfreeze = paramsIn; 

    /*
     * Get lambda
     */
    paramsIn = paramsIn + ny*nz;
    double lambda = paramsIn[0]; 

    /*
     * Get gamma
     */
    paramsIn = paramsIn + 1;
    double gamma = paramsIn[0]; 

    /*
     * Get rhomin
     */
    paramsIn = paramsIn + 1;
    double rhomin = paramsIn[0]; 

    /*
     * Output variable
     */

    /*
     * Get rhow
     */
    paramsIn = paramsIn + 1;
    double* rhow = paramsIn;

    /*
     * Get Ew
     */
    paramsIn = paramsIn + nx*ny*nz;
    double* Ew = paramsIn;

    /*
     * Get pxw
     */
    paramsIn = paramsIn + nx*ny*nz;
    double* pxw = paramsIn;

    /*
     * Get pyw
     */
    paramsIn = paramsIn + nx*ny*nz;
    double* pyw = paramsIn;

    /*
     * Get pzw
     */
    paramsIn = paramsIn + nx*ny*nz;
    double* pzw = paramsIn;

    /*
     * Get pressb
     */
    paramsIn = paramsIn + nx*ny*nz;
    double* pressb = paramsIn;


    //printf("GPU: GEBUG LOG 2\n");
    fluidQtys[0] = gamma;
    fluidQtys[1] = gamma-1.0;
    fluidQtys[2] = gamma*(gamma-1.0);
    fluidQtys[3] = rhomin;
    // assert     cs > cs_min
    //     g P / rho > g rho_min^(g-1)
    // (g-1) e / rho > rho_min^(g-1)
    //             e > rho rho_min^(g-1)/(g-1)
    fluidQtys[4] = pow(rhomin, gamma-1.0)/(gamma-1.0);
    fluidQtys[5] = 1.0 - .5*gamma;
    fluidQtys[6] = ALFVEN_FACTOR - .5*(gamma-1.0)*gamma;

    double lambdaqtr = 0.25 * lambda; 
 
#define FLUXLa_OFFSET 0
#define FLUXLb_OFFSET (BLOCKLENP4)
#define FLUXRa_OFFSET (2*(BLOCKLENP4))
#define FLUXRb_OFFSET (3*(BLOCKLEN+4))
    #define FLUXA_DECOUPLE(i) fluxArray[FLUXLa_OFFSET+tid] = q_i[i]*C_f - w_i; fluxArray[FLUXRa_OFFSET+tid] = q_i[i]*C_f + w_i;
    #define FLUXB_DECOUPLE(i) fluxArray[FLUXLb_OFFSET+tid] = q_i[i]*C_f - w_i; fluxArray[FLUXRb_OFFSET+tid] = q_i[i]*C_f + w_i;

    #define FLUXA_DELTA lambdaqtr*(fluxArray[FLUXLa_OFFSET+tid] - fluxArray[FLUXLa_OFFSET+tid+1] + fluxArray[FLUXRa_OFFSET+tid] - fluxArray[FLUXRa_OFFSET+tid-1])
    #define FLUXB_DELTA lambdaqtr*(fluxArray[FLUXLb_OFFSET+tid] - fluxArray[FLUXLb_OFFSET+tid+1] + fluxArray[FLUXRb_OFFSET+tid] - fluxArray[FLUXRb_OFFSET+tid-1])

#define momhalfsq momhalfsq

    double C_f, velocity;
    double q_i[3];
    double w_i;
    double velocity_half;

    double* fluxArray = (double *) gemtcSharedMemory();
    //double fluxArray[4*BLOCKLENP4];

    double* freezeSpeed = fluxArray + 4*(BLOCKLENP4);
    //double freezeSpeed[BLOCKLENP4];
    //return;

    /*
     * Traverse 3-D data and solve the equations
     */
    for(int i = 0; i < ny; i++) {
        for(int j = 0; j < nz; j++) { 
            //printf("GPU: Iteration %d\n", j);
            freezeSpeed[tid] = 0;

            /*
             * Do NOT memset(), instead chop the setting functions for 32 threads to process
             */
            for(int k = tid; k < 4*(BLOCKLENP4); k+=32) {
                fluxArray[k] = 0.0;
            }
            //if(tid == 0) memset(fluxArray, 0, sizeof(double)*4*(BLOCKLENP4));

            /*
             * Step 0 - obligatory setup 
             */
            int I0 = nx*(i + j*ny);
            int Xindex = (tid-2);
            int Xtrack = Xindex;
            Xindex += nx*(tid < 2);

            int x; /* = Xindex % nx; */
            bool doIflux = (tid > 1) && (tid < BLOCKLEN+2);

            /*
             * Step 1 - calculate W values 
             */
            C_f = Cfreeze[i + j*ny];

            double locPsq;
            double locE;

            /*
             * int stopme = (blockIdx.x == 0) && (blockIdx.y == 0); // For cuda-gdb
             */

            while(Xtrack < nx+2) {
                x = I0 + (Xindex % nx);

           //printf("GPU DEBUGING 1 %d \n", tid);
                /*
                 * rho q_i[0] = inputPointers[0][x];  Preload these out here 
                 * E q_i[1] = inputPointers[1][x];  So we avoid multiple loops 
                 * px q_i[2] = inputPointers[2][x];  over them inside the flux loop 
                 * py q_i[3] = inputPointers[3][x];  
                 * pz q_i[4] = inputPointers[4][x];  
                 */

                q_i[0] = rho[x];
                q_i[1] = px[x];
                q_i[2] = E[x];
                locPsq = Ptot[x];

                velocity = q_i[1] / q_i[0];

                w_i = velocity*(q_i[2]+locPsq); /* E flux = v*(E+P) */
                FLUXA_DECOUPLE(2)
                w_i = (velocity*q_i[1] + locPsq); /* px flux = v*px + P */
                FLUXB_DECOUPLE(1)
                //__syncthreads();

           //printf("GPU DEBUGING 2 %d \n", tid);
                if(doIflux && (Xindex < nx)) {
                   locE = q_i[2] - FLUXA_DELTA; /* Calculate Ehalf */
                   velocity_half = locPsq = q_i[1] - FLUXB_DELTA; /* Calculate Pxhalf */
                   pxw[x] = locPsq; /* store pxhalf */
                }
                //__syncthreads();
           //printf("GPU DEBUGING 3 %d \n", tid);

               locPsq *= locPsq; /* store p^2 in locPsq */

               q_i[0] = py[x];
               q_i[2] = pz[x];
               w_i = velocity*q_i[0]; /* py flux = v*py */
               FLUXA_DECOUPLE(0)
               w_i = velocity*q_i[2]; /* pz flux = v pz */
               FLUXB_DECOUPLE(2)

               //__syncthreads();

           //printf("GPU DEBUGING 4 %d \n", tid);
               if(doIflux && (Xindex < nx)) {
                   q_i[0] -= FLUXA_DELTA;
                   locPsq += q_i[0]*q_i[0];
                   pyw[x] = q_i[0];
                   q_i[2] -= FLUXB_DELTA;
                   locPsq += q_i[2]*q_i[2]; /* Finished accumulating p^2 */
                   pzw[x] = q_i[2];
               }
               //__syncthreads();

               q_i[0] = rho[x];
               w_i = q_i[1]; /* rho flux = px */
               FLUXA_DECOUPLE(0)
               //__syncthreads();
           //printf("GPU DEBUGING 5 %d \n", tid);

               if(doIflux && (Xindex < nx)) {
                   q_i[0] -= FLUXA_DELTA; /* Calculate rho_half */
                   //      outputPointers[0][x] = q_i[0];
                   q_i[0] = (q_i[0] < FLUID_MINMASS) ? FLUID_MINMASS : q_i[0]; /* Enforce minimum mass density */
                   rhow[x] = q_i[0];

                   velocity_half /= q_i[0]; /* calculate velocity at the halfstep for doing C_freeze */

        
                   locPsq = (locE - .5*(locPsq/q_i[0])); /* Calculate epsilon = E - T */
                   //      P[x] = FLUID_GM1*locPsq; /* Calculate P = (gamma-1) epsilon */

                   // For now we have to store the above before fixing them so the original freezeAndPtot runs unperturbed
                   // but assert the corrected P, C_f values below to see what we propose to do.
                   // it should match the freezeAndPtot very accurately.

                   // assert   cs^2 > cs^2(rho minimum)
                   //     g P / rho > g rho_min^(g-1) under polytropic EOS
                   //g(g-1) e / rho > g rho_min^(g-1)
                   //             e > rho rho_min^(g-1)/(g-1) = rho FLUID_MINEINT
                   if(locPsq < q_i[0]*FLUID_MINEINT) {
                       locE = locE - locPsq + q_i[0]*FLUID_MINEINT; // Assert minimum E = T + epsilon_min
                       locPsq = q_i[0]*FLUID_MINEINT; // store minimum epsilon.
                   } /* Assert minimum temperature */

           //printf("GPU DEBUGING 6 %d, x=%d\n", tid, x);
                   pressb[x] = FLUID_GM1*locPsq; /* Calculate P = (gamma-1) epsilon */
                   Ew[x] = locE; /* store total energy: We need to correct this for negativity shortly */

                   /* calculate local freezing speed */
                   locPsq = abs(velocity_half) + sqrt(FLUID_GG1*locPsq/q_i[0]);
                   if(locPsq > freezeSpeed[tid]) {
                       if((Xtrack > 2) && (Xtrack < (nx-3))) freezeSpeed[tid] = locPsq;
                   }
               }

               Xindex += BLOCKLEN;
               Xtrack += BLOCKLEN;
           //printf("GPU DEBUGING 7 %d \n", tid);
               //__syncthreads();
           }

           //printf("REDUCTION STARTS\n");
           /* 
            * We have a block of 32 threads. Skip computations wisely
            */
           if(tid < 16) {

               if(freezeSpeed[tid+16] > freezeSpeed[tid]) freezeSpeed[tid] = freezeSpeed[tid+16];
               //__syncthreads();
               if(tid < 8) {

                   if(freezeSpeed[tid+8] > freezeSpeed[tid]) freezeSpeed[tid] = freezeSpeed[tid+8];
                   //__syncthreads();
                   if(tid > 4) {

                       if(freezeSpeed[tid+4] > freezeSpeed[tid]) freezeSpeed[tid] = freezeSpeed[tid+4];
                       //__syncthreads();
                       if(tid < 2) {

                           if(freezeSpeed[tid+2] > freezeSpeed[tid]) freezeSpeed[tid] = freezeSpeed[tid+2];
                           //__syncthreads();
                           if(tid < 1) {
                               /*if(tid > 0) return;
                               for(x = 0; x < BLOCKLENP4; x++) { if(freezeSpeed[x] > freezeSpeed[0]) freezeSpeed[0] = freezeSpeed[x]; }
                               Cfreeze[blockIdx.x + gridDim.x * blockIdx.y] = freezeSpeed[0];*/

                               Cfreeze[i + j*ny] = (freezeSpeed[1] > freezeSpeed[0]) ? freezeSpeed[1] : freezeSpeed[0];
                           }
                       }
                   }
               }
           }
           //printf("REDUCTION STOPS\n");
       }
   }
   //printf("Exiting thread %d\n", threadIdx.x);
    /*
     * Kernel Benchmarking parameters
     * Uncomment to benchmark inside CUDA kernel.
     * Don't uncomment otherwise else it will lead to unnecessary console logs.
     */
    //stop = clock();
    //float time;
    //if (stop > start) {
    //    time = (float)(stop - start)/(float)SHADER_CLOCK;
    //} else {
    //    time = (float)(stop + (0xffffffff - start))/(float)SHADER_CLOCK;
    //}
    //printf("Time taken %f ms\n", time);    
}

