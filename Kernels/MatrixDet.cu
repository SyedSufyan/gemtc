
#include <hip/hip_runtime.h>
#include <stdio.h>


__device__ void MatrixDeterminant(void *param)
{ 
    float *input = (float *) param;
    int warp_size=32;
    int n = (int)input[0];
    float* matrix = input+1;
    int thread = threadIdx.x % warp_size;
    float value;
    float *det = input + n*n;    
    if(n < 1){
    //Error return 0
    value = 0; 
    }
    else {
    if(n==1) 
     value = matrix[0];
    else if(n==2) 
     value =  matrix[0] * matrix[3] - matrix[2] * matrix[1];
     

    }
    *det = value;
    
}

