
#include <hip/hip_runtime.h>
#include <stdio.h>
//__device__ float Determinant(float *a,int n,float *temp);
__device__ __shared__ float result[3];
__device__ void MatrixDeterminant(void *param)
{ 
    float *input = (float *) param;
    int warp_size=32;
    int n = (int)input[0];
    float* matrix = input+1;
    int thread = threadIdx.x % warp_size;
    float value =0;
    float *det = matrix +n*n;   
    if(n < 1){
    //Error return 0
    value = 0; 
    }
    else {
    if(n==1) 
     value = matrix[0];
    else if(n==2) 
     value =  matrix[0] * matrix[3] - matrix[2] * matrix[1];
    else if (n==3){
      if(thread < 3){
      result[thread] = pow(-1.0,thread) *(matrix[thread]*(matrix[1*n + (thread+1)%3]*matrix[2*n + (thread+2)%3] - matrix[1*n + (thread+2)%3]*matrix[2 *n + (thread+1)%3]));
    } 
    }
    else
	value = 0;//This program works only for n=1 to 3   
    }
    if(n==3 && thread ==0)
    {
     for(int i=0; i < n; i++)
     {
     value = value + result[i];
     }
    *det = value;
    }
   else if(n<3) 
    *det = value;
    
}

//Recursive function not working 
/*
__device__ float Determinant(float *a,int n,float *m)
{

   int i,j,j1,j2;
   float det = 0;
   printf("%dInput\n",n);
   if (n < 1) { * Error 

   } else if (n == 1) { /* Shouldn't get used 
      det = a[0];
   } else if (n == 2) {
      det =  a[0] * a[3] - a[2] * a[1];
   } else {
      det = 0;
      for (j1=0;j1<n;j1++) {
  //       m = (float *)malloc((n-1)*(n-1) * sizeof(float));
        // for (i=0;i<n-1;i++)
          //  m[i] = (float *)malloc((n-1)*sizeof(float));
         for (i=1;i<n;i++) {
            j2 = 0;
            for (j=0;j<n;j++) {
               if (j == j1)
                  continue;
               m[(i-1)*n+j2] = a[i * n + j];
               j2++;
            }
         }
         det += pow(-1.0,1.0+j1+1.0) * a[j1] * Determinant(m,n-1,a);
	 printf("%f Intermidiate det\n", det);
        // free(m);
      }
   }
   return(det);
}*/
