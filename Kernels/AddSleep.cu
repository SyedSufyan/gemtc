
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ int addSleep(void *p_us_time)
{ 
    //This method will sleep for clockRate*kernel_time many clock ticks
    // which is equivalent to sleeping for kernel_time microseconds
    int *time = (int *) p_us_time;


    float AddPerUs = 17.69911504424; //Ben
    //float AddPerUs = 9.89759943623274; //Scott Mainh.cu
    //float AddPerUs = 18.3952025; //Scott Main.c
    //float AddPerUs = 1; // Test

    int adds = (*time)*AddPerUs;

    int save_time = *time;

    //    int temp=0;
    //while(temp<adds){
      //   temp++;
       //}
    while(adds>0){
       adds--;
       (*time)--;
    }

    *time = save_time;

    return *time;
}
