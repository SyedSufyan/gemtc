#include "hip/hip_runtime.h"
#include <stdio.h>

__device__ int hack(int *result){
  *result = (*result)-1;
  return *result;
}

__device__ int addSleep(void *p_us_time)
{ 
    //This method will sleep for clockRate*kernel_time many clock ticks
    // which is equivalent to sleeping for kernel_time microseconds
    int *time = (int *) p_us_time;


    float AddPerUs = 17.69911504424; //Ben
    //float AddPerUs = 9.89759943623274; //Scott Mainh.cu

    //float AddPerUs = 18.3952025; //Scott Main.c
    //float AddPerUs = 1; // Test

    /* //Regular Addsleep
    int adds = (*time)*AddPerUs;

    int save_time = *time;

    while(adds>0){
       adds--;
       (*time)--;
    }
    *time = save_time;
    */
    //Shared Memory AddSleep
    int *shared = (int *) gemtcSharedMemory();
    *shared = (*time)*AddPerUs;

    int save_time = *time;

    while((*shared)>0){
       (*shared)--;
       (*time)--;
    }
    *time = save_time;
    
    return *time;
}
