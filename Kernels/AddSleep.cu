
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ int hack(int *result){
  *result = (*result)-1;
  return *result;
}

__device__ int addSleep(void *p_us_time)
{ 
    //This method will sleep for clockRate*kernel_time many clock ticks
    // which is equivalent to sleeping for kernel_time microseconds
    int *time = (int *) p_us_time;

    // float AddPerUs = 17.69911504424; //Ben
    float AddPerUs = 9.89759943623274; //Scott Mainh.cu
    //float AddPerUs = 18.3952025; //Scott Main.c
    //float AddPerUs = 1; // Test

    int adds = (*time)*AddPerUs;

    /*    
    int temp=0;
    while(temp<adds){
         temp++;
       }
    */
    /*
    int save_time = *time;

    while(adds>0){
       adds = adds-1;
       *time = (*time)-1;
    }
    *time = save_time;
    */
    while(adds>0){
       adds = hack(&adds);
    }
    return *time;
}
