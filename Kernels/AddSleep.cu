
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ int addSleep(void *p_us_time)
{ 
    //This method will sleep for clockRate*kernel_time many clock ticks
    // which is equivalent to sleeping for kernel_time milliseconds
    int time = *((int *) p_us_time);

    //float AddPerUs = 10.26188; //Ben
    //        float AddPerUs = 9.89759943623274; //Scott Mainh.cu
    float AddPerUs = 18.3952025; //Scott Main.c
    //    float AddPerUs = 1; // Test

    float adds = time*AddPerUs;


    int temp=0;
    while(temp<adds){
       temp++;
    }
    return temp;
}
