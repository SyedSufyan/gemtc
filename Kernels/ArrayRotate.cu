#include "hip/hip_runtime.h"
/*
 * Application:- Imogen
 * Purpose:-
 *     To perform array rotation.
 */

/*
 * Maximum size of the array which can be entertained is 32*32
 * In case more thant that is needed we need to increase the below
 * definition
 */
/*
 * Transpose 2D array
 */
__device__ void ArrayTranspose2D(void *params)
{
    /*
     * Declare shared array, this is shared between all threads
     */
    double *tmp = (double *) gemtcSharedMemory();

    /*
     * Get input parameters
     */
    double* paramsIn = (double*)params;

    /*
     * First argument is basis of getting into this function
     */
    paramsIn = paramsIn + 1;
    int nx = (int)paramsIn[0];
    
    paramsIn = paramsIn + 1;
    int ny = (int)paramsIn[0];

    paramsIn = paramsIn + 1;
    double* src = (double*)paramsIn;

    paramsIn = paramsIn + nx*ny;
    double* dst = (double*)paramsIn;

    /*
     * CUDA Threads
     */
    int warp_size = 32;
    int tid = threadIdx.x % warp_size;

    /*
     * Local parameters
     */
    int myx    = tid;
    int myy    = threadIdx.y;
    int myAddr = myx*ny + myy;

    /*
     * Each thread has to copy data from source into shared memory 
     */
    while (myx < nx) {
        myy = threadIdx.y;
        while (myy < ny) {
            myAddr = myx*ny + myy;
            tmp[myy*ny + myx] = src[myAddr];
            myy++;
        }
        myx += warp_size;
    }

    /*
     * No need to sync within warp? Sync up
     * http://stackoverflow.com/questions/10205245/cuda-syncthreads-usage-within-a-warp ?
     */
    //__syncthreads();
#if 1
    myx    = tid;
    myy    = threadIdx.y;
    myAddr = myx*ny + myy;

    /*
     * Transpose using the shared memory 
     */
    while (myx < nx) {
        myy = threadIdx.y;
        while (myy < ny) {
            myAddr = myx*ny + myy;
            dst[myAddr] = tmp[myx*ny + myy];
            myy++;
        }
        myx += warp_size;
    }
#endif
}

__device__ void ArrayExchangeY(void *params)
{
    /*
     * Declare shared array, this is shared between all threads
     */
    double *tmp = (double *) gemtcSharedMemory();

    /*
     * Get input parameters
     */
    double* paramsIn = (double*)params;

    /*
     * First argument is basis of getting into this function
     */
    paramsIn = paramsIn + 1;
    int nx = (int)paramsIn[0];
    
    paramsIn = paramsIn + 1;
    int ny = (int)paramsIn[0];

    paramsIn = paramsIn + 1;
    int nz = (int)paramsIn[0];

    paramsIn = paramsIn + 1;
    double* src = (double*)paramsIn;

    paramsIn = paramsIn + nx*ny;
    double* dst = (double*)paramsIn;

    /*
     * CUDA Threads
     */
    int warp_size = 32;
    int tid = threadIdx.x % warp_size;

    /*
     * Local parameters
     */
    int myx       = tid;
    int myy       = threadIdx.y;
    int myz       = 0;
    int myAddr    = myx + nx*myy;

    /*
     * Each thread has to copy data from source into shared memory 
     */
    while (myz < nz) {
       /*
        * For each z dimension's iteration, save data into 
        * temporary shared memory
        */
        myx = tid;
        while (myx < nx) {
            myy = threadIdx.y;
            while (myy < ny) {
                myAddr = myx + nx*myy + (nx*ny)*(myz);
                *((double*)tmp + myy*ny + myx) = src[myAddr];
                myy++;
            }
            myx += warp_size;
        }

        /*
         * Synchronize
         */
        //__syncthreads();

        myx = tid;
        while (myx < nx) {
            myy = threadIdx.y;
            while (myy < ny) {
                myAddr = myx + nx*myy + (nx*ny)*(myz);
                dst[myAddr] = *((double*)tmp + myx*ny + myy);
                myy++;
            }
            myx += warp_size;
        }
        /*
         * Synchronize
         */
        //__syncthreads();

        myz++;
    }
}

__device__ void ArrayExchangeZ(void *params)
{
    /*
     * Declare shared array, this is shared between all threads
     */
    double *tmp = (double *) gemtcSharedMemory();

    /*
     * Get input parameters
     */
    double* paramsIn = (double*)params;

    /*
     * First argument is basis of getting into this function
     */
    paramsIn = paramsIn + 1;
    int nx = (int)paramsIn[0];
    
    paramsIn = paramsIn + 1;
    int ny = (int)paramsIn[0];

    paramsIn = paramsIn + 1;
    int nz = (int)paramsIn[0];

    paramsIn = paramsIn + 1;
    double* src = (double*)paramsIn;

    paramsIn = paramsIn + nx*ny;
    double* dst = (double*)paramsIn;

    /*
     * CUDA Threads
     */
    int warp_size = 32;
    int tid = threadIdx.x % warp_size;

    /*
     * Local parameters
     */
    int myx       = tid;
    int myz       = tid + threadIdx.y;
    int myy       = 0;
    int myAddr    = myx + nx*myy;

    /*
     * Each thread has to copy data from source into shared memory 
     */
    while (myy < ny) {
       /*
        * For each y dimension's iteration, save data into 
        * temporary shared memory
        */
        myx = tid;
        while (myx < nx) {
            myz = tid + threadIdx.y;
            while (myz < nz) {
                myAddr = myx + nx*myy + (nx*ny)*(myz);
                *((double*)tmp + myy*ny + myx) = src[myAddr];
                myz++;
            }
            myx += warp_size;
        }

        /*
         * Synchronize
         */
        //__syncthreads();

        myx = tid;
        while (myx < nx) {
            myz = tid + threadIdx.y;
            while (myz < nz) {
                myAddr = myx + nx*myy + (nx*ny)*(myz);
                dst[myAddr] = *((double*)tmp + myx*ny + myy);
                myz++;
            }
            myx += warp_size;
        }
       
        /*
         * Synchronize
         */
        //__syncthreads();

        myz++;
    }
}

__device__ void ArrayRotate(void *params)
{
    double *operation = (double*)params;

    switch((int)*operation) {
    case 1:
        ArrayExchangeY(params);
        break;
    case 2:
        ArrayExchangeZ(params);
        break;
    case 3:
        ArrayTranspose2D(params);
        break;
    };
}
