#include "hip/hip_runtime.h"
/*
 * Application:- Imogen's ported kernel from "gpuImogen/gpuclass/cudaArrayRotate.cu"
 * Purpose:-
 *     To perform array rotation.
 */

/*
 * Shader frequency of GTX 480 
 * Better will be to deriver this in case we are simulation in a different GPU
 * But calling function to derive frequncy so many times will be still expensive
 * Tip:- Get this from the caller as input parameter.
 */
#define SHADER_CLOCK 1401000

/*
 * Transpose 2D array
 */
__device__ void ArrayTranspose2D(void *params)
{
    int j;

    /*
     * Kernel Benchmarking parameters
     * Uncomment to benchmark inside CUDA kernel.
     * Don't uncomment otherwise else it will lead to unnecessary console logs.
     */
    //clock_t start, stop;
    //start = clock();
    //printf("Initiating\n");

    /*
     * Declare shared array, this is shared between all threads
     * Do NOT use it, it will restrict the maximum data-set on which we can work.
     * Till mid-semester report we were using it.
     */
    //double *tmp = (double *) gemtcSharedMemory();

    /*
     * Get input parameters, Unpack parameters
     */
    double* paramsIn = (double*)params;

    /*
     * First argument is basis of getting into this function
     * Remmeber that while unpacking we will be deriving offsets.
     * These offsets are the positions where we will find input parameters.
     * Be careful on pointer arithmetic. If you get segmentation-faul this is 
     * the first place to check for.
     */
    paramsIn = paramsIn + 1;
    
    /*
     * Get nx (x-dimension)
     */
    int nx = (int)paramsIn[0];
    
    /*
     * Get ny (y-dimension)
     */
    paramsIn = paramsIn + 1;
    int ny = (int)paramsIn[0];

    /*
     * Get source 2D array
     */
    paramsIn = paramsIn + 1;
    double* src = (double*)paramsIn;

    /*
     * Get destination 2D array
     */
    paramsIn = paramsIn + nx*ny;
    double* dst = (double*)paramsIn;

    /*
     * CUDA Threads
     */
    int warp_size = 32;
    /*
     * Get thread tid (we should keep it from 0-31 range only)
     */
    int tid = threadIdx.x % warp_size;

    /*
     * Fill the destination 2-D array in rotated fashion.
     * One thread incharge of 1 row, hence skip next 32 entries 
     * for each thread.
     */
    for (; tid < nx; tid+=warp_size) {
        for (j = 0; j < ny; j++) {
            dst[tid*ny + j] = src[j*nx + tid];
        }
    }

    /*
     * Kernel Benchmarking parameters
     * Uncomment to benchmark inside CUDA kernel.
     * Don't uncomment otherwise else it will lead to unnecessary console logs.
     */
    //printf("DONE\n");
    //stop = clock();
    //float time = (float)(stop - start)/(float)SHADER_CLOCK;
    //printf("Time taken %f ms\n", time);    
}

__device__ void ArrayExchangeY(void *params)
{
    /*
     * Declare shared array, this is shared between all threads
     */
    double *tmp = (double *) gemtcSharedMemory();

    /*
     * Get input parameters
     */
    double* paramsIn = (double*)params;

    /*
     * First argument is basis of getting into this function
     */
    paramsIn = paramsIn + 1;
    int nx = (int)paramsIn[0];
    
    paramsIn = paramsIn + 1;
    int ny = (int)paramsIn[0];

    paramsIn = paramsIn + 1;
    int nz = (int)paramsIn[0];

    paramsIn = paramsIn + 1;
    double* src = (double*)paramsIn;

    paramsIn = paramsIn + nx*ny;
    double* dst = (double*)paramsIn;

    /*
     * CUDA Threads
     */
    int warp_size = 32;
    int tid = threadIdx.x % warp_size;

    /*
     * Local parameters
     */
    int myx       = tid;
    int myy       = threadIdx.y;
    int myz       = 0;
    int myAddr    = myx + nx*myy;

    /*
     * Each thread has to copy data from source into shared memory 
     */
    while (myz < nz) {
       /*
        * For each z dimension's iteration, save data into 
        * temporary shared memory
        */
        myx = tid;
        while (myx < nx) {
            myy = threadIdx.y;
            while (myy < ny) {
                myAddr = myx + nx*myy + (nx*ny)*(myz);
                *((double*)tmp + myy*ny + myx) = src[myAddr];
                myy++;
            }
            myx += warp_size;
        }

        /*
         * Synchronize
         */
        //__syncthreads();

        myx = tid;
        while (myx < nx) {
            myy = threadIdx.y;
            while (myy < ny) {
                myAddr = myx + nx*myy + (nx*ny)*(myz);
                dst[myAddr] = *((double*)tmp + myx*ny + myy);
                myy++;
            }
            myx += warp_size;
        }
        /*
         * Synchronize
         */
        //__syncthreads();

        myz++;
    }
}

__device__ void ArrayExchangeZ(void *params)
{
    /*
     * Declare shared array, this is shared between all threads
     */
    double *tmp = (double *) gemtcSharedMemory();

    /*
     * Get input parameters
     */
    double* paramsIn = (double*)params;

    /*
     * First argument is basis of getting into this function
     */
    paramsIn = paramsIn + 1;
    int nx = (int)paramsIn[0];
    
    paramsIn = paramsIn + 1;
    int ny = (int)paramsIn[0];

    paramsIn = paramsIn + 1;
    int nz = (int)paramsIn[0];

    paramsIn = paramsIn + 1;
    double* src = (double*)paramsIn;

    paramsIn = paramsIn + nx*ny;
    double* dst = (double*)paramsIn;

    /*
     * CUDA Threads
     */
    int warp_size = 32;
    int tid = threadIdx.x % warp_size;

    /*
     * Local parameters
     */
    int myx       = tid;
    int myz       = tid + threadIdx.y;
    int myy       = 0;
    int myAddr    = myx + nx*myy;

    /*
     * Each thread has to copy data from source into shared memory 
     */
    while (myy < ny) {
       /*
        * For each y dimension's iteration, save data into 
        * temporary shared memory
        */
        myx = tid;
        while (myx < nx) {
            myz = tid + threadIdx.y;
            while (myz < nz) {
                myAddr = myx + nx*myy + (nx*ny)*(myz);
                *((double*)tmp + myy*ny + myx) = src[myAddr];
                myz++;
            }
            myx += warp_size;
        }

        /*
         * Synchronize
         */
        //__syncthreads();

        myx = tid;
        while (myx < nx) {
            myz = tid + threadIdx.y;
            while (myz < nz) {
                myAddr = myx + nx*myy + (nx*ny)*(myz);
                dst[myAddr] = *((double*)tmp + myx*ny + myy);
                myz++;
            }
            myx += warp_size;
        }
       
        /*
         * Synchronize
         */
        //__syncthreads();

        myz++;
    }
}

/*
 * Sub-kernel selection function, Superkernel will call this function only
 */
__device__ void ArrayRotate(void *params)
{
    /*
     * Get the selection option
     */
    double *operation = (double*)params;

    switch((int)*operation) {
    /*
     * Call exchange y for selection option 1
     */
    case 1:
        ArrayExchangeY(params);
        break;
    /*
     * Call exchange z for selection option 1
     */
    case 2:
        ArrayExchangeZ(params);
        break;
    /*
     * Call exchange transpose2D for selection option 1
     */
    case 3:
        ArrayTranspose2D(params);
        break;
    };
}
