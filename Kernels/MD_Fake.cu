
#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>

__device__ void FakeCompute(void* params){
  
  //void *table = *((void**)params);
  //int offset = *((int*)(((void **)params) + 1)); 
  
  //Extract all the values. 
  int np = *((int*) params);
  int nd = *(((int*) params) +1);

  int size = np * nd;

  double *mass = (double*)(((int*)params)+2);
  double *pos = mass + 1; 
  double *vel = pos + size; 
  double *f = vel + size;

  double *pe = f + size;
  double *ke = pe + size;

  int i;
  for(i=0; i<size; i++){
    pos[i] = i;
    vel[i] = i*2;
    f[i] = i*3;
    pe[i] = i*4;
    ke[i] = i*5;
  }
}


__device__ void FakeInit(void *params){ 
  
  int *np = (int*)(params);
  int *nd = np + 1;

  int size = (*np) * (*nd);

  double *acc = ((double*)(params)) + 1;
  double *vel = acc + size; 
  double *pos = vel + size;
  double *box = pos + size; 

  int *seed = (int*)(box + *nd); //box size is ND.  
  
  int i;
  for(i=0; i<size; i++){
    acc[i] = i;
    vel[i] = i*2;
    pos[i] = i*3;
  }

  box[0] = 107;
  box[1] = 107;

  *np = 1;
  *nd = 2;
  *seed = 3; 
}
