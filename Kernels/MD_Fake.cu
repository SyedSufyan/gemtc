
#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>

__device__ void FakeCompute(void* params){
  
  //void *table = *((void**)params);
  //int offset = *((int*)(((void **)params) + 1)); 
  
  //Extract all the values. 
  int np = *((int*) params);
  int nd = *(((int*) params) +1);

  int size = np * nd;

  double *mass = (double*)(((int*)params)+2);
  double *pos = mass + 1; 
  double *vel = pos + size; 
  double *f = vel + size;

  double *pe = f + size;
  double *ke = pe + size;

  int i;
  for(i=0; i<size; i++){
    pos[i] = i;
    vel[i] = i*2;
    f[i] = i*3;
    pe[i] = i*4;
    ke[i] = i*5;
  }
}


__device__ double r8_uniform_01(int *seed){
  
  int k = *seed/127773;
  *seed = 16807 * (*seed - k * 127773) - k * 2836; 

  if(*seed < 0 ){
    *seed += 2147483647;
  }

  double r = (double) (*seed) * 4.65661275E-10; 
  return r; 
}

__device__ void FakeInit(void *params){

  //Extract all the values if they are going to be passed in. 
  int *np = (int*)params;
  int *nd = np + 1;
  int *seed = nd + 1;

  *np = 1;
  *nd = 2;
  *seed = 3; 
}
