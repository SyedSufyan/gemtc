
#include <hip/hip_runtime.h>
///////////////////////////////////////////////////
//////////// Multi-Node Kernels ///////////////////
///////////////////////////////////////////////////

__device__ void ComputeParticles_Multi(void* params){  
  
  //Extract all the values. 
  long int np = *((long int*) params);
  long int nd = *(((long int*) params)+1);

  int size = np * nd;

  double *mass = (((double*) params)+2);
  double *pos = mass + 1;
  double *vel = pos + size; 
  double *acc = vel + size;
  double *f = acc + size;
  double *pe = f + size;
  double *ke = pe + size;

  int i, j;

  double d, d2; 
  double PI2 = 3.141592653589793 / 2.0;
  double rij[3];

  //CUDA Threads
  int warp_size = 32;
  int tid = threadIdx.x%warp_size;
 
  //Compute all the potential energy and forces.
  //for(k=0; k<np; k++){
  while(tid<np){
    for(i=0; i<nd; i++){
      f[i+tid*nd] = 0.0;
    }
    
    for(j=0; j<np; j++){
      if(tid == j){ continue; }
      
      d = 0.0; 
      for(i=0; i<nd; i++){
	rij[i] = pos[tid*nd+i] - pos[j*nd+i];
	d += pow(rij[i], 2); 
      }
      
      d = sqrt(d); 
      d2 = d < PI2? d : PI2; 
      
      pe[tid] +=  0.5 * pow(sin(d2), 2);
      
      for(i=0; i<nd; i++){
	f[i+tid*nd] = f[i+tid*nd] - rij[i] *sin(1.0 * d2) / d;
      }
    }
    tid += warp_size;
  }

  int tid2 = threadIdx.x%warp_size;
  //  for(k=0;k<np;k++){
  while(tid<np){
    // compute kinetic
    for(i=0; i<nd; i++){
      ke[tid2] += vel[i+tid2*nd] * vel[i+tid2*nd];
    }
    ke[tid2] *= 0.5 * (*mass);
    tid2 += warp_size;
  }	
}
__device__ void UpdatePosVelAccel_Multi(void* params){
  
  //Unpack Table
  long int np = *((long int*)params);
  long int nd = *(((long int*)params) + 1);
  
  int size = np * nd;

  double mass = *(((double*)params) + 2); 
   
  double *pos = ((double*)params) + 3;
  double *vel = pos + size;
  double *acc = vel + size; 
  double *f = acc + size;  
  double dt = .0001; 
  int i,j;
  double rmass = 1.0 / mass;

  //O(np*nd)
  //Begin computation
  for(j=0; j<np; j++){
    for ( i = 0 ; i < nd; i ++){
      pos[i+j*nd] += vel[i+j*nd] * dt + 0.5 * acc[i+j*nd] * dt * dt;   
      vel[i+j*nd] += 0.5 * dt * (f[i+j*nd] * rmass + acc[i+j*nd]);
      acc[i+j*nd] = f[i+j*nd] * rmass;
    }
  }
}

__device__ void MDProxy(void* params){
  ComputeParticles_Multi(params);
  UpdatePosVelAccel_Multi(params);
}
