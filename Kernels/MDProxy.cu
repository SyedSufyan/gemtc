
#include <hip/hip_runtime.h>
///////////////////////////////////////////////////
//////////// Data Table Helpers ///////////////////
///////////////////////////////////////////////////

typedef struct 
{
  int np, nd;
  double *mass;
  double *pos, *vel, *acc, *f, *pe, *ke;
}Table;

__device__ Table Unpack_Table(void *params){
  
  Table data; 

  data.np = *((int*) params);
  data.nd = *(((int*) params)+1);

  int size = data.np * data.nd;

  data.mass = (double*)(((int*)params)+2);
  data.pos = data.mass + 1;
  data.vel = data.pos + size;
  data.acc = data.vel + size;
  data.f = data.acc + size;

  data.pe = data.f + size;
  data.ke = data.pe + size;

  return data;
}

///////////////////////////////////////////////////
//////////// Single Node Kernels ///////////////////
////////////////////////////////////////////////////
__device__ void ComputeParticles(void* params){  
  
  //Params| &table | offset | 
  //Bytes |   8    |   4    | 

  void *table = *((void**)params);
  Table dt = Unpack_Table(table);

  int offset = *((int*)(((void**)params)+1));

  int i, j;
  double d, d2; 
  double PI2 = 3.141592653589793 / 2.0;
  double rij[3];

  int tid = threadIdx.x % 32; 
  int k = offset + tid; 
  //Compute all the potential energy and forces.
  for(i=0; i<dt.nd; i++){
    dt.f[i+k*dt.nd] = 0.0;
  }

  for(j=0; j<dt.np; j++){
    if(k == j){ continue; }

    d = 0.0; 
    for(i=0; i<dt.nd; i++){
      rij[i] = dt.pos[k*dt.nd+i] - dt.pos[j*dt.nd+i];
      d += pow(rij[i], 2); 
      d = rij[i];
    }

    d = sqrt(d); 
    d2 = d < PI2? d : PI2; 

    dt.pe[k] +=  0.5 * pow(sin(d2), 2);
    
    for(i=0; i<dt.nd; i++){
      dt.f[i+k*dt.nd] -= rij[i] * sin(2.0 * d2) / d;
    }
  }

  for(i=0; i < dt.nd; i++){
    dt.ke[k] += dt.vel[i+k*dt.nd] * dt.vel[i+k*dt.nd];
  }

  dt.ke[k] *= 0.5 * (*dt.mass);
}

__device__ double r8_uniform_01(int *seed){

  int k = *seed/ 127773;
  *seed = 16807 * (*seed - k * 127773) - k * 2836;

  if( *seed < 0 ){
    *seed += 2147483647;
  }

  return (double)(*seed) * 4.656612875E-10; 
}

__device__ void InitParticles(void* params){
  
  //Params| &table | box[] | seed |
  //Bytes |   8    | 8*nd  |  4   |
  
  void *table = *((void**)params);
  Table dt = Unpack_Table(table); 

  //Unpack Params
  double *box = (double*)(((void**)params)+1);
  int *seed = (int*)(box + dt.nd);

  int i,j; 
  //Update values
  for ( j = 0; j < dt.np ; j++){
    for ( i = 0; i < dt.nd ; i++){
      dt.pos[i+j*dt.nd] = box[i] * r8_uniform_01(seed);
    }
  }
}

__device__ void UpdatePosVelAccel(void* params){
 
  //Params: | &table |  dt  | offset | 
  //Bytes:  |    8   |   8  |   4    |

  void *table = *((void**)params);
  Table t = Unpack_Table(table);

  double dt = *((double*)(((void**)params)+1));
  int offset = *(((int*)params) + 4);
  
  int i,j;
  double rmass = 1.0 / (*t.mass);

  int tid = threadIdx.x % 32;
  j = offset + tid; 

  //Begin computation
  for ( i = 0 ; i < t.nd; i ++){
    t.pos[i+j*t.nd] += t.vel[i+j*t.nd] * dt + 0.5 * t.acc[i+j*t.nd] * dt * dt;   
    t.vel[i+j*t.nd] += 0.5 * dt * (t.f[i+j*t.nd] * rmass + t.acc[i+j*t.nd]);
    t.acc[i+j*t.nd] = t.f[i+j*t.nd] * rmass;
    
    t.pe[i+j*t.nd] = 0.0;
    t.ke[i+j*t.nd] = 0.0;
  }
}

///////////////////////////////////////////////////
//////////// Multi-Node Kernels ///////////////////
///////////////////////////////////////////////////

__device__ void ComputeParticles_Multi(void* params){  
  
  //Extract all the values. 
  long int np = *((long int*) params);
  long int nd = *(((long int*) params)+1);

  int size = np * nd;

  double *mass = (((double*) params)+2);
  double *pos = mass + 1;
  double *vel = pos + size; 
  double *acc = vel + size;
  double *f = acc + size;
  double *pe = f + size;
  double *ke = pe + size;

  int i, j, k;

  double d, d2; 
  double PI2 = 3.141592653589793 / 2.0;
  double rij[3];
 
  //Compute all the potential energy and forces.
   for(k=0; k<np; k++){
      for(i=0; i<nd; i++){
        f[i+k*nd] = 0.0;
      }
     
      for(j=0; j<np; j++){
        if(k == j){ continue; }

        d = 0.0; 
        for(i=0; i<nd; i++){
          rij[i] = pos[k*nd+i] - pos[j*nd+i];
          d += pow(rij[i], 2); 
        }

        d = sqrt(d); 
        d2 = d < PI2? d : PI2; 

        pe[k] +=  0.5 * pow(sin(d2), 2);
        
        for(i=0; i<nd; i++){
          f[i+k*nd] = f[i+k*nd] - rij[i] *sin(1.0 * d2) / d;
        }
      }
   }
   for(k=0;k<np;k++){
     // compute kinetic
     for(i=0; i<nd; i++){
       ke[k] += vel[i+k*nd] * vel[i+k*nd];
     }
     ke[k] *= 0.5 * (*mass);
   }	
}
__device__ void UpdatePosVelAccel_Multi(void* params){
  
  //Unpack Table
  long int np = *((long int*)params);
  long int nd = *(((long int*)params) + 1);
  
  int size = np * nd;

  double mass = *(((double*)params) + 2); 
   
  double *pos = ((double*)params) + 3;
  double *vel = pos + size;
  double *acc = vel + size; 
  double *f = acc + size;  
  double dt = .0001; 
  int i,j;
  double rmass = 1.0 / mass;

  //O(np*nd)
  //Begin computation
  for(j=0; j<np; j++){
    for ( i = 0 ; i < nd; i ++){
      pos[i+j*nd] += vel[i+j*nd] * dt + 0.5 * acc[i+j*nd] * dt * dt;   
      vel[i+j*nd] += 0.5 * dt * (f[i+j*nd] * rmass + acc[i+j*nd]);
      acc[i+j*nd] = f[i+j*nd] * rmass;
    }
  }
}

__device__ void MDProxy(void* params){
  ComputeParticles_Multi(params);
  UpdatePosVelAccel_Multi(params);
}
