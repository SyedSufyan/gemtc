
#include <hip/hip_runtime.h>
__device__ void ComputeParticles(void* params){  
  
  //Params| &table | offset | 
  //Bytes |   8    |   4    | 
  
 void *table = *((void**)params);
 int offset = *((int*)(((void**)params)+1));
  
  //Extract all the values. 
  int np = *((int*) table);
  int nd = *(((int*) table)+1);

  int size = np * nd;

  double *mass = (double*)(((int*)table)+2);
  double *pos = mass + 1;
  double *vel = pos + size; 
  double *acc = vel + size;
  double *f = acc + size;

  double *pe = f + size;
  double *ke = pe + size;

  int i;
   
  double d, d2; 
  double PI2 = 3.141592653589793 / 2.0;
  double rij[3];
  
  int j;
  int tid = threadIdx.x % 32; 
  int k = offset + tid; 
  //Compute all the potential energy and forces.
      for(i=0; i<nd; i++){
        f[i+k*nd] = 0.0;
      }

      for(j=0; j<np; j++){
        if(k == j){ continue; }

        d = 0.0; 
        for(i=0; i<nd; i++){
          rij[i] = pos[k*nd+i] - pos[j*nd+i];
          d += pow(rij[i], 2); 
          d = rij[i];
        }

        d = sqrt(d); 
        d2 = d < PI2? d : PI2; 

        pe[k] +=  0.5 * pow(sin(d2), 2);
        
        for(i=0; i<nd; i++){
          f[i+k*nd] -= rij[i] * sin(2.0 * d2) / d;
        }
      }

      for(i=0; i < nd; i++){
        ke[k] += vel[i+k*nd] * vel[i+k*nd];
      }

      ke[k] *= 0.5 * (*mass);
}

__device__ double r8_uniform_01(int *seed){
  int k; 
  double r;

  k = *seed/ 127773;
  *seed = 16807 * (*seed - k * 127773) - k * 2836;

  if( *seed < 0 ){
    *seed += 2147483647;
  }

  r = (double)(*seed) * 4.656612875E-10;
  return r; 
}

__device__ void InitParticles(void* params){
  
  //Params| &table | box[] | seed | offset |  
  //Bytes |   8    | 8*nd  |  4   |   4    |
  
  void *table = *((void**)params); 

  //Unpack Table  
  int np = *((int*)table);
  int nd = *(((int*)table) + 1);
  
  int size = np * nd;

  double *pos = ((double*)table) + 2;
  double *vel = pos + size;
  double *acc = vel + size;
  double *f = acc + size;
  double *pe = f + size;
  double *ke = pe + size; 

  //Unpack Params
  double *box = (double*)(((void**)params)+1);
  int *seed = (int*)(box + nd);
  //int *offset = seed + 1; 

  int i,j; 
  //int tid = (threadIdx.x % 32) + *offset; 
  //Update values
  for ( j = 0; j < np ; j++){
    for ( i = 0; i < nd ; i++){
      pos[i+j*nd] = box[i] * r8_uniform_01(seed);
      vel[i+j*nd] = 0.0;
      acc[i+j*nd] = 0.0;
      f[i+j*nd] = 0.0;
      pe[i+j*nd] = 0.0;
      ke[i+j*nd] = 0.0;
    }
  }

}

__device__ void UpdatePosVelAccel(void* params){
 
  //Params: | &table |  dt  | offset | 
  //Bytes:  |    8   |   8  |   4    |

  void *table = *((void**)params);
  double dt = *((double*)(((void**)params)+1));
  int offset = *(((int*)params) + 4);
  
  //Unpack Table
  int np = *((int*)table);
  int nd = *(((int*)table) + 1);
  
  int size = np * nd;

  double mass = *(((double*)table) + 1); 
   
  double *pos = ((double*)table) + 2;
  double *vel = pos + size;
  double *acc = vel + size; 
  double *f = acc + size; 

  int i,j;
  double rmass = 1.0 / mass;

  int tid = threadIdx.x % 32;
  j = offset + tid; 

  //Begin computation
  for ( i = 0 ; i < nd; i ++){
    pos[i+j*nd] += vel[i+j*nd] * dt + 0.5 * acc[i+j*nd] * dt * dt;   
    vel[i+j*nd] += 0.5 * dt * (f[i+j*nd] * rmass + acc[i+j*nd]);
    acc[i+j*nd] = f[i+j*nd] * rmass; 
  }
}
