
#include <hip/hip_runtime.h>
__device__ void ComputeParticles(void* params){
  
  void *table = *((void**)params);
  int offset = *((int*)(((void **)params) + 1)); 
  
  //Extract all the values. 
  int np = *((int*) table);
  int nd = *(((int*) table)+1);

  int size = np * nd;

  double *pos = (double*)(((int*) table)+2);
  double *vel = pos + size; 
  double *mass = vel + size; 
  double *f = mass + 1;

  double *pe = f + size;
  double *ke = pe + size;

  double d, d2; 
  double PI2 = 3.141592653589793 / 2.0;
  double rij[3];
  
  int i,j;
  int tid = threadIdx.x % 32; 
  int k = offset + tid; 
  //Compute all the potential energy and forces.
      for(i=0; i<nd; i++){
        f[i+k*nd] = 0.0;
      }

      for(j=0; j<np; j++){
        if(k == j){ continue; }

        d = 0.0; 
        for(i=0; i<nd; i++){
          rij[i] = pos[k*nd+i] - pos[j*nd+i];
          d += pow(rij[i], 2); 
        }
        d = sqrt(d); 
        d2 = d < PI2? d : PI2; 

        pe[k] +=  0.5 * pow(sin(d2), 2);
        
        for(i=0; i<nd; i++){
          f[i+k*nd] -= rij[i] * sin(2.0 * d2) / d;
        }
      }

      for(i=0; i < nd; i++){
        ke[k] += vel[i+k*nd] * vel[i+k*nd];
      }

  ke[k] *= 0.5 * (*mass);
}
