
#include <hip/hip_runtime.h>
__device__ void ComputeParticles(void* params){  
  //Extract all the values. 
  int np = *((int*) params);
  int nd = *(((int*) params)+1);

  int size = np * nd;

  double *mass = (double*)(((int*)params)+2);
  double *pos = mass + 1;
  double *vel = pos + size; 
  double *f = vel + size;

  double *pe = f + size;
  double *ke = pe + size;
  
  double d, d2; 
  double PI2 = 3.141592653589793 / 2.0;
  double rij[3];
  
  int i,j;
  int tid = threadIdx.x % 32; 
  //int k = offset + tid; 
  int k = tid;
  //Compute all the potential energy and forces.
      for(i=0; i<nd; i++){
        f[i+k*nd] = 0.0;
      }

      for(j=0; j<np; j++){
        if(k == j){ continue; }

        d = 0.0; 
        for(i=0; i<nd; i++){
          rij[i] = pos[k*nd+i] - pos[j*nd+i];
          d += pow(rij[i], 2); 
          d = rij[i];
        }

        d = sqrt(d); 
        d2 = d < PI2? d : PI2; 

        pe[k] +=  0.5 * pow(sin(d2), 2);
        
        for(i=0; i<nd; i++){
          f[i+k*nd] -= rij[i] * sin(2.0 * d2) / d;
        }
      }

      for(i=0; i < nd; i++){
        ke[k] += vel[i+k*nd] * vel[i+k*nd];
      }

      ke[k] *= 0.5 * (*mass);
}

__device__ double r8_uniform_01(int *seed){
  int k; 
  double r;

  k = *seed/ 127773;
  *seed = 16807 * (*seed - k * 127773) - k * 2836;

  if( *seed < 0 ){
    *seed += 2147483647;
  }

  r = (double)(*seed) * 4.656612875E-10;
  return r; 
}

__device__ void InitParticles(void* params){
  
  //Params| &table | box[] | seed | offset |  
  //Bytes |   8    | 8*nd  |  4   |   4    |
  
  void *table = *((void**)params); 

  //Unpack Table  
  int np = *((int*)table);
  int nd = *(((int*)table) + 1);
  
  int size = np * nd;

  double *pos = ((double*)table) + 1;
  double *vel = pos + size;
  double *acc = vel + size; 

  //Unpack Params
  double *box = (double*)(((void**)params)+1);
  int *seed = (int*)(box + nd);
  //int *offset = seed + 1; 

  int i,j; 
  //int tid = (threadIdx.x % 32) + *offset; 
  //Update values
  for ( j = 0; j < np ; j++){
    for ( i = 0; i < nd ; i++){
      pos[i+j*nd] = box[i] * r8_uniform_01(seed);
      vel[i+j*nd] = 0.0;
      acc[i+j*nd] = 0.0;
    }
  }

}

__device__ void UpdatePosVelAccel(void* params){
 
  //Params: | np | nd |  *pos  |  *vel  |   *f   |  *acc  | mass | dt | 
  //Bytes:  | 4  | 4  | 8*size | 8*size | 8*size | 8*size |  8   |  8 | 

  //Extract Values 
  int np = *((int*)params);
  int nd = *(((int*)params)+1);

  int size = np * nd; 

  double *pos = ((double*)(params) + 1);
  double *vel = pos + size; 
  double *f = vel + size;
  double *acc = f + size;

  double mass = *(acc + size);
  double dt = *(acc + size + 1); 
  
  int i,j; 
  double rmass = 1.0 / mass;

  //Begin computation
  for ( j = 0; j < np ; j++){
    for ( i = 0 ; i < nd; i ++){
      pos[i+j*nd] += vel[i+j*nd] * dt + 0.5 * acc[i+j*nd] * dt * dt;   
      vel[i+j*nd] += 0.5 * dt * (f[i+j*nd] * rmass + acc[i+j*nd]);
      acc[i+j*nd] = f[i+j*nd] * rmass; 

    }
  }
}
