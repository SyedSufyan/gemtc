#include "hip/hip_runtime.h"
__device__ void* compute(void* params){
  //Extract all the values. 
  int np = *((int*) params);
  int nd = *(np + 1); 

  int size = np * nd;

  double *pos = nd + 1;
  double *vel = pos + size; 

  double mass = *(vel + size); 

  double *f = mass + 1;

  double pot = *(f + size);
  double kin = *(pot + 1);

  double ke, pe, d, d2; 
  double PI2 = 3.141592653589793 / 2.0;
  double rij[3];
  int i,j,k;

  for(k=0; k<np; k++){
    //Compute all the potential energy and forces.
      for(i=0; i<nd; i++){
        f[i+k*nd] = 0.0;
      }

      for(j=0; j<np; j++){
        if(k == j){ continue };

        d = dist( nd, pos+k*nd, pos+j*nd, rij);
        
        if( d < PI2){
          d2 = d; 
        }
        else{
          d2 = PI2;
        }

        pe = pe + 0.5 * pow( sin (d2), 2);
        
        for(i=0; i<nd; i++){
          f[i+k*nd] -= rij[i] * sin(2.0 * d2) / d;
        }
      }

      for(i=0; i < nd; i++){
        ke += vel[i+k*ned] * vel[i+k*nd];
      }
  }

  ke *= 0.5 * mass;
  //*pot = pe;
  //*kin = ke;
}
