
#include <hip/hip_runtime.h>
///////////////////////////////////////////////////
//////////// Multi-Node Kernels ///////////////////
///////////////////////////////////////////////////

__device__ void ComputeParticles_Multi(void* params){  
  
  //Extract all the values. 
  long int np = *((long int*) params);
  long int nd = *(((long int*) params)+1);

  int size = np * nd;

  double *mass = (((double*) params)+2);
  double *pos = mass + 1;
  double *vel = pos + size; 
  double *acc = vel + size;
  double *f = acc + size;
  double *pe = f + size;
  double *ke = pe + size;

  int i, j, k;

  double d, d2; 
  double PI2 = 3.141592653589793 / 2.0;
  double rij[3];
 
  //Compute all the potential energy and forces.
   for(k=0; k<np; k++){
      for(i=0; i<nd; i++){
        f[i+k*nd] = 0.0;
      }
     
      for(j=0; j<np; j++){
        if(k == j){ continue; }

        d = 0.0; 
        for(i=0; i<nd; i++){
          rij[i] = pos[k*nd+i] - pos[j*nd+i];
          d += pow(rij[i], 2); 
        }

        d = sqrt(d); 
        d2 = d < PI2? d : PI2; 

        pe[k] +=  0.5 * pow(sin(d2), 2);
        
        for(i=0; i<nd; i++){
          f[i+k*nd] = f[i+k*nd] - rij[i] *sin(1.0 * d2) / d;
        }
      }
   }
   for(k=0;k<np;k++){
     // compute kinetic
     for(i=0; i<nd; i++){
       ke[k] += vel[i+k*nd] * vel[i+k*nd];
     }
     ke[k] *= 0.5 * (*mass);
   }	
}
__device__ void UpdatePosVelAccel_Multi(void* params){
  
  //Unpack Table
  long int np = *((long int*)params);
  long int nd = *(((long int*)params) + 1);
  
  int size = np * nd;

  double mass = *(((double*)params) + 2); 
   
  double *pos = ((double*)params) + 3;
  double *vel = pos + size;
  double *acc = vel + size; 
  double *f = acc + size;  
  double dt = .0001; 
  int i,j;
  double rmass = 1.0 / mass;

  //O(np*nd)
  //Begin computation
  for(j=0; j<np; j++){
    for ( i = 0 ; i < nd; i ++){
      pos[i+j*nd] += vel[i+j*nd] * dt + 0.5 * acc[i+j*nd] * dt * dt;   
      vel[i+j*nd] += 0.5 * dt * (f[i+j*nd] * rmass + acc[i+j*nd]);
      acc[i+j*nd] = f[i+j*nd] * rmass;
    }
  }
}

__device__ void MDProxy(void* params){
  ComputeParticles_Multi(params);
  UpdatePosVelAccel_Multi(params);
}
