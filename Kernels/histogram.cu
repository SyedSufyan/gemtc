
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

__device__ void histogram(
        void *input
)
{
	uint * inputIn = (uint *) input;
        uint byteCount = inputIn[0];

        uint *d_Data = inputIn +1;
        uint *d_Histogram = d_Data + byteCount;
	//printf("Thread #: %d\n",threadIdx.x);
		int i = threadIdx.x %32;
		
                //if(i==0)
		 //printf("Thread #: %d\n",i);	
		while (i < byteCount)
		{
				//atomicAdd( &(d_Histogram[d_Data[i]]), 1 );
				d_Histogram[d_Data[i]]++;
				i+= 32;
		}

}
