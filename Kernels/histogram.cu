#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

<<<<<<< HEAD
__device__ void histogram(
        void *input
)
{
	uint * inputIn = (uint *) input;
        uint byteCount = inputIn[0];

        uint *d_Data = inputIn +1;
        uint *d_Histogram = d_Data + byteCount;
	//printf("Thread #: %d\n",threadIdx.x);
		int i = threadIdx.x %32;
		
	
		while (i < byteCount)
		{
				//atomicAdd( &(d_Histogram[d_Data[i]]), 1 );
				d_Histogram[d_Data[i]]++;
				i+= 32;
		}

=======
__device__ void histogram(void *input){
  uint * inputIn = (uint *) input;
  uint byteCount = inputIn[0];
  uint *d_Data = inputIn +1;
  uint *d_Histogram = d_Data + byteCount;
  int i = threadIdx.x %32;
  /*  
  if(i==0){
    printf("Thread #: %d\n",i);
    printf("Thread #: %d\n",threadIdx.x);
  }
  */	
  while (i < byteCount){
    //atomicAdd( &(d_Histogram[d_Data[i]]), 1 );
    d_Histogram[d_Data[i]]++;
    i+= 32;
  } 
>>>>>>> 35324e28d2a39cf81e2b4470ecc1ac70764c6034
}
