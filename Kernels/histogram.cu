
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

__device__ void histogram(void *input){
  uint * inputIn = (uint *) input;
  uint byteCount = inputIn[0];
  uint *d_Data = inputIn +1;
  uint *d_Histogram = d_Data + byteCount;
  int i = threadIdx.x %32;
  /*  
  if(i==0){
    printf("Thread #: %d\n",i);
    printf("Thread #: %d\n",threadIdx.x);
  }
  */	
  while (i < byteCount){
    //atomicAdd( &(d_Histogram[d_Data[i]]), 1 );
    d_Histogram[d_Data[i]]++;
    i+= 32;
  } 
}
