
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

__device__ void histogram(
        void *input
)
{
        uint * inputIn = (uint *) input;
        uint byteCount = inputIn[0];

        uint *d_Data = inputIn +1;
        uint *d_Histogram = d_Data + byteCount;


		int i = threadIdx.x %32;
		
	
		while (i < byteCount)
		{
				atomicAdd( &(d_Histogram[d_Data[i]]), 1 );
				i+= 32;
		}

}
