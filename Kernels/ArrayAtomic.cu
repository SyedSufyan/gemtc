
#include <hip/hip_runtime.h>
/*
 * Application:- Imogen's ported kernel from "gpuImogen/gpuclass/cudaArrayAtomic.cu" 
 * Purpose:-
 *     To perform operation on single array. Useful when parameters like density
 * are required to be kept to have certain minimum value or certain maximum value
 * or require NaNs to be replaced by zeros.
 */ 

/*
 * Shader frequency of GTX 480 
 * Better will be to deriver this in case we are simulation in a different GPU
 * But calling function to derive frequncy so many times will be still expensive
 * Tip:- Get this from the caller as input parameter.
 */
#define SHADER_CLOCK 1401000
/*
 * This will set all array elements with less than minimum 
 * threshold to value specified in input parameter
 */
__device__ void ArraySetMin(void *params)
{
    /*
     * Kernel Benchmarking parameters
     * Uncomment to benchmark inside CUDA kernel.
     * Don't uncomment otherwise else it will lead to unnecessary console logs.
     */
    //clock_t start, stop;
    //start = clock();
    //printf("Thread %d\n", threadIdx.x);
    /*
     * CUDA Threads
     */
    int warp_size = 32;

    /*
     * Get thread tid (we should keep it from 0-31 range only)
     */
    int tid = threadIdx.x % warp_size;

    /*
     * Get input parameters
     */
    double* paramsIn = (double*)params;

    /*
     * First argument is basis of getting into this function
     */
    paramsIn = paramsIn + 1;

    /*
     * Get minimum threshold parameter
     */
    double min = (double)paramsIn[0];

    /*
     * Get number of elements in the array
     */
    paramsIn = paramsIn + 1;
    int n   = (int)paramsIn[0];

    /*
     * Get the array
     */
    paramsIn       = paramsIn + 1;
    double *array  = paramsIn;

    /*
     * Loop and set values
     */
    while (tid < n) {
        if (array[tid] < min) {
            array[tid] = min;
        }
        //printf("tid = %d, n = %d\n", tid, n);
        //printf("incrementing Thread %d\n", threadIdx.x);
        /*
         * Skip next 32 entries, other threads will take care of them
         */
        tid += warp_size;
    }
    /*
     * No need of synchronization within 1 warp
     */
    //__syncthreads();
    /*
     * Kernel Benchmarking parameters
     * Uncomment to benchmark inside CUDA kernel.
     * Don't uncomment otherwise else it will lead to unnecessary console logs.
     */
    //printf("Thread %d\n", threadIdx.x);
    //stop = clock();
    //float time = (float)(stop - start)/(float)SHADER_CLOCK;
    //printf("Time taken %f ms\n", time);    
}

/*
 * This will set all array elements with greater than maximum 
 * threshold to value specified in input parameter
 */
__device__ void ArraySetMax(void *params)
{
    /*
     * CUDA Threads
     */
    int warp_size = 32;
    int tid = threadIdx.x % warp_size;

    /*
     * Get input parameters
     */
    double* paramsIn = (double*)params;

    /*
     * First argument is basis of getting into this function
     */
    paramsIn = paramsIn + 1;

    /*
     * Get maximum threshold
     */
    double max = (double)paramsIn[0];

    /*
     * Get number of array elements
     */
    paramsIn = paramsIn + 1;
    int n   = (int)paramsIn[0];

    /*
     * Get the array itself
     */
    paramsIn       = paramsIn + 1;
    double *array = paramsIn;

    /*
     * Loop and set values
     */
    while (tid < n) {
        if (array[tid] > max) {
            array[tid] = max;
        }
        /*
         * Skip next 32 entries, other threads will take care of them
         */
        tid += warp_size;
    }
}

/*
 * This will set all array elements which are not a number
 * to value specified in input parameter
 */
__device__ void ArraySetNaN(void *params)
{
    /*
     * CUDA Threads
     */
    int warp_size = 32;
    int tid = threadIdx.x % warp_size;

    /*
     * Get input parameters
     */
    double* paramsIn = (double*)params;

    /*
     * First argument is basis of getting into this function
     */
    paramsIn = paramsIn + 1;

    /*
     * Get fixed values to replace NaNs
     */
    double fixval = (double)paramsIn[0];

    /*
     * Get number of array elements
     */
    paramsIn = paramsIn + 1;
    int n   = (int)paramsIn[0];

    /*
     * Get the array itself
     */
    paramsIn       = paramsIn + 1;
    double *array = paramsIn;

    /*
     * Loop and set values
     */
    while (tid < n) {
        if (isnan(array[tid])) {
            array[tid] = fixval;
        }
        /*
         * Skip next 32 entries, other threads will take care of them
         */
        tid += warp_size;
    }
}

/*
 * Sub-kernel selection function, Superkernel will call this function only
 */
__device__ void ArrayAtomic(void *params)
{
    /*
     * Get the selection option
     */
    double *operation = (double*)params;

    switch((int)*operation) {
    /*
     * Call set min for selection option 1
     */
    case 1: ArraySetMin(params);
        break;
    /*
     * Call set max for selection option 2
     */
    case 2: ArraySetMax(params);
        break;
    /*
     * Call set NaN for selection option 3
     */
    case 3: ArraySetNaN(params);
        break;
    }
}
