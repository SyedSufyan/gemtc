
#include <hip/hip_runtime.h>
/*
 * Application:- Imogen
 * Purpose:-
 *     To perform operation on single array. Useful when parameters like density
 * are required to be kept to a minimum value or NaNs or replaced by zeros.
 */ 

/*
 * This will set all array elements with less than minimum 
 * threshold to value specified in input parameter
 */
__device__ void ArraySetMin(void *params)
{
    /*
     * CUDA Threads
     */
    int warp_size = 32;
    int tid = threadIdx.x % warp_size;

    /*
     * Get input parameters
     */
    double* paramsIn = (double*)params;

    /*
     * First argument is basis of getting into this function
     */
    paramsIn = paramsIn + 1;

    double min = (double)paramsIn[0];

    paramsIn = paramsIn + 1;
    int n   = (int)paramsIn[0];

    paramsIn       = paramsIn + 1;
    double *array  = paramsIn;

    while (tid < n) {
        if (array[tid] < min) {
            array[tid] = min;
        }
        tid += warp_size;
    }
}

/*
 * This will set all array elements with greater than maximum 
 * threshold to value specified in input parameter
 */
__device__ void ArraySetMax(void *params)
{
    /*
     * CUDA Threads
     */
    int warp_size = 32;
    int tid = threadIdx.x % warp_size;

    /*
     * Get input parameters
     */
    double* paramsIn = (double*)params;

    /*
     * First argument is basis of getting into this function
     */
    paramsIn = paramsIn + 1;

    double max       = (double)paramsIn[0];

    paramsIn = paramsIn + 1;
    int n   = (int)paramsIn[0];

    paramsIn       = paramsIn + 1;
    double *array = paramsIn;

    while (tid < n) {
        if (array[tid] > max) {
            array[tid] = max;
        }
        tid += warp_size;
    }
}

/*
 * This will set all array elements which are not a number
 * to value specified in input parameter
 */
__device__ void ArraySetNaN(void *params)
{
    /*
     * CUDA Threads
     */
    int warp_size = 32;
    int tid = threadIdx.x % warp_size;

    /*
     * Get input parameters
     */
    double* paramsIn = (double*)params;

    /*
     * First argument is basis of getting into this function
     */
    paramsIn = paramsIn + 1;

    double fixval    = (double)paramsIn[0];

    paramsIn = paramsIn + 1;
    int n   = (int)paramsIn[0];

    paramsIn       = paramsIn + 1;
    double *array = paramsIn;

    while (tid < n) {
        if (isnan(array[tid])) {
            array[tid] = fixval;
        }
        tid += warp_size;
    }
}

__device__ void ArrayAtomic(void *params)
{
    double *operation = (double*)params;

    switch((int)*operation) {
    case 1: ArraySetMin(params);
        break;
    case 2: ArraySetMax(params);
        break;
    case 3: ArraySetNaN(params);
        break;
    }
}
