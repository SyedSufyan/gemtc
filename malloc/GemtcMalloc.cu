#include "hip/hip_runtime.h"
#include<stdlib.h>
#include<hip/hip_runtime.h>

/*
This file contains the SubAllocator for Gemtc

The purpose of the suballocator is to allow efficient mallocs and
frees of global memory from the Host

hipMalloc gets poor performance, so this will use hipMalloc to
get a large block of memory and sub-divide it as mallocs are needed


Model:
A circular linked list of free memory blocks is kept in host memory
The elements of this list are kept in order of increasing device addr

When memory is allocated, a header is written infront of it with its
size information. Beyond this, mallocs are not tracked.

When memory is malloced, an existing node will removed or have its
size reduced.

When memory is freed, an existing is extended if its adjacent to the
new memory, or a new node is created and inserted in order.

NOTE: Currently malloc and free must search the linked list, which 
      may be O(n). A future work is to change the data structure to
      get O(log(n)) scaling.
 */



pthread_mutex_t memoryListLock;

typedef struct memoryPointer MemoryPointer;

//The struct for a node of the memory linked list
struct memoryPointer{
  MemoryPointer *ptr;
  unsigned size;
  unsigned *data;
};

static MemoryPointer base;

//A pointer to somewhere in the list
static MemoryPointer *freep = NULL;

//All memory is allocated in mulitples of this amount
int CHUNK_SIZE=1024;
//This is the size of the header infront of each malloc
int headerSize=8;

//Minimum size that will be malloced from CUDA when more
//  memory is needed
int MIN_BULK_AMOUNT = 10000000; //~10million


//Adds a node to the list of memory, or merges it with a node that is
//  its precessor or succesor
void gemtcAddList(MemoryPointer *bp){
  MemoryPointer *p;
  //freep start of list of free memory
  for(p = freep; (bp->data < p->data || bp->data > (p->ptr)->data); p = p->ptr){
     if(p->data >= (p->ptr)->data && (bp->data > p->data || bp->data < (p->ptr)->data)){
       break;
     }
  }
  if( (((char *)bp->data) + bp->size) == (char *)p->ptr->data){
    bp->size += (p->ptr)->size;
    bp->ptr = (p->ptr)->ptr;
    if(p->ptr != &base){
      free(p->ptr);
    }
  }else
    bp->ptr = p->ptr;

  if( (((char *)p->data) + p->size) == (char *)bp->data){
    p->size += bp->size;
    p->ptr = bp->ptr;
    free(bp);
  }else{
    p->ptr = bp;
  }
  freep = p;
}

//This will read the size information of the memory, create a node
// for the memory and hand it off the gemtcAddList(..)
void gemtcFree(void *loc){
  pthread_mutex_lock(&memoryListLock);

  loc = ((void *)(((char *)loc)-headerSize));
  MemoryPointer *v = (MemoryPointer *) malloc(sizeof(MemoryPointer));
  cudaSafeMemcpy(&v->size, loc, sizeof(unsigned), hipMemcpyDeviceToHost,
                 stream_dataOut, "Reading size of freed memory");
  
  v->data = (unsigned *) loc;

  gemtcAddList(v);

  pthread_mutex_unlock(&memoryListLock);
}

//This is called by malloc when more memory is needed
//It will call hipMalloc then hand it off to gemtcAddList to
//  Add the new memory to the list
static MemoryPointer *morecore(unsigned nu){
  void *cp;
  MemoryPointer *up = (MemoryPointer *)malloc(sizeof(MemoryPointer));
  if (nu < MIN_BULK_AMOUNT) nu = MIN_BULK_AMOUNT;
  hipMalloc(&cp, nu);

  up->data = (unsigned *)cp;

  up->size = nu;

  gemtcAddList(up);
  return freep;
}

//Searches the list for a large enough block of consecutive memory
//If none are found, manycore() wil be called to get a large 
//  enough block
void *gemtcMalloc(unsigned nbytes){
  pthread_mutex_lock(&memoryListLock);
  MemoryPointer *p, *prevp;
  if ((prevp = freep)==NULL){
    base.ptr = freep = prevp = &base;
    base.size = 0;
  }
  nbytes+=headerSize;
  if(nbytes%CHUNK_SIZE!=0)nbytes+=(CHUNK_SIZE-nbytes%CHUNK_SIZE);
  char *loc;
  for(p = prevp->ptr; ;prevp = p, p = p->ptr){
    if(p->size >= nbytes){
      if(p->size == nbytes){
        prevp->ptr = p->ptr;
        loc = (char *) p->data;
        free(p);
      }else{
        p->size -= nbytes;
        loc =((char *) p->data)+p->size;
      }
      freep = prevp;

      cudaSafeMemcpy(loc,&nbytes,sizeof(unsigned),hipMemcpyHostToDevice,
                     stream_dataIn, "Writing size on newly allocated memory");
      pthread_mutex_unlock(&memoryListLock);
 
      return (void *)(loc+headerSize);
    }
    if (p == freep){
      if((p = morecore(nbytes))==NULL){
        pthread_mutex_unlock(&memoryListLock);
	return NULL;
      }
    }
  }
}

