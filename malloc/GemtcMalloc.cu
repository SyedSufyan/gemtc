#include<stdlib.h>
#include<hip/hip_runtime.h>

typedef struct memoryPointer MemoryPointer;

struct memoryPointer{
  MemoryPointer *ptr;
  unsigned size;
  unsigned *data;
};

static MemoryPointer base;
static MemoryPointer *freep = NULL;

int CHUNK_SIZE=2048;
int headerSize=16;
int MIN_BULK_AMOUNT = 1000000; //2^20 ~1million

void gemtcAddList(MemoryPointer *bp){
  MemoryPointer *p;
  // printf("start\n");
  //freep start of list of free memory
    for(p = freep; (bp->data < p->data || bp->data > (p->ptr)->data); p = p->ptr){
  //  for(p = freep; !(bp->data > p->data && bp->data < (p->ptr)->data); p = p->ptr){
      //printf("in for p:%p  bp:%p\n", p->data, bp->data);
    if(p->data >= (p->ptr)->data && (bp->data > p->data || bp->data < (p->ptr)->data)){
      // printf("break\n");
      break;
    }
  }
  if(bp->data==p->data)return; //memory is freed twice
                               // This is a bad patch for the problem

    // printf("step 1\n");
  if( (((char *)bp->data) + bp->size) == (char *)p->ptr->data){
    // printf("step 1 else\n");
    bp->size += (p->ptr)->size;
    bp->ptr = (p->ptr)->ptr;
    cudaSafeMemcpy(bp->data, &bp->size, sizeof(unsigned), hipMemcpyHostToDevice,
                   stream_dataIn, "Merging freed memory in old block");
    // printf("step 2\n");
    if(p->ptr != &base)free(p->ptr);
  }else
    bp->ptr = p->ptr;

  if( (((char *)p->data) + p->size) == (char *)bp->data){
    p->size += bp->size;
    p->ptr = bp->ptr;
    cudaSafeMemcpy(p->data, &p->size, sizeof(unsigned), hipMemcpyHostToDevice,
	       stream_dataIn, "Merging old memory into new block");
    // printf("step 3\n");
    free(bp);
  }else
    p->ptr = bp;
  
  freep = p;
}

void gemtcFree(void *loc){
  loc = ((void *)(((char *)loc)-headerSize));
  MemoryPointer *v = (MemoryPointer *) malloc(sizeof(MemoryPointer));
  cudaSafeMemcpy(&v->size, loc, sizeof(unsigned), hipMemcpyDeviceToHost,
                 stream_dataOut, "Reading size of freed memory");
  v->data = (unsigned *) loc;
  // printf("start add list\n");
  gemtcAddList(v);
  // printf("end add list\n");
}


static MemoryPointer *morecore(unsigned nu){
  void *cp;
  MemoryPointer *up = (MemoryPointer *)malloc(sizeof(MemoryPointer));
  if (nu < MIN_BULK_AMOUNT) nu = MIN_BULK_AMOUNT;
  hipMalloc(&cp, nu);

  up->data = (unsigned *)cp;

  up->size = nu;
  cudaSafeMemcpy(cp,&nu,sizeof(unsigned),hipMemcpyHostToDevice,
		 stream_dataIn, "Writing size of new block from hipMalloc");

  gemtcAddList(up);
  return freep;
}


void *gemtcMalloc(unsigned nbytes){
  MemoryPointer *p, *prevp;
  if ((prevp = freep)==NULL){
    base.ptr = freep = prevp = &base;
    base.size = 0;
  }
  nbytes+=headerSize;
  if(nbytes%CHUNK_SIZE!=0)nbytes+=(CHUNK_SIZE-nbytes%CHUNK_SIZE);
  char *loc;
  for(p = prevp->ptr; ;prevp = p, p = p->ptr){
    if(p->size >= nbytes){
      if(p->size == nbytes){
        prevp->ptr = p->ptr;
        loc = (char *) p->data;
        free(p);
      }else{
        p->size -= nbytes;
        loc =((char *) p->data)+p->size;
      }
      freep = prevp;
      cudaSafeMemcpy(loc,&nbytes,sizeof(unsigned),hipMemcpyHostToDevice,
                     stream_dataIn, "Writing size on newly allocated memory");
      return (void *)(loc+headerSize);
    }
    if (p == freep)
      if((p = morecore(nbytes))==NULL)
        return NULL;
  }
}

