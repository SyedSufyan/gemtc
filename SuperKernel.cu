#include "hip/hip_runtime.h"
#include <stdio.h>

#include "Kernels/gemtcKernelLib.cu"
#include "Kernels/AddSleep.cu"
#include "Kernels/VecAdd.cu"
#include "Kernels/VecDot.cu"
#include "Kernels/MatrixSquare.cu"
#include "Kernels/MatrixMultiply.cu"
#include "Kernels/MatrixTranspose.cu"
#include "Kernels/MatrixVector.cu"
#include "Kernels/MatrixInverse.cu"
#include "Kernels/StencilCopy.cu"
#include "Kernels/StencilUpdate.cu"
#include "Kernels/BlackScholes.cu"
#include "Kernels/ArrayMin.cu"
#include "Kernels/ArrayMax.cu"
#include "Kernels/ArrayAvg.cu"
#include "Kernels/LCSubstring.cu"
#include "Kernels/MDProxy.cu"
#include "Kernels/MDFake.cu"
//#include "Kernels/Sort.cu"


__device__ JobPointer executeJob(volatile JobPointer currentJob);

__global__ void superKernel(volatile Queue incoming, 
                            volatile Queue results, volatile int *kill)
{ 
    int warp_size = 32;

    int threadID = threadIdx.x % warp_size;
    int warpID = threadIdx.x / warp_size;   //add depenency on block?

    //Init shared memory to hold Task descriptions
    __shared__ JobPointer currentJobs[32];

    //Init general purpose shared memory
    // TODO: make this work correctly
    __shared__ char shared_mem[8192];  //8kB for the 8 warps, so 1kB each
    gemtcInitSharedMemory(shared_mem, 8192, 8);

    while(!(*kill))
    {
      //dequeue a task
      if(threadID==0)
          FrontAndDequeueJob(incoming, &currentJobs[warpID], kill);
      if(*kill)return;
      //execute the task
      volatile JobPointer retval;
      if(threadID<(currentJobs[warpID]->numThreads)) 
          retval = executeJob(currentJobs[warpID]);
      if(*kill)return;
      //enqueue the result
      if(threadID==0) EnqueueResult(retval, results, kill);
    }
    return;
}

__device__ JobPointer executeJob(JobPointer currentJob){

  int JobType = currentJob->JobType;

  // large switch
  switch(JobType){
    case 0:
      addSleep(currentJob->params);
      break;
    case 1:
      VecAdd(currentJob->params);
      break;
    case 2:
      MatrixSquare(currentJob->params);
      break;
    case 3:
      //      Sort(currentJob->params);
      break;
    case 4:
      VecDot(currentJob->params);
      break;
    case 5:
      MatrixMultiply(currentJob->params);
      break;
    case 6:
      MatrixTranspose(currentJob->params);
      break;
    case 7:
      MatrixVector(currentJob->params);
      break;
    case 8:
      MatrixInverse(currentJob->params);
      break;
    case 9:
      StencilCopy(currentJob->params);
    case 10:
      StencilUpdate(currentJob->params);
      break;
    case 11:
      BlackScholes(currentJob->params);
    case 12:
      ArrayMin(currentJob->params);
      break;
    case 13:
      ArrayMax(currentJob->params);
      break;
    case 14:
      ArrayAvg(currentJob->params);
      break;
    case 15:
      LCSubstring(currentJob->params);
      break;
    case 16:
      ComputeParticles(currentJob->params);
      break;
    case 17:
      InitParticles(currentJob->params);
      break; 
    case 18:
      UpdatePosVelAccel(currentJob->params);
      break;
    case 19:
      UnpackTable(currentJob->params);
      break;
    case 20:
      FakeInit(currentJob->params);
      break;
    case 21:
      FakeUpdate(currentJob->params);
      break; 
  }
  return currentJob;
}

