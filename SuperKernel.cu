#include "hip/hip_runtime.h"
#include <stdio.h>
#include "Kernels/gemtcKernelLib.cu"
#include "Kernels/AddSleep.cu"
#include "Kernels/Square.cu"
#include "Kernels/VecAdd.cu"
#include "Kernels/VecDot.cu"
#include "Kernels/VecSqrt.cu"
#include "Kernels/MatrixSquare.cu"
#include "Kernels/MatrixMultiply.cu"
#include "Kernels/MatrixTranspose.cu"
#include "Kernels/MatrixVector.cu"
#include "Kernels/MatrixInverse.cu"
#include "Kernels/StencilCopy.cu"
#include "Kernels/StencilUpdate.cu"
#include "Kernels/BlackScholes.cu"
#include "Kernels/ArrayMin.cu"
#include "Kernels/ArrayMax.cu"
#include "Kernels/ArrayAvg.cu"
#include "Kernels/LCSubstring.cu"
#include "Kernels/MDProxy.cu"
#include "Kernels/MDFake.cu"
#include "Kernels/ArrayAtomic.cu"
#include "Kernels/ArrayRotate.cu"
#include "Kernels/SciColSim.cu"
#include "Kernels/FluidW.cu"
#include "Kernels/freezeAndPtot.cu"
#include "Kernels/FluidTVD.cu"
#include "Kernels/Pi.cu"
#include "Kernels/MatrixDet.cu"
//#include "Kernels/Sort.cu"

/*
 * Update this value whenever we add new kernel
 */
#define GEMTC_MAX_KERNELS 30

__device__ JobPointer executeJob(volatile JobPointer currentJob);

__global__ void superKernel(volatile Queue incoming, 
                            volatile Queue results, volatile int *kill)
{ 
    int warp_size = 32;

    int threadID = threadIdx.x % warp_size;
    int warpID = threadIdx.x / warp_size;   //add depenency on block?

    //Init shared memory to hold Task descriptions
    __shared__ JobPointer currentJobs[32];

    //Init general purpose shared memory
    // TODO: make this work correctly
    __shared__ char shared_mem[8192];  //8kB for the 8 warps, so 1kB each
    gemtcInitSharedMemory(shared_mem, 8192, 8);

    while(!(*kill))
    {
      //dequeue a task
      if(threadID==0)
          FrontAndDequeueJob(incoming, &currentJobs[warpID], kill);
      if(*kill)return;
      //execute the task
      volatile JobPointer retval;
      if(threadID<(currentJobs[warpID]->numThreads)) 
          retval = executeJob(currentJobs[warpID]);
      if(*kill)return;
      //enqueue the result
      if(threadID==0) EnqueueResult(retval, results, kill);
    }
    return;
}

__device__ JobPointer executeJob(JobPointer currentJob){

  int JobType = currentJob->JobType;

  // large switch
  switch(JobType){
    case 0:
      addSleep(currentJob->params);
      break;
    case 1:
      VecAdd(currentJob->params);
      break;
    case 2:
      MatrixSquare(currentJob->params);
      break;
    case 3:
      //      Sort(currentJob->params);
      break;
    case 4:
      VecDot(currentJob->params);
      break;
    case 5:
      MatrixMultiply(currentJob->params);
      break;
    case 6:
      MatrixTranspose(currentJob->params);
      break;
    case 7:
      MatrixVector(currentJob->params);
      break;
    case 8:
      MatrixInverse(currentJob->params);
      break;
    case 9:
      StencilCopy(currentJob->params);
    case 10:
      StencilUpdate(currentJob->params);
      break;
    case 11:
      BlackScholes(currentJob->params);
    case 12:
      ArrayMin(currentJob->params);
      break;
    case 13:
      ArrayMax(currentJob->params);
      break;
    case 14:
      ArrayAvg(currentJob->params);
      break;
    case 15:
      LCSubstring(currentJob->params);
      break;
    case 16:
      //ComputeParticles(currentJob->params);
      break;
    case 17:
      //InitParticles(currentJob->params);
      break; 
    case 18:
      //UpdatePosVelAccel(currentJob->params);
      break;
    case 19:
      ComputeTest(currentJob->params);
      break; 
    case 20:
      //Only use this kernel if you are doing multi-node.
      MDProxy(currentJob->params);
      break;
    case 21:
      VecSqrt(currentJob->params);
      break;
    case 22:
      VecSqrt(currentJob->params);
      break;
    case 23:
      Square(currentJob->params);
      break;
    case 24:
      /*
       * Imogen's ported kernel from "gpuImogen/gpuclass/cudaArrayAtomic.cu"
       */
      ArrayAtomic(currentJob->params);
      break;
    case 25:
      /*
       * Imogen's ported kernel from "gpuImogen/gpuclass/cudaArrayRotate.cu"
       */
      ArrayRotate(currentJob->params);
      break;
    case 26:
      /*
       * SciColSim's expensive function update_probabilities_all_visible() from 
       * "scicolsim-2013-03-07/src/optimizer.cpp"
       */
      gemtc_scicolsim(currentJob->params);
      break;
    case 27:
      /*
       * Imogen's ported cukern_Wstep_hydro_uniform() kernel from "gpuImogen/gpuclass/cudaFluidW.cu"
       */
      cukern_Wstep_hydro_uniform(currentJob->params);
      break;
    case 28:
      /*
       * Imogen's ported cukern_FreezeSpeed_hydro() kernel from "gpuImogen/gpuclass/freezeAndPtot.cu"
       */
      cukern_FreezeSpeed_hydro(currentJob->params);
      break;
    case 29:
      /*
       * Imogen's ported cukern_TVDStep_hydro_uniform() kernel from "gpuImogen/gpuclass/cudaFluidTVD.cu"
       */
      cukern_TVDStep_hydro_uniform(currentJob->params);
      break;
    case 30:
      /*
       * Calculates PI using Monte Carlo simulation 
       */
      gemtc_pi(currentJob->params);
      break;
    case 31:
      //Matrix Determinant ... Newly added by Karthik
      MatrixDeterminant(currentJob->params);
      break;	
  }
  return currentJob;
}

