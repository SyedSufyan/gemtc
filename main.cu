#include "gemtc.cu"
#include "Tests/test-mc/tests.cu"
#include <hip/hip_runtime.h>

int main(int argc, char **argv){

  // creates two queues each size of half this param
  setupGemtc(2560);

  // doing work on gpu
  int i;
  for(i=0; i<1; i++){
    int sleepTime = 6;
    // runs a task on the gpu
    //testArrayAvg();
    //testArrayMax();
    //testArrayMin();
    //testAdd();
    //testVectorProduct();
    //testMatrixMultiply();
    //testMatrixTranspose();
    //testMatrixSquare();
    //testMatrixVector();
    //testMatrixInverse();
    //testStencil();
    testBlackScholes();
    //void *ret = run(0, 32, &sleepTime, sizeof(int));
    //printf("%d : Finished job with parameter: %d\n",i, *(int *)ret);
  }

  // stops the superkernel and cleans up some memory
  cleanupGemtc();

  return 0;
}
