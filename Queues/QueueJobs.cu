#include <stdlib.h>
#include "QueueHelpers.cu"

////////////////////////////////////////////////////////////
// Constructor and Deconsturctor
////////////////////////////////////////////////////////////

Queue CreateQueue(int MaxElements) {
  Queue Q = (Queue) malloc (sizeof(struct QueueRecord));

  Q->Array = (JobPointer *) gemtcMalloc(sizeof(JobPointer)*MaxElements);

  Q->Capacity = MaxElements;
  Q->Front = 1;
  Q->Rear = 0;
  Q->ReadLock = 0;

  Queue d_Q = (Queue) gemtcMalloc(sizeof(struct QueueRecord));
  cudaSafeMemcpy(d_Q, Q, sizeof(struct QueueRecord), 
                 hipMemcpyHostToDevice, stream_dataIn, 
                 "Copying initial queue to device");
  free(Q);
  return d_Q;
}

void DisposeQueue(Queue d_Q) {
  Queue h_Q = (Queue) malloc(sizeof(struct QueueRecord));
  cudaSafeMemcpy(h_Q, d_Q, sizeof(struct QueueRecord), 
                 hipMemcpyDeviceToHost, stream_dataIn,
                 "DisposeQueue, Copying Queue to get array pointer");
  gemtcFree((void *)h_Q->Array);
  free(h_Q);
  gemtcFree(d_Q);
}

////////////////////////////////////////////////////////////
// Host Functions to Change Queues
////////////////////////////////////////////////////////////

void EnqueueJob(JobPointer h_JobDescription, Queue Q) {
//called by CPU

  int copySize= sizeof(struct QueueRecord);

  Queue h_Q = (Queue) malloc(sizeof(struct QueueRecord));
  cudaSafeMemcpy(h_Q, Q, copySize, hipMemcpyDeviceToHost, stream_dataIn,
                 "EnqueueJob, Getting Queue");
  int c=0;
  while(h_IsFull(h_Q)){
    if(c%500==0)printf("Looping alot\n");
    pthread_yield();
    cudaSafeMemcpy(h_Q, Q, copySize, hipMemcpyDeviceToHost, stream_dataIn,
                    "EnqueueJob, Getting Queue again...");
    c++;
  }

  // floating point exception from mod capacity if 0 or -n
  h_Q->Rear = (h_Q->Rear+1)%(h_Q->Capacity);

  JobPointer d_JobDescription = 
      (JobPointer) gemtcMalloc(sizeof(struct JobDescription));

  cudaSafeMemcpy( d_JobDescription,
                  h_JobDescription, 
                  sizeof(struct JobDescription),
                  hipMemcpyHostToDevice, 
                  stream_dataIn,
                  "EnqueueJob, Writing JobDescription");

  // set job description
  cudaSafeMemcpy( (void *)&h_Q->Array[h_Q->Rear],
                  &d_JobDescription, 
                  sizeof(JobPointer),
                  hipMemcpyHostToDevice, 
                  stream_dataIn,
                  "EnqueueJob, Writing JobPointer");

  cudaSafeMemcpy(movePointer(Q, 12), movePointer(h_Q, 12), 
		 sizeof(int), hipMemcpyHostToDevice, stream_dataIn,
                 "EnqueueJob, Updating Queue");
  free(h_Q);
}

JobPointer MaybeFandD(Queue Q){
  int copySize= sizeof(struct QueueRecord);

  Queue h_Q = (Queue) malloc(sizeof(struct QueueRecord));
  
  cudaSafeMemcpy(h_Q, Q, copySize, hipMemcpyDeviceToHost, stream_dataOut,
                 "MaybeFandDJob, Getting Queue");
  //printf("Reading queue\n");
  //printf("Front %d\n", h_Q->Front);
  //printf("Rear %d\n", h_Q->Rear);

  if(h_IsEmpty(h_Q)){
    free(h_Q);
    return NULL;
  }else{
    JobPointer *resultP = (JobPointer *) malloc(sizeof(JobPointer));
    JobPointer result = (JobPointer) malloc(sizeof(struct JobDescription));

    //Read task Description Location
    cudaSafeMemcpy(resultP, (void *)&h_Q->Array[h_Q->Front], sizeof(JobPointer), 
                   hipMemcpyDeviceToHost, stream_dataOut,
                   "FandDJob, Getting JobPointer");
    //Read task description
    cudaSafeMemcpy(result, (void *)*resultP, sizeof(struct JobDescription), 
                   hipMemcpyDeviceToHost, stream_dataOut,
                   "FandDJob, Getting JobDescription");
    //Free task description
    gemtcFree(*resultP);


    //Update Queue metadata
    h_Q->Front = (h_Q->Front+1)%(h_Q->Capacity);
    cudaSafeMemcpy( movePointer(Q, 16), movePointer(h_Q, 16), 
                    sizeof(int), hipMemcpyHostToDevice, stream_dataOut,
                    "FandDJob, Updating Queue");

    free(h_Q);
    free(resultP);
    return result;
  }
}

JobPointer FrontResult(Queue Q) {
//called by CPU
  int copySize= sizeof(struct QueueRecord);

  Queue h_Q = (Queue) malloc(sizeof(struct QueueRecord));
  
  cudaSafeMemcpy(h_Q, Q, copySize, hipMemcpyDeviceToHost, stream_dataOut,
                 "FandDJob, Getting Queue");
  while(h_IsEmpty(h_Q)){
    pthread_yield();
    cudaSafeMemcpy(h_Q, Q, copySize, hipMemcpyDeviceToHost, stream_dataOut,
                   "FandDJob, Getting Queue again...");
  }
  //  printf("job pointer start\n");
  JobPointer *resultP = (JobPointer *) malloc(sizeof(JobPointer));
  JobPointer result = (JobPointer) malloc(sizeof(struct JobDescription));
  // printf("job pointer end\n");

  cudaSafeMemcpy(resultP, (void *)&h_Q->Array[h_Q->Front], sizeof(JobPointer), 
                 hipMemcpyDeviceToHost, stream_dataOut,
                 "FandDJob, Getting JobPointer");

  cudaSafeMemcpy(result, (void *)*resultP, sizeof(struct JobDescription), 
                 hipMemcpyDeviceToHost, stream_dataOut,
                 "FandDJob, Getting JobDescription");
  // printf("free begin\n");
  free(h_Q);

  free(resultP);

  return result;
}
void DequeueResult(Queue Q) {
//called by CPU
  int copySize= sizeof(struct QueueRecord);

  Queue h_Q = (Queue) malloc(sizeof(struct QueueRecord));

  cudaSafeMemcpy(h_Q, Q, copySize, hipMemcpyDeviceToHost, stream_dataOut,
                 "FandDJob, Getting Queue");

  while(h_IsEmpty(h_Q)){
    pthread_yield();
    cudaSafeMemcpy(h_Q, Q, copySize, hipMemcpyDeviceToHost, stream_dataOut,
                   "FandDJob, Getting Queue again...");
  }

  JobPointer *resultP = (JobPointer *) malloc(sizeof(JobPointer));
  //Pointer to Front task description

  cudaSafeMemcpy(resultP, (void *)&h_Q->Array[h_Q->Front], sizeof(JobPointer), 
                 hipMemcpyDeviceToHost, stream_dataOut,
                 "FandDJob, Getting JobPointer");
  
  gemtcFree(*resultP);


  h_Q->Front = (h_Q->Front+1)%(h_Q->Capacity);

  cudaSafeMemcpy( movePointer(Q, 16), movePointer(h_Q, 16), 
		  sizeof(int), hipMemcpyHostToDevice, stream_dataOut,
                  "FandDJob, Updating Queue");

  free(h_Q);
}


////////////////////////////////////////////////////////////
// Device Functions to Change Queues
////////////////////////////////////////////////////////////
__device__ void FrontAndDequeueJob(volatile Queue Q, volatile JobPointer *pResult, 
                                   volatile int *kill) {
//called by GPU
  getLock(Q);

  int count = 0;
  while(d_IsEmpty(Q)){
    if(*kill)return;
    count++;
  }
  volatile int *front = &Q->Front;

  volatile JobPointer *ppResult = Q->Array + *front;

  *pResult = *ppResult;

  *front = (*front+1)%(Q->Capacity);

  releaseLock(Q);
}

__device__ void EnqueueResult(volatile JobPointer X, volatile Queue Q, volatile int *kill) {
//called by GPU
  getLock(Q);

  int count =0;
  while(d_IsFull(Q)){
    count++;
    if(*kill)return;
  }
  volatile int *rear = &Q->Rear;
  int temp = (*rear + 1)%(Q->Capacity);

  volatile JobPointer *pLoc = Q->Array + temp;
  *pLoc = X;

  *rear = temp;

  releaseLock(Q);
}




