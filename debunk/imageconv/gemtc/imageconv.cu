#include "../../../gemtc.cu"
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>

//#define DEBUG 1
// flag == 0 Input, flag ==1 Output
void printStart(void *param,int flag){
  float *input = (float *) param;
  int IW = (int)input[0];   //Image Width
  int MW = (int)input[1]; //MASK_WIDTH;
  float* image = input+2;
  float* mask = image + IW;
  float* imageout = image + MW + IW;

  int i;
  if(flag ==0){
    printf("Printing Image: \n");
   for(i=0; i<IW; i++){
      if (i%IW == 0 && i!=0)
        printf("\n");
    printf("%f ", image[i]);
  }
        printf("\nPrinting Mask: \n");

  for(i=0; i<MW; i++){
      if (i%MW == 0 && i!=0)
        printf("\n");
   printf("%f", mask[i]);
  }
 }
 if(flag ==1){
 printf("\nPrinting Output\n");
  for(i=0; i<IW; i++){
    if (i%IW == 0 && i!=0)
           printf("\n");

  printf("%f ", imageout[i]);
   }
  }
}

int main(int argc, char **argv){
  int NUM_TASKS, IMAGE_WIDTH, MASK_WIDTH;
  int warps;
  int Overfill = 0;
  hipDeviceProp_t devProp;
  hipGetDeviceProperties(&devProp, 0);
  int blocks = devProp.multiProcessorCount;
  if(argc>2){

    IMAGE_WIDTH = atoi(argv[1]);
    MASK_WIDTH = atoi(argv[2]);

  }else{
    printf("This test requires four parameters:\n");
    printf("   int IMAGE_WIDTH, int MSAK_WIDTH\n");
    printf("       IMAGE_WIDTH is the number of pixels in an image in one dimensional\n");
    printf("       MASK_WIDTH is the width of the mask to be applied on the image\n");


    exit(1);
  }
  
  if(Overfill==1){
    warps = devProp.maxThreadsPerBlock/32;
  }
	if(Overfill==0){
		int coresPerSM = _ConvertSMVer2Cores(devProp.major, devProp.minor);
		warps = coresPerSM/16;  //A warp runs on 16 cores
	}
	if(Overfill==2){
		warps =1;
		blocks = 1;
	}
	NUM_TASKS = warps * blocks;
	IMAGE_WIDTH = IMAGE_WIDTH/NUM_TASKS;

  gemtcSetup(25600, 1);
  int size = sizeof(float)*(2+2 * IMAGE_WIDTH + MASK_WIDTH);

  int j;
  int temp_size = IMAGE_WIDTH + MASK_WIDTH;

  float *h_params = (float *) malloc(size);

  h_params[0] = IMAGE_WIDTH;
  h_params[1] = MASK_WIDTH;

  for(j=2; j<temp_size+2; j++){
    float r = (float)(rand() % 100);
    h_params[j] = r;
  }
  //0 for printing inputs
  #ifdef DEBUG
  printStart(h_params,0);
  #endif
  //Starting timing
  struct timeval tim;
  double t1,t2;
  gettimeofday(&tim, NULL);
  t1=tim.tv_sec+(tim.tv_usec/1000000.0);
  for(j=0; j<NUM_TASKS; j++){
	  float *d_params = (float *) gemtcGPUMalloc(size);

	  gemtcMemcpyHostToDevice(d_params, h_params, size);
	  gemtcPush(32, 32, j, d_params);
   
    }
	
	  void *ret=NULL;
	  int id;
	  while(ret==NULL){
		gemtcPoll(&id, &ret);
	  }

     gemtcMemcpyDeviceToHost(h_params, ret, size);
     gettimeofday(&tim, NULL);
     t2=tim.tv_sec+(tim.tv_usec/1000000.0);
      // Free the device pointer
     gemtcGPUFree(ret);
    

      // Do we need to do this?
      ret = NULL;

  // 1 for printing output
  #ifdef DEBUG
  printStart(h_params,1);
  #endif
  printf("%.4lf\t", (t2-t1));
  gemtcCleanup();
  free(h_params);
  return 0;
}
