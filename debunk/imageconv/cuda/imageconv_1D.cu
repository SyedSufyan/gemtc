#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "../../saxpy/saxpy.c"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
//#define DEBUG 0

#define CHECK_ERR(x)                                    \
  if (x != hipSuccess) {                               \
    fprintf(stderr,"%s in %s at line %d\n",             \
            hipGetErrorString(err),__FILE__,__LINE__); \
    exit(-1);                                           \
  }   

__global__ void image_1D_convolution(float *M, float *N, float *C, int mask_width, int width,int num_threads)
{
int threadId = blockIdx.x * blockDim.x + threadIdx.x;
float value =0;
int start;
int index;
//this function includes 2 floating point operations
while(threadId < width)
{
start = threadId - (mask_width/2);
for(int i=0; i<mask_width;i++){
        index= start + i;
        if(index >=0 && index <width)
                value = value + N[index] * M[i];
}
threadId = threadId + num_threads;
C[threadId] = value;
}
}

void print(float* result,int size){
	printf("Printing array....\n");
	for(int i=0;i<size;i++){
		printf(" %f ", result[i]);
	}
printf("\n");
}

int main(int argc, char *argv[]){
  //mask_width, filter width
  int IMAGE_WIDTH, MASK_WIDTH,NUM_THREADS,FLAG;
  float *h_M, *h_N, *h_C;
  float *d_M, *d_N, *d_C;
  size_t size_M,size_N;
  hipError_t err;
  if(argc!=5)
    {
      printf("This test requires two parameters:\n");
      printf("   int IMAGE_WIDTH, int MASK_WIDTH, int NUM_THREADS \n");
      printf("where  IMAGE_WIDTH is the number of pixels in an image in one dimensional\n");
      printf("       MASK_WIDTH is the width of the mask to be applied on the image\n");
      printf("       NUM_THREADS is the number of threads to be executed in parallel\n");
      printf("       FLAG to decide flops including data copy or not. 1 for flops with data copy and 0 for only execution of gpu function.\n");
      
      exit(1);
    }
  srand (time(NULL));
  IMAGE_WIDTH = atoi(argv[1]);
  MASK_WIDTH  = atoi(argv[2]);
  NUM_THREADS = atoi(argv[3]);
  FLAG = atoi(argv[4]);
  
  // allocate host
  size_M = sizeof(float) * MASK_WIDTH;
  size_N = sizeof(float) * IMAGE_WIDTH;
  h_N = (float *) malloc(size_N);
  h_M = (float *) malloc(size_M);
  h_C = (float *) malloc(size_N);
  
  // allocate device
  err=hipMalloc((void **) &d_M, size_M);
  CHECK_ERR(err);
  err=hipMalloc((void **) &d_N, size_N);
  CHECK_ERR(err);
  err=hipMalloc((void **) &d_C, size_N);
  CHECK_ERR(err);
  
  // pop arrays
  populateRandomFloatArray(IMAGE_WIDTH,h_N);
  populateRandomFloatArray(MASK_WIDTH,h_M);
  
#ifdef DEBUG
  print(h_N,IMAGE_WIDTH);
  print(h_M, MASK_WIDTH);
#endif
  
  // Start the timer
  struct timeval tim;
  double t1,t2;
  
  if(FLAG){
    gettimeofday(&tim, NULL);
    t1=tim.tv_sec+(tim.tv_usec/1000000.0);
  }

  err = hipMemcpy(d_M,h_M,size_M,hipMemcpyHostToDevice);
  CHECK_ERR(err);
  err = hipMemcpy(d_N,h_N,size_N, hipMemcpyHostToDevice);
  CHECK_ERR(err);
  
  if(!FLAG){
    gettimeofday(&tim, NULL);
    t1=tim.tv_sec+(tim.tv_usec/1000000.0);
  }

  image_1D_convolution<<<1,NUM_THREADS>>>(d_M,d_N,d_C,MASK_WIDTH,IMAGE_WIDTH,NUM_THREADS);
  hipDeviceSynchronize();
  if(!FLAG){
    gettimeofday(&tim, NULL);
    t2=tim.tv_sec+(tim.tv_usec/1000000.0);
  }

  //Copy back the results from the device
  //printf("%x %x %d\n", h_C, d_C, size_N);
  
  float * temp = (float *)malloc(size_N);
  //  err = hipMemcpy((void *)h_C, (void *)d_C, size_N, hipMemcpyDeviceToHost);
  err = hipMemcpy((void *)temp, (void *)d_C, size_N, hipMemcpyDeviceToHost);
  CHECK_ERR(err);
  //printf("AFTER COPY BACK!\n");

#ifdef DEBUG
  print(h_C,IMAGE_WIDTH);
#endif
  
  // free device
  hipFree(d_C);
  hipFree(d_M);
  hipFree(d_N);
  
  if(FLAG){
    gettimeofday(&tim, NULL);
    t2=tim.tv_sec+(tim.tv_usec/1000000.0);
  }
  
  // Print timing information
  printf("%.4lf\t",(t2-t1));

  // free cpu
  free(h_M);
  free(h_N);
  free(h_C);
}
