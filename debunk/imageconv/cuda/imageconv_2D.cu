#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "../saxpy/saxpy.c"
#include "../../utils/logger.h"
#include "../../utils/logger.c"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#define DEBUG 1 

#define CHECK_ERR(x)                                    \
  if (x != hipSuccess) {                               \
    fprintf(stderr,"%s in %s at line %d\n",             \
            hipGetErrorString(err),__FILE__,__LINE__); \
    exit(-1);                                           \
  }   

__global__ void image_1D_convolution(float *M, float *N, float *C, int mask_width, int width,int num_threads)
{
int threadIdX = blockIdx.x * blockDim.x + threadIdx.x;
int threadIdY = blockIdx.y * blockDim.y + threadIdx.y;
float value =0;
int startx, starty;
int indexx,indexy;
printf("%d - %d", threadIdX,threadIdY);
//this function includes 2 floating point operations
while(threadIdX < width)
{
startx = threadIdX - (mask_width/2);

while(threadIdY < width)
{
starty = threadIdY - (mask_width/2);

for(int i=0; i<mask_width;i++){
        indexx  = startx + i;
        for(int j=0; j<mask_width;j++){
                indexy  = starty + j;
                if(indexx >=0 && indexx <width && indexy >=0 && indexy <width)
                        value = value + N[indexx * width + indexy] * M[i * mask_width + j];
        }
}


threadIdY = threadIdY + num_threads;
C[threadIdX * width + threadIdY] = value;
printf("%d - %d\n", threadIdX,threadIdY);
}
threadIdX = threadIdX + num_threads;
printf("X : %d\n", threadIdX);
}
}

void print(float* result,int size){
	printf("Printing array....\n");
	for(int i=0;i<size;i++){
		printf(" %f ", result[i]);
	}
printf("\n");
}

int main(int argc, char *argv[]){
//mask_width, filter width
int IMAGE_WIDTH, MASK_WIDTH,NUM_THREADS,FLAG;
float *h_M, *h_N, *h_C;
float *d_M, *d_N, *d_C;
size_t size_M,size_N;
hipError_t err;
if(argc!=5)
{
	printf("This test requires two parameters:\n");
    printf("   int IMAGE_WIDTH, int MASK_WIDTH, int NUM_THREADS \n");
    printf("where  IMAGE_WIDTH is the number of pixels in an image in one dimensional\n");
    printf("       MASK_WIDTH is the width of the mask to be applied on the image\n");
    printf("       NUM_THREADS is the number of threads to be executed in parallel\n");
    printf("       FLAG to decide flops including data copy or not. 1 for flops with data copy and 0 for only execution of gpu function.\n");

	exit(1);
}
srand (time(NULL));
IMAGE_WIDTH = atoi(argv[1]);
MASK_WIDTH  = atoi(argv[2]);
NUM_THREADS = atoi(argv[3]);
FLAG = atoi(argv[4]);

size_M = sizeof(float) * MASK_WIDTH * MASK_WIDTH;
size_N = sizeof(float) * IMAGE_WIDTH * IMAGE_WIDTH;
h_N = (float *) malloc(size_N);
h_M = (float *) malloc(size_M);
h_C = (float *) malloc(size_N);

err=hipMalloc((void **) &d_M, size_M);
CHECK_ERR(err);
err=hipMalloc((void **) &d_N, size_N);
CHECK_ERR(err);
err=hipMalloc((void **) &d_C, size_N);
CHECK_ERR(err);

populateRandomFloatArray(IMAGE_WIDTH * IMAGE_WIDTH,h_N);
populateRandomFloatArray(MASK_WIDTH * MASK_WIDTH,h_M);
#ifdef DEBUG
print(h_N,IMAGE_WIDTH * IMAGE_WIDTH);
print(h_M, MASK_WIDTH * MASK_WIDTH);
#endif
// Start the timer
struct timeval tim;
double t1,t2;
if(FLAG){
gettimeofday(&tim, NULL);
t1=tim.tv_sec+(tim.tv_usec/1000000.0);
}

err = hipMemcpy(d_M,h_M,size_M,hipMemcpyHostToDevice);
CHECK_ERR(err);
err = hipMemcpy(d_N,h_N,size_N, hipMemcpyHostToDevice);
CHECK_ERR(err);

if(!FLAG){
gettimeofday(&tim, NULL);
t1=tim.tv_sec+(tim.tv_usec/1000000.0);
}
image_1D_convolution<<<1,NUM_THREADS>>>(d_M,d_N,d_C,MASK_WIDTH,IMAGE_WIDTH,NUM_THREADS);
hipDeviceSynchronize();
if(!FLAG){
gettimeofday(&tim, NULL);
t2=tim.tv_sec+(tim.tv_usec/1000000.0);
}

//Copy back the results from the device
err = hipMemcpy(h_C,d_C,size_N,hipMemcpyDeviceToHost);
CHECK_ERR(err);
#ifdef DEBUG
print(h_C,IMAGE_WIDTH * IMAGE_WIDTH);
#endif
hipFree(d_C);
hipFree(d_M);
hipFree(d_N);

if(FLAG){
gettimeofday(&tim, NULL);
t2=tim.tv_sec+(tim.tv_usec/1000000.0);
}

// Print timing information
printf("%.6lf\t", (((2*MASK_WIDTH*IMAGE_WIDTH)/(t2-t1))/1000000000));

free(h_M);
free(h_N);
free(h_C);
}
