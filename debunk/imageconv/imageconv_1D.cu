#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "../saxpy/saxpy.c"
#include <stdio.h>
#include <stdlib.h>
#define CHECK_ERR(x)                                    \
  if (x != hipSuccess) {                               \
    fprintf(stderr,"%s in %s at line %d\n",             \
            hipGetErrorString(err),__FILE__,__LINE__); \
    exit(-1);                                           \
  }   

__global__ void image_1D_convolution(float *M, float *N, float *C, int mask_width, int width)
{
int threadId = blockIdx.x * blockDim.x + threadIdx.x;
float value =0;
int start = threadId - (mask_width/2);
int index;
//this function includes 2 floating point operations
for(int i=0; i<width;i++){
	index= start + i;
	if(index >=0 && index <width)
		value = value + N[index] * M[i];
}
C[threadId] = value;	
}

void print(float* result,int size){
	printf("Printing array....\n");
	for(int i=0;i<size;i++){
		printf(" %f ", result[i]);
	}
printf("\n");
}

int main(int argc, char *argv[]){
//mask_width, filter width
int IMAGE_WIDTH, MASK_WIDTH;
float *h_M, *h_N, *h_C;
float *d_M, *d_N, *d_C;
size_t size_M,size_N;
hipError_t err;
if(argc!=3)
{
	printf("This test requires two parameters:\n");
    printf("   int IMAGE_WIDTH, int MASK_WIDTH \n");
    printf("where  IMAGE_WIDTH is the number of pixels in an image in one dimensional\n");
    printf("       MASK_WIDTH is the width of the mask to be applied on the image\n");
	exit(1);
}
IMAGE_WIDTH = atoi(argv[1]);
MASK_WIDTH  = atoi(argv[2]);
size_M = sizeof(float) * MASK_WIDTH;
size_N = sizeof(float) * IMAGE_WIDTH;
h_N = (float *) malloc(size_N);
h_M = (float *) malloc(size_M);
h_C = (float *) malloc(size_N);

err=hipMalloc((void **) &d_M, size_M);
CHECK_ERR(err);
err=hipMalloc((void **) &d_N, size_N);
CHECK_ERR(err);
err=hipMalloc((void **) &d_C, size_N);
CHECK_ERR(err);

populateRandomFloatArray(IMAGE_WIDTH,h_N);
populateRandomFloatArray(MASK_WIDTH,h_M);

print(h_N,IMAGE_WIDTH);
print(h_M, MASK_WIDTH);

err = hipMemcpy(d_M,h_M,size_M,hipMemcpyHostToDevice);
CHECK_ERR(err);
err = hipMemcpy(d_N,h_N,size_N, hipMemcpyHostToDevice);
CHECK_ERR(err);

image_1D_convolution<<<1,256>>>(d_M,d_N,d_C,MASK_WIDTH,IMAGE_WIDTH);
hipDeviceSynchronize();

//Copy back the results from the device
err = hipMemcpy(h_C,d_C,size_N,hipMemcpyDeviceToHost);
CHECK_ERR(err);
print(h_C,IMAGE_WIDTH);
hipFree(d_C);
hipFree(d_M);
hipFree(d_N);


free(h_M);
free(h_N);
free(h_C);
printf("Number of floating point operations: %d\n", IMAGE_WIDTH*2);
}
