#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#define BIN_COUNT 256
#define NUM_RUNS 8 
#define CHECK_ERR(x)                                    \
  if (x != hipSuccess) {                               \
    fprintf(stderr,"%s in %s at line %d\n",             \
            hipGetErrorString(err),__FILE__,__LINE__); \
    exit(-1);                                           \
  }

__global__ void
histogram( unsigned char *buffer,long size,unsigned int *histo ) {
  __shared__ unsigned int temp[256];
  temp[threadIdx.x] = 0;
   __syncthreads();
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;
  while (i < size)
  {
          atomicAdd( &(temp[buffer[i]]), 1 );
              i += stride;
  }
  __syncthreads();
  atomicAdd( &(histo[threadIdx.x]), temp[threadIdx.x] );
}

void print(unsigned int *histo){
int i;
for(i=0;i<BIN_COUNT;i++){
printf("%d\t",histo[i]);
}
}

int main(int argc, char *argv[])
{
 unsigned char * h_data;
 unsigned int h_histogram[BIN_COUNT];
 unsigned char * d_data;
 unsigned int * d_histogram;
 unsigned int byteCount = 1024;
 size_t size; 
 hipError_t err;
 StopWatchInterface *hTimer = NULL;
 int iter; 
 sdkCreateTimer(&hTimer);
 for(iter =0 ; iter< NUM_RUNS;iter++){
  
  sdkResetTimer(&hTimer);
  sdkStartTimer(&hTimer);
  srand (time(NULL));
 size = sizeof(unsigned char) * byteCount;
 h_data = (unsigned char *) malloc(sizeof(unsigned char) * byteCount);

 err=hipMalloc((void **) &d_data, size);
 CHECK_ERR(err);
 err=hipMalloc((void **) &d_histogram, sizeof(unsigned int) * BIN_COUNT);
 CHECK_ERR(err);

 for (unsigned int i = 0; i < byteCount; i++)
 {
	h_data[i] = rand() % 256;
 } 
 err = hipMemcpy(d_data,h_data,size,hipMemcpyHostToDevice);
 CHECK_ERR(err);
 err = hipMemcpy(d_histogram,h_histogram,sizeof(unsigned int) * BIN_COUNT, hipMemcpyHostToDevice);
 CHECK_ERR(err);

 hipDeviceProp_t prop;
 checkCudaErrors( hipGetDeviceProperties( &prop, 0 ) );
 int blocks = prop.multiProcessorCount;

 histogram<<<blocks*2,BIN_COUNT>>>(d_data,byteCount,d_histogram);
 hipDeviceSynchronize();

//Copy back the results from the device
 err = hipMemcpy(h_histogram,d_histogram,sizeof(unsigned int) * BIN_COUNT,hipMemcpyDeviceToHost);
 CHECK_ERR(err);

//print(h_histogram);
 hipFree(d_data);
 hipFree(d_histogram);
 free(h_data);
 sdkStopTimer(&hTimer);
 
 double dAvgSecs = 1.0e-3 * (double)sdkGetTimerValue(&hTimer) / 1.0;
 printf("%u\t%.4f\t%.5f\n",
                        byteCount,(1.0e-6 * (double)byteCount / dAvgSecs), dAvgSecs);
 byteCount = byteCount * 10;
 
 }
// Print timing information

  sdkDeleteTimer(&hTimer);
}

