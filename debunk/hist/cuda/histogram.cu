#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#define BIN_COUNT 256 
//#define NUM_RUNS 5 
//#define NUM_TEST 10.0
#define BYTE_COUNT 25600
#define CHECK_ERR(x)                                    \
  if (x != hipSuccess) {                               \
    fprintf(stderr,"%s in %s at line %d\n",             \
            hipGetErrorString(err),__FILE__,__LINE__); \
    exit(-1);                                           \
  }

__global__ void
histogram( unsigned char *buffer,long size,unsigned int *histo ) {
  __shared__ unsigned int temp[BIN_COUNT];
  temp[threadIdx.x] = 0;
   __syncthreads();
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;
  while (i < size)
  {
          atomicAdd( &(temp[buffer[i]]), 1 );
              i += stride;
  }
  __syncthreads();
  atomicAdd( &(histo[threadIdx.x]), temp[threadIdx.x] );
  
}

void print(unsigned int *histo){
    int i;
    for(i=0;i<BIN_COUNT;i++){
        printf("%d\t",histo[i]);
    }
}
int main(int argc, char *argv[])
{
    if (argc != 5){
        printf("invalid parameters, use: <NUM_ELEMENTS> <NUM_THREADS> <NUM_TASKS> <NUM_TEST> \n");
    return -1;
    }
    unsigned char * h_data;
    unsigned int h_histogram[BIN_COUNT];
    unsigned char * d_data;
    unsigned int * d_histogram;
    unsigned int byteCount = atoi(argv[1])/atoi(argv[3]);//BYTE_COUNT;
    size_t size;
    hipError_t err;

    //int NUM_RUNS = atoi(argv[1]);
    int NUM_THREADS = atoi(argv[2]);
    int NUM_TEST = atoi(argv[4]);
    
    StopWatchInterface *hTimer = NULL;
    //int iter;
    sdkCreateTimer(&hTimer);
    hipDeviceProp_t prop;
    checkCudaErrors( hipGetDeviceProperties( &prop, 0 ) );
    int blocks = prop.multiProcessorCount;
    //for(iter =0 ; iter < NUM_RUNS;iter++){
        srand (time(NULL));
        size = sizeof(unsigned char) * byteCount;
        h_data = (unsigned char *) malloc(sizeof(unsigned char) * byteCount);
        for (unsigned int i = 0; i < byteCount; i++)
        {
            h_data[i] = rand() % 256;
        }
        sdkResetTimer(&hTimer);
        sdkStartTimer(&hTimer);
        int j;
        for(j=0; j <  NUM_TEST; j++) {
            err=hipMalloc((void **) &d_data, size);
            CHECK_ERR(err);
            err=hipMalloc((void **) &d_histogram, sizeof(unsigned int) * BIN_COUNT);
            CHECK_ERR(err);
            err = hipMemcpy(d_data,h_data,size,hipMemcpyHostToDevice);
            CHECK_ERR(err);
            err = hipMemcpy(d_histogram,h_histogram,sizeof(unsigned int) * BIN_COUNT, hipMemcpyHostToDevice);
            CHECK_ERR(err);
            histogram<<<blocks,NUM_THREADS>>>(d_data,byteCount,d_histogram);
            hipDeviceSynchronize();
            //Copy back the results from the device
            err = hipMemcpy(h_histogram,d_histogram,sizeof(unsigned int) * BIN_COUNT,hipMemcpyDeviceToHost);
            CHECK_ERR(err);
            //print(h_histogram);
            hipFree(d_data);
            hipFree(d_histogram);
        }
        sdkStopTimer(&hTimer);
        free(h_data);
        //unsigned int problem_size = byteCount * 4;
        double dAvgSecs = 1.0e-3 * (double)sdkGetTimerValue(&hTimer) / (double) NUM_TEST;
        //printf("%.4f\t%.4f\t%.5f",
        //(1.0e-6 * (double)problem_size / dAvgSecs),(3.0 * (double) byteCount/dAvgSecs), dAvgSecs);
        printf("%.5f\t",dAvgSecs);
        //byteCount = byteCount * 10;
    //}
    // Print timing information
    sdkDeleteTimer(&hTimer);
}
