#include "../../../gemtc.cu"
#include<stdio.h>
#include<stdlib.h>
#define BIN_COUNT 256
#define NUM_RUNS 6 
#define AVG_RUNS 5 
#include <helper_functions.h>
#include <hip/hip_runtime.h>

int main(int argc, char **argv){
  int NUM_TASKS, LOOP_SIZE;
  uint byteCount = 1024;
  int Overfill =1;
  if(argc>2){
    NUM_TASKS = atoi(argv[1]);
    LOOP_SIZE = atoi(argv[2]);

  }else{
    printf("This test requires four parameters:\n");
    printf("   int NUM_TASKS, int LOOP_SIZE, int MATRIX_SIZE, int STATIC_VALUE\n");
    printf("where  NUM_TASKS is the total numer of vector add tasks to be sent to gemtc\n");
    printf("       LOOP_SIZE is the number of tasks should be sent to gemtc before waiting for results\n");
    exit(1);
  }
  hipDeviceProp_t devProp;
  hipGetDeviceProperties(&devProp, 0);
  StopWatchInterface *hTimer = NULL; 
  int iter,warps;
  int blocks = devProp.multiProcessorCount;
   sdkCreateTimer(&hTimer);   
  for(iter=0; iter < NUM_RUNS; iter++) {

  gemtcSetup(25600, Overfill);
  int d_size = sizeof(unsigned int) * byteCount;
  int h_size = sizeof(int) * BIN_COUNT;
  int size = 1 + d_size + h_size;
  int j;
  int k;
  uint *h_params = (uint *) malloc(size);
  double dAvgSecs;
  if(Overfill==1){
    warps = devProp.maxThreadsPerBlock/32;
  }
  if(Overfill==0){
    int coresPerSM = _ConvertSMVer2Cores(devProp.major, devProp.minor);
    warps = coresPerSM/16;  //A warp runs on 16 cores
  }
  if(Overfill==2){
  warps =1;
  blocks = 1;
  }
   srand(2009);
   h_params[0] = byteCount;
   for (uint i = 1; i <= byteCount; i++)
   {
        h_params[i] = rand() % 256;
   }
  sdkResetTimer(&hTimer);
  sdkStartTimer(&hTimer); 
  for(k=0; k < AVG_RUNS ; k++) {
   for(j=0; j<NUM_TASKS/LOOP_SIZE; j++){
        int i;
        for(i=0; i<LOOP_SIZE; i++){
                uint *d_params = (uint *) gemtcGPUMalloc(size);
                gemtcMemcpyHostToDevice(d_params, h_params, size);
                gemtcPush(34, 32, i+j*LOOP_SIZE, d_params);
        }

        for(i=0; i<LOOP_SIZE; i++){
                void *ret=NULL;
                int id;
                while(ret==NULL){
                gemtcPoll(&id, &ret);
                }
      // Copy back the results
                gemtcMemcpyDeviceToHost(h_params, ret, size);

      // Free the device pointer
                gemtcGPUFree(ret);
        }
  }
  }
  free(h_params);
  sdkStopTimer(&hTimer);
  dAvgSecs = 1.0e-3 * (double)sdkGetTimerValue(&hTimer) / (double) AVG_RUNS;
  dAvgSecs = dAvgSecs/(NUM_TASKS/LOOP_SIZE);
  printf("%u\t%.4f\t%.5f\n",
  byteCount,(1.0e-6 * warps * (double)byteCount / dAvgSecs), dAvgSecs);
  byteCount *= 10;
  gemtcCleanup();
  }
  //printf("Completed\n");
  sdkDeleteTimer(&hTimer);
  return 0;
}

