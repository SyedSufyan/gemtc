#include "../../../gemtc.cu"
#include<stdio.h>
#include<stdlib.h>
#define BIN_COUNT 256
#define NUM_RUNS 5 
#define AVG_RUNS 10.0 
#define BYTE_COUNT 25600
#include <helper_functions.h>
#include <hip/hip_runtime.h>

int main(int argc, char **argv){
    int NUM_TASKS, LOOP_SIZE;
    uint byteCount = BYTE_COUNT;
    int Overfill = 0;
    /*if(argc&gt;2){
        NUM_TASKS = atoi(argv[1]);
        LOOP_SIZE = atoi(argv[2]);
        
        }else{
        printf("This test requires four parameters:\n");
        printf("   int NUM_TASKS, int LOOP_SIZE, int MATRIX_SIZE, int STATIC_VALUE\n");
        printf("where  NUM_TASKS is the total numer of vector add tasks to be sent to gemtc\n");
        printf("       LOOP_SIZE is the number of tasks should be sent to gemtc before waiting for results\n");
        exit(1);
    }*/
    
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, 0);
    StopWatchInterface *hTimer = NULL;
    int iter,warps;
    int blocks = devProp.multiProcessorCount;
    sdkCreateTimer(&hTimer);
    
    //Starting Iterating
    for(iter=0; iter < NUM_RUNS; iter++) {
        if(Overfill==1){
            warps = devProp.maxThreadsPerBlock/32;
        }
        if(Overfill==0){
            int coresPerSM = _ConvertSMVer2Cores(devProp.major, devProp.minor);
            warps = coresPerSM/16;  //A warp runs on 16 cores
        }
        if(Overfill==2){
            warps =1;
            blocks = 1;
        }
        NUM_TASKS = warps * blocks;
        LOOP_SIZE = 1;
        byteCount = byteCount / NUM_TASKS;
        
        gemtcSetup(25600, Overfill);
        int d_size = sizeof(unsigned int) * byteCount;
        int h_size = sizeof(int) * BIN_COUNT;
        int size = 1 + d_size + h_size;
        int j;
        int k;
        uint *h_params = (uint *) malloc(size);
        double dAvgSecs;
        
        
        srand(2009);
        h_params[0] = byteCount;
        //printf("ByteCount :%d , NUM_TASKS : %d \n", byteCount,NUM_TASKS);
        for (uint i = 1; i <= byteCount; i++)
        {
            h_params[i] = rand() % 256;
        }
        sdkResetTimer(&hTimer);
        sdkStartTimer(&hTimer);
        for(k=0; k < AVG_RUNS ; k++) {
            for(j=0; j <NUM_TASKS/LOOP_SIZE; j++){
                int i;
                for(i=0; i < LOOP_SIZE; i++){
                    uint *d_params = (uint *) gemtcGPUMalloc(size);
                    gemtcMemcpyHostToDevice(d_params, h_params, size);
                    gemtcPush(34, 32, i+j*LOOP_SIZE, d_params);
                }
                
                for(i=0; i < LOOP_SIZE; i++){
                    void *ret=NULL;
                    int id;
                    while(ret==NULL){
                        gemtcPoll(&id, &ret);
                    }
                    gemtcMemcpyDeviceToHost(h_params, ret, size);
                    gemtcGPUFree(ret);
                }
            }
        }
        free(h_params);
        sdkStopTimer(&hTimer);
        dAvgSecs = 1.0e-3 * (double)sdkGetTimerValue(&hTimer)/(double) AVG_RUNS;
        unsigned int problem_size = (byteCount * 4) * NUM_TASKS;
        
        //dAvgSecs = dAvgSecs/(NUM_TASKS/LOOP_SIZE);
        printf("%u \t%.4f\t%.5f\n",
        problem_size,(1.0e-6 * (double) problem_size / dAvgSecs), dAvgSecs);
        byteCount = byteCount * NUM_TASKS * 10;
        gemtcCleanup();
    }
    sdkDeleteTimer(&hTimer);
    return 0;
}
