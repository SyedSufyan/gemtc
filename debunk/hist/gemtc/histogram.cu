#include "../../../gemtc.cu"
#include<stdio.h>
#include<stdlib.h>
#define BIN_COUNT 256
//#define NUM_RUNS  5
//#define AVG_RUNS 10.0 
#define BYTE_COUNT 1000
#include <helper_functions.h>
#include <hip/hip_runtime.h>

int main(int argc, char **argv){
    int NUM_TASKS, LOOP_SIZE;
    uint byteCount = BYTE_COUNT;
    int Overfill = 0;
    if (argc != 3){
        printf("invalid parameters, use: <NUM_INPUTS> <NUM_TEST>\n");
    return -1;
    }
    int NUM_RUNS = atoi(argv[1]);
    int AVG_RUNS = atoi(argv[2]);    


    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, 0);
    StopWatchInterface *hTimer = NULL;
    int iter,warps;
    int blocks = devProp.multiProcessorCount;
    sdkCreateTimer(&hTimer);
    
    //Starting Iterating
    for(iter=0; iter < NUM_RUNS; iter++) {
        if(Overfill==1){
            warps = devProp.maxThreadsPerBlock/32;
        }
        if(Overfill==0){
            int coresPerSM = _ConvertSMVer2Cores(devProp.major, devProp.minor);
            warps = coresPerSM/16;  //A warp runs on 16 cores
        }
        if(Overfill==2){
            warps =1;
            blocks = 1;
        }
        NUM_TASKS = warps * blocks*AVG_RUNS;
        LOOP_SIZE = 1;
        byteCount = byteCount / NUM_TASKS;
        
        gemtcSetup(25600, Overfill);
        int d_size = sizeof(unsigned int) * byteCount;
        int h_size = sizeof(int) * BIN_COUNT;
        int size = 1 + d_size + h_size;
        int j;
        int k;
        uint *h_params = (uint *) malloc(size);
        double dAvgSecs;
        
        
        srand(2009);
        h_params[0] = byteCount;
        //printf("ByteCount :%d , NUM_TASKS : %d \n", byteCount,NUM_TASKS);
        for (uint i = 1; i <= byteCount; i++)
        {
            h_params[i] = rand() % 256;
        }
        sdkResetTimer(&hTimer);
        sdkStartTimer(&hTimer);
        //for(k=0; k < AVG_RUNS ; k++) {
            for(j=0; j <NUM_TASKS; j++){
                //for(i=0; i < LOOP_SIZE; i++){
                    uint *d_params = (uint *) gemtcGPUMalloc(size);
                    gemtcMemcpyHostToDevice(d_params, h_params, size);
                    gemtcPush(34, 32, j*LOOP_SIZE, d_params);
                //}
                
                /*for(i=0; i < LOOP_SIZE; i++){
                    void *ret=NULL;
                    int id;
                    while(ret==NULL){
                        gemtcPoll(&id, &ret);
                    }
                    gemtcMemcpyDeviceToHost(h_params, ret, size);
                    gemtcGPUFree(ret);
                }*/
            }
		    void *ret=NULL;
                    int id;
                    while(ret==NULL){
                        gemtcPoll(&id, &ret);
                    }
                    gemtcMemcpyDeviceToHost(h_params, ret, size);
                    gemtcGPUFree(ret);

        //}
        free(h_params);
        sdkStopTimer(&hTimer);
        dAvgSecs = 1.0e-3 * (double)sdkGetTimerValue(&hTimer);///(double) AVG_RUNS;
        unsigned int problem_size = (byteCount * 4) * NUM_TASKS;
        
        //dAvgSecs = dAvgSecs/(NUM_TASKS/LOOP_SIZE);
        printf("%u \t%.4f\t%.5f\n",
        problem_size,(1.0e-6 * (double) problem_size / dAvgSecs), dAvgSecs);
        byteCount = byteCount * NUM_TASKS * 10;
        gemtcCleanup();
    }
    sdkDeleteTimer(&hTimer);
    return 0;
}
