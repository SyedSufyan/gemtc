#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include "hist.h"
#define BIN_COUNT 256

//#define NUM_RUNS 5 
//#define NUM_TEST 10.0
#define BYTE_COUNT 25600
#define CHECK_ERR(x)                                    \
  if (x != hipSuccess) {                               \
    fprintf(stderr,"%s in %s at line %d\n",             \
            hipGetErrorString(err),__FILE__,__LINE__); \
    exit(-1);                                           \
  }

__global__ void
histogram(double *buffer,long size,unsigned int *histo ) {
  __shared__ unsigned int temp[256];
  temp[threadIdx.x] = 0;
   __syncthreads();
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;
  while (i < size)
  {
          atomicAdd( &(temp[(int)buffer[i]]), 1 );
              i += stride;
  }
  __syncthreads();
  atomicAdd( &(histo[threadIdx.x]), temp[threadIdx.x] );
  
}

double* hist(double *h_data, int byteCount)
{
    //unsigned char * h_data;
    unsigned int h_histogram[BIN_COUNT];
    double * d_data;
    unsigned int * d_histogram;
    //unsigned int byteCount = BYTE_COUNT;
    size_t size;
    hipError_t err;

    int NUM_RUNS = 1;
    int NUM_TEST =10;

    StopWatchInterface *hTimer = NULL;
    int iter;
    sdkCreateTimer(&hTimer);
    hipDeviceProp_t prop;
    checkCudaErrors( hipGetDeviceProperties( &prop, 0 ) );
    int blocks = prop.multiProcessorCount;
    for(iter =0 ; iter < NUM_RUNS;iter++){
        srand (time(NULL));
        size = sizeof(double) * byteCount;
        /*h_data = (unsigned char *) malloc(sizeof(unsigned char) * byteCount);
        for (unsigned int i = 0; i < byteCount; i++)
        {
            h_data[i] = rand() % 256;
        }*/
        sdkResetTimer(&hTimer);
        sdkStartTimer(&hTimer);
        int j;
        for(j=0; j <  NUM_TEST; j++) {
            err=hipMalloc((void **) &d_data, size);
            CHECK_ERR(err);
            err=hipMalloc((void **) &d_histogram, sizeof(unsigned int) * BIN_COUNT);
            CHECK_ERR(err);
            err = hipMemcpy(d_data,h_data,size,hipMemcpyHostToDevice);
            CHECK_ERR(err);
            err = hipMemcpy(d_histogram,h_histogram,sizeof(unsigned int) * BIN_COUNT, hipMemcpyHostToDevice);
            CHECK_ERR(err);
            histogram<<<blocks,BIN_COUNT>>>(d_data,byteCount,d_histogram);
            hipDeviceSynchronize();
            //Copy back the results from the device
            err = hipMemcpy(h_histogram,d_histogram,sizeof(unsigned int) * BIN_COUNT,hipMemcpyDeviceToHost);
            CHECK_ERR(err);
            //print(h_histogram);
            hipFree(d_data);
            hipFree(d_histogram);
        }
        sdkStopTimer(&hTimer);
        //free(h_data);
        unsigned int problem_size = byteCount * 8;
        double dAvgSecs = 1.0e-3 * (double)sdkGetTimerValue(&hTimer) / (double) NUM_TEST;
        printf("%u\t%.4f\t%.5f\n",
        problem_size,(1.0e-6 * (double)problem_size / dAvgSecs), dAvgSecs);
        byteCount = byteCount * 10;
    }
    // Print timing information
    sdkDeleteTimer(&hTimer);
    double* result = (double *)malloc(sizeof(double));
    result[0] = 0;
    return result;
}
