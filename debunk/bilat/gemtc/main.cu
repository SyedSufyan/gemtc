#include "hip/hip_runtime.h"
#include "../../../gemtc.cu"
#include<stdio.h>
#include<stdlib.h>
#include <helper_functions.h>
#include <hip/hip_runtime.h>
#define NUM_TEST 10
#define TEST_RUN 7
#define IMAGE_SIZE 1024

typedef struct {
        float R;
        float G;
        float B;

} RGB;



int main(int argc, char **argv){
    int NUM_TASKS, LOOP_SIZE;
    int Overfill = 0;
    if (argc != 5){
        printf("invalid parameters, use:  <channels> <neighborhood radius> <spatial sigma> <range sigma>\n");
	return -1;
    }
    const unsigned int channels = atoi(argv[1]);
    unsigned int radius = atoi(argv[2]);
    float sigma_spatial = (float)atof(argv[3]);
    float sigma_range = (float)atof(argv[4]);
	 
    unsigned int width = IMAGE_SIZE;
    unsigned int height = IMAGE_SIZE;
    
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, 0);
    StopWatchInterface *hTimer = NULL;
    sdkCreateTimer(&hTimer);
    
    /*int warps;
    int blocks = devProp.multiProcessorCount;
	
	if(Overfill==1){
		warps = devProp.maxThreadsPerBlock/32;
	}
	if(Overfill==0){
		int coresPerSM = _ConvertSMVer2Cores(devProp.major, devProp.minor);
		warps = coresPerSM/16;  //A warp runs on 16 cores
	}
	if(Overfill==2){
		warps =1;
		blocks = 1;
	}
     */
	for(int i = 0; i < TEST_RUN; i++)
    {
		int d_size = sizeof(RGB) * width * height;
		
        int size = 2 + 2*d_size;
		
        RGB *data=(RGB *) malloc (size);
        
		RGB r_size;
		r_size.R = width * height;
		r_size.G = width;
		r_size.B = height;
		
		data[0] = r_size;
				
		RGB r;
		r.R = channels;
		r.G =  radius;
		r.B = sigma_spatial;
		data[1] = r;
		
		RGB r1;
		r1.R = sigma_range;
		data[2] = r1;
		
		
		for(int j = 3; j < (width * height) + 3; j++)
         {
            RGB c;
            c.R = rand() % 255;
            c.G = rand() % 255;
            c.B =  rand() % 255;
            data[j] = c;
         }

		gemtcSetup(25600, Overfill);
		
		
		sdkResetTimer(&hTimer);
		sdkStartTimer(&hTimer);
		
		for(int k=0; k < NUM_TEST ; k++) {
            for(int j=0; j <NUM_TASKS/LOOP_SIZE; j++){
                int i;
                for(i=0; i < LOOP_SIZE; i++){
                    float3 *d_params = (float3 *) gemtcGPUMalloc(size);
                    gemtcMemcpyHostToDevice(d_params, data, size);
                    gemtcPush(34, 32, i+j*LOOP_SIZE, d_params);
                }
                
                for(i=0; i < LOOP_SIZE; i++){
                    void *ret=NULL;
                    int id;
                    while(ret==NULL){
                        gemtcPoll(&id, &ret);
                    }
                    gemtcMemcpyDeviceToHost(data, ret, size);
                    gemtcGPUFree(ret);
                }
            }
        }
		free(data);
		sdkStopTimer(&hTimer);
		double dAvgSecs = 1.0e-3 * (double)sdkGetTimerValue(&hTimer) / (double) NUM_TEST;
		printf("%u\t%.5f\n",width*height, dAvgSecs);
		gemtcCleanup();
		width *= 2;
		
	}
    sdkDeleteTimer(&hTimer);
    return 0;	
	
}

