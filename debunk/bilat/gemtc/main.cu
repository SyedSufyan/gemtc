#include "hip/hip_runtime.h"
#include "../../../gemtc.cu"
#include<stdio.h>
#include<stdlib.h>
#include <helper_functions.h>
#include <hip/hip_runtime.h>
//#define NUM_TEST 1
//#define TEST_RUN 4 
#define IMAGE_SIZE 1024

typedef struct {
        float R;
        float G;
        float B;

} RGB;

int main(int argc, char **argv){

    int NUM_TASKS=1, LOOP_SIZE=1;
    int Overfill = 0;
    if (argc != 3){
        printf("invalid parameters, use:  <NUM_INPUTS> <NUM_TEST>\n");
        return -1;
    }
    const unsigned int channels = 1;//atoi(argv[1]);
    unsigned int radius = 1;//atoi(argv[2]);
    float sigma_spatial = 1.0;//(float)atof(argv[3]);
    float sigma_range = 1.0;//(float)atof(argv[4]);

    unsigned int width = IMAGE_SIZE;
    unsigned int height = IMAGE_SIZE;

    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, 0);
    StopWatchInterface *hTimer = NULL;
    sdkCreateTimer(&hTimer);
    int TEST_RUN = atoi(argv[1]);
    int NUM_TEST = atoi(argv[2]);

    /*int warps;
    int blocks = devProp.multiProcessorCount;

        if(Overfill==1){
                warps = devProp.maxThreadsPerBlock/32;
        }
        if(Overfill==0){
                int coresPerSM = _ConvertSMVer2Cores(devProp.major, devProp.minor);
                warps = coresPerSM/16;  //A warp runs on 16 cores
        }
        if(Overfill==2){
                warps =1;
                blocks = 1;
        }
     */
	for(int i = 0; i < TEST_RUN; i++)
    	{
		int d_size = sizeof(RGB) * width * height;

        	int size = 3 + 2*d_size;

	        RGB *data=(RGB *) malloc (size);

		RGB r_size;
		r_size.R = width * height;
		r_size.G = width;
		r_size.B = height;

		data[0] = r_size;

		RGB r;
		r.R = channels;
		r.G =  radius;
		r.B = sigma_spatial;
		data[1] = r;

		RGB r1;
		r1.R = sigma_range;
		data[2] = r1;


		for(int j = 3; j < (width * height) + 3; j++)
		 {
			RGB c;
			c.R = rand() % 255;
			c.G = rand() % 255;
			c.B =  rand() % 255;
			data[j] = c;
		 }

		//gemtcSetup(25600, Overfill);


		sdkResetTimer(&hTimer);
		sdkStartTimer(&hTimer);

		for(int k=0; k < NUM_TEST ; k++) {
			gemtcSetup(25600, Overfill);
			for(int j=0; j <NUM_TASKS/LOOP_SIZE; j++){
				int x;
				for(x=0; x < LOOP_SIZE; x++){
					float3 *d_params = (float3 *) gemtcGPUMalloc(size);
					gemtcMemcpyHostToDevice(d_params, data, size);
					gemtcPush(35, 32, i+j*LOOP_SIZE, d_params);
				}

				for(x=0; x < LOOP_SIZE; x++){
					void *ret=NULL;
					int id;
					while(ret==NULL){
						gemtcPoll(&id, &ret);
					}
					gemtcMemcpyDeviceToHost(data, ret, size);
					gemtcGPUFree(ret);
				}
			}
			gemtcCleanup();
		}
		free(data);
		sdkStopTimer(&hTimer);
		double dAvgSecs = 1.0e-3 * (double)sdkGetTimerValue(&hTimer) / (double) NUM_TEST;
		printf("%u\t%.5f\n",width*height, dAvgSecs);
		//gemtcCleanup();
		width *= 2;

	}
    sdkDeleteTimer(&hTimer);
    return 0;

}

