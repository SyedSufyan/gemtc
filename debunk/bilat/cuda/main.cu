#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include "../helper/uint_util.hcu"
#include "../helper/float_util.hcu"
//#include "common/error.h"

#include <math.h>

#define PI 3.14159265
#define IMAGE_SIZE 1024

#define CHECK_ERR(x)                                    \
  if (x != hipSuccess) {                               \
    fprintf(stderr,"%s in %s at line %d\n",             \
            hipGetErrorString(err),__FILE__,__LINE__); \
    exit(-1);                                           \
  }


unsigned int radius;
float sigma_spatial;
float sigma_range;
typedef struct {
        float R;
        float G;
        float B;

} RGB;

__host__ __device__
float gaussian1d(float x, float sigma)
{
        float variance = pow(sigma,2);
        float exponent = -pow(x,2)/(2*variance);
        return expf(exponent) / sqrt(2 * PI * variance);
}

__host__ __device__
float gaussian2d(float x, float y, float sigma)
{
        float variance = pow(sigma,2);
        float exponent = -(pow(x,2) + pow(y,2))/(2*variance);
        return expf(exponent) / (2 * PI * variance);
}
__global__
void bilateralFilterGPU_v1(float3* input, float3* output, uint2 dims, int radius, float sigma_spatial, float sigma_range)
{
        const unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

        uint2 pos = idx_to_co(idx,dims);
        int img_x = pos.x;
        int img_y = pos.y;

        if(img_x >= dims.x || img_y >= dims.y) return;

        float3 currentColor = input[idx];

        float3 res = make_float3(0.0f,0.0f,0.0f);
        float3 normalization = make_float3(0.0f,0.0f,0.0f);;


        for(int i = -radius; i <= radius; i++) {
                for(int j = -radius; j <= radius; j++) {
                        int x_sample = img_x+i;
                        int y_sample = img_y+j;

                        //mirror edges
                        if( x_sample < 0) x_sample = -x_sample;
                        if( y_sample < 0) y_sample = -y_sample;
                        if( x_sample > dims.x - 1) x_sample = dims.x - 1 - i;
                        if( y_sample > dims.y - 1) y_sample = dims.y - 1 - j;


                        float3 tmpColor = input[co_to_idx(make_uint2(x_sample,y_sample),dims)];

                        float gauss_spatial = gaussian2d(i,j,sigma_spatial);
                        float3 gauss_range;
                        gauss_range.x = gaussian1d(currentColor.x - tmpColor.x, sigma_range);
                        gauss_range.y = gaussian1d(currentColor.y - tmpColor.y, sigma_range);
                        gauss_range.z = gaussian1d(currentColor.z - tmpColor.z, sigma_range);

                        float3 weight = gauss_spatial * gauss_range;
                        normalization = normalization + weight;
                        res = res + (tmpColor * weight);

                }
        }

        res.x /= normalization.x;
        res.y /= normalization.y;
        res.z /= normalization.z;
        output[idx] = res;
}

void bilateralFiltering_v1(RGB* data, int width, int height ,int radius, float sigma_spatial, float sigma_range) {
    unsigned int numElements = width * height;
    hipError_t err;
    // copy data to device
    float3* d_data;
    err= hipMalloc( (void**) &d_data, numElements*sizeof(RGB));
    CHECK_ERR(err);
    err= hipMemcpy( d_data, data, numElements*sizeof(RGB), hipMemcpyHostToDevice );
    CHECK_ERR(err);

        //Output image
    float3* d_result;
    err= hipMalloc( (void**) &d_result, numElements*sizeof(RGB));
    CHECK_ERR(err);

    // setup dimensions of grid/blocks.
    dim3 blockDim(512,1,1);
    dim3 gridDim((unsigned int) ceil((double)(numElements/blockDim.x)), 1, 1 );

    // invoke kernel
        bilateralFilterGPU_v1<<< gridDim, blockDim >>>( d_data, d_result, make_uint2(width,height), radius, sigma_spatial, sigma_range);

    // copy data to host
    err= hipMemcpy( data, d_result, numElements*sizeof(RGB), hipMemcpyDeviceToHost );
    CHECK_ERR(err);
        hipFree(d_data);
        hipFree(d_result);

}

/*(RGB*) populateRGB(int width, int height){
int i=0;
int j=0;
srand (time(NULL));
RGB* colors = (RGB *) malloc (sizeof(RGB) * width * height);
for(int i = 0; i < numElements; i++)
        {
                RGB c;
                c.R = rand() % 255;
                c.G = rand() % 255;
                c.B =  rand() % 255;
                colors[i] = c;
        }
return colors;
}
*/
int main(int argc, char** argv) {
    if (argc != 3){
        printf("invalid parameters, use: <NUM_INPUTS> <NUM_TEST>\n");
    return -1;
    }
    
        //const unsigned int channels = 1;//atoi(argv[1]);
	StopWatchInterface *hTimer = NULL;
	unsigned int width = IMAGE_SIZE;
	unsigned int height = IMAGE_SIZE;
	sdkCreateTimer(&hTimer);
        radius = 1;//atoi(argv[2]);
        sigma_spatial = 1.0; //(float)atof(argv[3]);
        sigma_range = 1.0; //(float)atof(argv[4]);
	int TEST_RUN = atoi(argv[1]);
	int NUM_TEST = atoi(argv[2]);
        for(int i = 0; i < TEST_RUN; i++)
        {
			RGB *data=(RGB *) malloc (sizeof(RGB) * width * height);
			for(int j = 0; j < (width * height); j++)
			{
					RGB c;
					c.R = rand() % 255;
					c.G = rand() % 255;
					c.B =  rand() % 255;
					data[j] = c;
			}
			
			
			sdkResetTimer(&hTimer);
			sdkStartTimer(&hTimer);
			for(int k = 0; k < NUM_TEST; k++)
			{
				bilateralFiltering_v1(data,width,height,radius,sigma_spatial,sigma_range);
			}
			sdkStopTimer(&hTimer);
			double dAvgSecs = 1.0e-3 * (double)sdkGetTimerValue(&hTimer) / (double) NUM_TEST;
			printf("%u\t%.5f\n",width*height, dAvgSecs);
			free(data);
			
			width *= 2;
		}
		
	sdkDeleteTimer(&hTimer);
    return EXIT_SUCCESS;
}

