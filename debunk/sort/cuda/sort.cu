#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#define NUM_RUNS 6 
#define BYTE_COUNT 1000
#define NUM_TEST 10
#define CHECK_ERR(x)                                    \
  if (x != hipSuccess) {                               \
    fprintf(stderr,"%s in %s at line %d\n",             \
            hipGetErrorString(err),__FILE__,__LINE__); \
    exit(-1);                                           \
  }


// Forward declaration of partition_by_bit(), called by sort()
__device__ void partition_by_bit(unsigned int *values, unsigned int bit,int n);

__global__ void sort(unsigned int *values,int n)
{
    int  bit;
    unsigned int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if(i < n) {
   if(threadIdx.x == 64)
    printf("%d - %d\n",i, blockIdx.x); 
    for( bit = 0; bit < 32; ++bit )
    {
        partition_by_bit(values, bit,n);
        __syncthreads();
    }
    
}
}

template<class T>
__device__ T plus_scan(T *x,int n)
{
    unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;

    //unsigned int i = threadIdx.x; // id of thread executing this instance

    unsigned int num = blockDim.x;// * (blockIdx.x +1);  // total number of threads in this block
    unsigned int offset;          // distance between elements to be added
   if(i < n){ 
   for( offset = 1; offset < num; offset *= 2) {
        T t;

        if ( i >= offset ) 
            t = x[i-offset];
        
        __syncthreads();

        if ( i >= offset ) 
            x[i] = t + x[i];      // i.e., x[i] = x[i] + x[i-1]

        __syncthreads();
    }
    //printf("Plus Scan - %d \n", x[i]);
    return x[i];
}
return 0;
}

__device__ void partition_by_bit(unsigned int *values, unsigned int bit,int n)
{
    unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int size = blockDim.x* (blockIdx.x + 1);
    if(i < n){
    unsigned int x_i = values[i];          // value of integer at position i
    unsigned int p_i = (x_i >> bit) & 1;   // value of bit at position bit

    values[i] = p_i;  

    // Wait for all threads to finish this.
    __syncthreads();

    unsigned int T_before = plus_scan(values,n);
    if(size -1 < n){
    unsigned int T_total  = values[size-1];
    unsigned int F_total  = size - T_total;
    __syncthreads();
	
    if (p_i)
        values[T_before-1 + F_total] = x_i;
    else
        values[i - T_before] = x_i;
   }
}
    //printf("BlockId Id %d - %d - %d \n", blockIdx.x,i,size);
    
}
void print(unsigned int *values, int size){
   for(int i=0; i < size; i++){
       printf("%d\n", values[i]);

   }
}
int main(int argc, char *argv[])
{
    unsigned int * h_data;
    unsigned int * d_data;

    size_t size;
    hipError_t err;
    StopWatchInterface *hTimer = NULL;
    int iter;
    sdkCreateTimer(&hTimer);
    hipDeviceProp_t prop;
    checkCudaErrors( hipGetDeviceProperties( &prop, 0 ) );
    int blocks = prop.multiProcessorCount;
    int byteCount = BYTE_COUNT;
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, 0);
    int numthreads = devProp.maxThreadsPerBlock;
    for(iter =0 ; iter < NUM_RUNS;iter++){
        srand (time(NULL));
        size = sizeof(unsigned int) * byteCount;
        h_data = (unsigned int *) malloc(sizeof(unsigned int) * byteCount);
        for (int i = 0; i < byteCount; i++)
        {
            h_data[i] = rand() % 1024;
//  	    printf("Rand %d\n", h_data[i]);
        }
        sdkResetTimer(&hTimer);
        sdkStartTimer(&hTimer);
        int j;
        for(j=0; j <  NUM_TEST; j++) {
            err=hipMalloc((void **) &d_data, size);
            CHECK_ERR(err);
            
            err = hipMemcpy(d_data,h_data,size,hipMemcpyHostToDevice);
            CHECK_ERR(err);
              
            sort<<<blocks,numthreads>>>(d_data, byteCount);
            hipDeviceSynchronize();
            //Copy back the results from the device
            err = hipMemcpy(h_data,d_data,size,hipMemcpyDeviceToHost);
            CHECK_ERR(err);
        //    print(h_data,byteCount);
	    		
            hipFree(d_data);
            
        }
        sdkStopTimer(&hTimer);
        free(h_data);
        unsigned int problem_size = byteCount * 4;
        double dAvgSecs = 1.0e-3 * (double)sdkGetTimerValue(&hTimer) / NUM_TEST;
        printf("%u\t%.4f\t%.5f\n",
        byteCount,(1.0e-6 * (double)problem_size / dAvgSecs), dAvgSecs);
        byteCount = byteCount * 10;
    }
    // Print timing information
    sdkDeleteTimer(&hTimer);
}

