#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "AddSleep.cu"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


int NUM_THREADS;
int JOBS_PER_THREAD;
int QUEUE_SIZE=12800;
int SLEEP_TIME;
int MALLOC_SIZE;
int LOOP_SIZE;

extern "C"
void sleep_wrapper(int SLEEP_TIME);

/*
int main(int argc, char **argv){
  printf("Starting AddSleep Test\n");
  if(argc>4){
    NUM_THREADS = atoi(argv[1]);
    JOBS_PER_THREAD = atoi(argv[2]);
    SLEEP_TIME = atoi(argv[3]);
    MALLOC_SIZE = atoi(argv[4]);
    LOOP_SIZE = atoi(argv[5]);
  }else{
    printf("This test requires five parameters:\n");
    printf("   int NUM_THREADS, int JOBS_PER_THREAD, int SLEEP_TIME, int MALLOC_SIZE, int LOOP_SIZE\n");
    printf("where  NUM_THREADS is the number of seperate threads that will be sending work into gemtc\n");
    printf("       JOBS_PER_THREAD is the number of tasks that a given thread will submit to gemtc\n");
    printf("       SLEEP_TIME is the parameter that will be given to each AddSleep micro-kernel, in microseconds\n");
    printf("       MALLOC_SIZE is the amount of memory that will be allocated and transfered with each sleep\n");
    printf("                   This number must be a multiple of 4, to comply with cuda's memory requirements\n");
    printf("       LOOP_SIZE is the number of tasks a thread will submit to gemtc before waiting for results\n");
    exit(1);
  }
  sleep_wrapper(SLEEP_TIME);
  return 0;
}
*/
void sleep_wrapper(int SLEEP_TIME){
  
  int nkernels = 1;               // number of concurrent kernels                                     
  int nstreams = nkernels + 1;    // use one more stream than concurrent kernel                       
  int nbytes = nkernels * sizeof(clock_t);   // number of data bytes                                  
  float kernel_time = SLEEP_TIME; // time the kernel should run in ms                                 
  float elapsed_time;   // timing variables                                                           
  int cuda_device = 0;
  
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  if (deviceCount == 0) {
    fprintf(stderr, "error: no devices supporting CUDA.\n");
    exit(EXIT_FAILURE);
  }
  int dev = 0;
  hipSetDevice(dev);
  hipDeviceProp_t prop;
  if (hipGetDeviceProperties(&prop, dev) == hipSuccess){
    printf("Using device %d:\n", dev);
    printf("%s; global mem: %dB; compute v%d.%d; clock: %d kHz\n",
	   prop.name, (int)prop.totalGlobalMem, (int)prop.major, 
	   (int)prop.minor, (int)prop.clockRate);
  }
  
  clock_t time_clocks = (clock_t)(kernel_time * (int)prop.clockRate);
  
  // allocate host memory                                                                             
  clock_t *a = 0;                     // pointer to the array data in host memory                     
  // allocate device memory                                                                           
  clock_t *d_a = 0;             // pointers to data and init value in the device memory              
  hipMalloc((void **)&d_a, nbytes);
  
  // run the task
  //  wrapAddSleep<<<1,1>>>(d_sleepTime);
  clock_block<<<1,1>>>(&d_a[0], time_clocks);
  
  // wait
  hipDeviceSynchronize();
  
  // return
  // return 0;
}
