#include <stdio.h>
#include <hip/hip_runtime.h>
#include <pthread.h>

hipStream_t stream_dataIn, stream_dataOut, stream_kernel;

pthread_mutex_t memcpyLock;
pthread_mutex_t enqueueLock;
pthread_mutex_t dequeueLock;

int *d_kill;

#include "DataMovement.cu"
#include "malloc/GemtcMalloc.cu"
#include "Queues/QueueJobs.cu"
#include "SuperKernel.cu"

Queue d_newJobs, d_finishedJobs;

/*
This file contains the functions that make up the API to gemtc
They are:
*** Initialize/Deconstruct ***
  gemtcSetup()
  gemtcCleanup()

*** EnQueue/DeQueue Tasks  ***
  gemtcBlockingRun()
  gemtcPush()
  gemtcPoll()

*** Memory Transfer Calls  ***
  gemtcMemcpyHostToDevice()
  gemtcMemcpyDeviceToHost()

****Memory Management Calls***
  gemtcGPUMalloc()
  gemtcGPUFree()
 */


/////////////////////
//Utility Functions//
/////////////////////
void *moveToCuda(void *val, int size){
  void *ret = gemtcMalloc(size);
  cudaSafeMemcpy(ret, val, size, 
                 hipMemcpyHostToDevice, stream_dataIn, 
                 "in moveToCuda of run()");
  return ret;
}
void *moveFromCuda(void *val, int size){
  void *ret = malloc(size);
  cudaSafeMemcpy(ret, val, size, 
                 hipMemcpyDeviceToHost, stream_dataOut, 
                 "in moveFromCuda of run()");
  return ret;
}

/////////////////
//API Functions//
/////////////////
extern "C"
void gemtcSetup(int QueueSize){
  //initialize locks
  pthread_mutex_init(&memcpyLock, NULL);
  pthread_mutex_init(&enqueueLock, NULL);
  pthread_mutex_init(&dequeueLock, NULL);
  pthread_mutex_init(&memoryListLock, NULL);

  //Default sizes for SuperKernel
  // Eventually this should read from a config file
  int warp_size = 32;
  int warps = 32;
  int blocks = 7;

  dim3 threads(warp_size*warps, 1, 1);
  dim3 grid(blocks, 1, 1);

  //Init Streams for the SuperKernel and various memory copies
  hipStreamCreate(&stream_kernel);
  hipStreamCreate(&stream_dataIn);
  hipStreamCreate(&stream_dataOut);
  
  //Initialize Device Memory with Queues
  d_newJobs = CreateQueue(QueueSize);
  d_finishedJobs = CreateQueue(QueueSize);

  //Initialize kill flag in device Memory
  int temp = 0;
  d_kill = (int *) moveToCuda((void *)&temp, sizeof(int));

  hipDeviceSynchronize();

//Launch the super kernel
  superKernel<<< grid, threads, 8192, stream_kernel>>>  //8192 = 8kBytes of shared Memory
             (d_newJobs, d_finishedJobs, d_kill);
}


extern "C"
void gemtcBlockingRun(int Type, int Threads, int ID, void *d_params){
  //This funcyion will enqueue the given task to the device
  //Then block until it returns
  //   This is busy blocking where it polls the GPU to see if it finished
  JobPointer h_JobDescription = (JobPointer) malloc(sizeof(JobDescription));
  h_JobDescription->JobType = Type;
  h_JobDescription->numThreads = Threads;
  h_JobDescription->params = d_params;
  h_JobDescription->JobID = ID;

  pthread_mutex_lock(&enqueueLock);  //Start Critical Section

  EnqueueJob(h_JobDescription, d_newJobs);

  pthread_mutex_unlock(&enqueueLock); //End Critical Section

  int first = 1;
  while(h_JobDescription->JobID!=ID || first){
    //Loop until our task is at the front of the result queue
    // Non-ideal because the task could finish but take awhile to move
    // through the queue. Cannot fix this problem with current DataStruct
    pthread_yield();
    pthread_mutex_lock(&dequeueLock);
    h_JobDescription = FrontResult(d_finishedJobs);
    if(h_JobDescription->JobID==ID)DequeueResult(d_finishedJobs);
    pthread_mutex_unlock(&dequeueLock);
    first = 0;
    printf("Current front:%d  MyID:%d\n", h_JobDescription->JobID, ID);
  }
  //printf("Recieved result on Job #%d\n", ID);
}

extern "C"
void gemtcCleanup(){
  int temp=1;
  cudaSafeMemcpy(d_kill, &temp, sizeof(int), hipMemcpyHostToDevice, 
                 stream_dataIn, "Writing the kill command to SuperKernel");

  //Wait for SuperKernel to die
  hipEvent_t Super_Kernel_Finished;
  hipEventCreate(&Super_Kernel_Finished);
  hipEventRecord(Super_Kernel_Finished, stream_kernel);
  hipEventSynchronize(Super_Kernel_Finished);

  hipError_t e = hipGetLastError();

  DisposeQueue(d_newJobs);

  DisposeQueue(d_finishedJobs);

  hipStreamDestroy(stream_kernel);
  hipStreamDestroy(stream_dataIn);
  hipStreamDestroy(stream_dataOut);

  pthread_mutex_destroy(&memcpyLock);
  pthread_mutex_destroy(&enqueueLock);
  pthread_mutex_destroy(&dequeueLock);
  pthread_mutex_destroy(&memoryListLock);
}

extern "C"
void gemtcPush(int taskType, int threads, int ID, void *d_parameters){
  //Enqueue the given task to the device
  //Returns as soon as the task is in Device Memory
  JobPointer h_JobDescription = (JobPointer) malloc(sizeof(JobDescription));
  h_JobDescription->JobType = taskType;
  h_JobDescription->numThreads = threads;
  h_JobDescription->params = d_parameters;
  h_JobDescription->JobID = ID;

  pthread_mutex_lock(&enqueueLock);  //Start Critical Section

  EnqueueJob(h_JobDescription, d_newJobs);

  pthread_mutex_unlock(&enqueueLock); //End Critical Section
}

extern "C"
void gemtcPoll(int *ID, void **params){
  //This function has no input parameters and two result parameters.
  //  ID and params are used as references to where the result will be written

  //This function will check the device queues for any tasks that finished
  //If none are found:
  //   Value at ID will be set to -1
  //   Value at params will be set to NULL
  //If a finished task is in the queue:
  //   Value at ID will be that task's ID
  //   Value at params will be a pointer to that tasks parameters

  JobPointer h_JobDescription = (JobPointer) malloc(sizeof(JobDescription));
  pthread_mutex_lock(&dequeueLock);  //Start Critical Section
  h_JobDescription = MaybeFandD(d_finishedJobs);//returns null if empty
  pthread_mutex_unlock(&dequeueLock); //End Critical Section
  if(h_JobDescription==NULL){
    free(h_JobDescription);
    *ID=-1;
    *params=NULL;
    return;
  }

  *ID = h_JobDescription->JobID;
  *params = h_JobDescription->params;

  free(h_JobDescription);
}

extern "C"
void gemtcMemcpyHostToDevice(void *device, void *host, int size){
  cudaSafeMemcpy(device, host, size, 
                 hipMemcpyHostToDevice, stream_dataIn, 
                 "HostToDevice API call");
}

extern "C"
void gemtcMemcpyDeviceToHost(void *host, void *device, int size){
  cudaSafeMemcpy(host, device, size, 
                 hipMemcpyDeviceToHost, stream_dataOut, 
                 "DeviceToHost API call");
}

extern "C"
void *gemtcGPUMalloc(int size){
  //This is defined in malloc/gemtcMalloc.cu
  return gemtcMalloc(size);
}

extern "C"
void gemtcGPUFree(void *p){
  //This is defined in malloc/gemtcMalloc.cu
  gemtcFree(p);
}
