#include <stdio.h>
#include <hip/hip_runtime.h>
#include <pthread.h>

hipStream_t stream_dataIn, stream_dataOut, stream_kernel;

pthread_mutex_t memcpyLock;
pthread_mutex_t enqueueLock;
pthread_mutex_t dequeueLock;

int *d_kill;

#include "DataMovement.cu"
#include "malloc/GemtcMalloc.cu"
#include "Queues/QueueJobs.cu"
#include "SuperKernel.cu"

Queue d_newJobs, d_finishedJobs;

/*
This file contains the functions that make up the API to gemtc
They are:
*** Initialize/Deconstruct ***
  gemtcSetup()
  gemtcCleanup()

*** EnQueue/DeQueue Tasks  ***
  gemtcBlockingRun()
  gemtcPush()
  gemtcPoll()

*** Memory Transfer Calls  ***
  gemtcMemcpyHostToDevice()
  gemtcMemcpyDeviceToHost()

****Memory Management Calls***
  gemtcGPUMalloc()
  gemtcGPUFree()
 */


/////////////////////
//Utility Functions//
/////////////////////
void *moveToCuda(void *val, int size){
  void *ret = gemtcMalloc(size);
  cudaSafeMemcpy(ret, val, size, 
                 hipMemcpyHostToDevice, stream_dataIn, 
                 "in moveToCuda of run()");
  return ret;
}
void *moveFromCuda(void *val, int size){
  void *ret = malloc(size);
  cudaSafeMemcpy(ret, val, size, 
                 hipMemcpyDeviceToHost, stream_dataOut, 
                 "in moveFromCuda of run()");
  return ret;
}

/////////////////
//API Functions//
/////////////////
extern "C"
void gemtcSetup(int QueueSize){
  //initialize locks
  pthread_mutex_init(&memcpyLock, NULL);
  pthread_mutex_init(&enqueueLock, NULL);
  pthread_mutex_init(&dequeueLock, NULL);
  pthread_mutex_init(&memoryListLock, NULL);

  //Default sizes for SuperKernel
  // Eventually this should read from a config file
  int warp_size = 32;
  int warps = 8;
  int blocks = 14;

  dim3 threads(warp_size*warps, 1, 1);
  dim3 grid(blocks, 1, 1);

  //Init Streams for the SuperKernel and various memory copies
  hipStreamCreate(&stream_kernel);
  hipStreamCreate(&stream_dataIn);
  hipStreamCreate(&stream_dataOut);
  
  //Initialize Device Memory with Queues
  d_newJobs = CreateQueue(QueueSize);
  d_finishedJobs = CreateQueue(QueueSize);

  //Initialize kill flag in device Memory
  int temp = 0;
  d_kill = (int *) moveToCuda((void *)&temp, sizeof(int));

  hipDeviceSynchronize();

//Launch the super kernel
  superKernel<<< grid, threads, 8192, stream_kernel>>>  //8192 = 8kBytes of shared Memory
             (d_newJobs, d_finishedJobs, d_kill);
}


extern "C"
void gemtcBlockingRun(int Type, int Threads, int ID, void *d_params){
  //This funcyion will enqueue the given task to the device
  //Then block until it returns
  //   This is busy blocking where it polls the GPU to see if it finished
  JobPointer h_JobDescription = (JobPointer) malloc(sizeof(JobDescription));
  h_JobDescription->JobType = Type;
  h_JobDescription->numThreads = Threads;
  h_JobDescription->params = d_params;
  h_JobDescription->JobID = ID;

  pthread_mutex_lock(&enqueueLock);  //Start Critical Section

  EnqueueJob(h_JobDescription, d_newJobs);

  pthread_mutex_unlock(&enqueueLock); //End Critical Section

  int first = 1;
  while(h_JobDescription->JobID!=ID || first){
    //Loop until our task is at the front of the result queue
    // Non-ideal because the task could finish but take awhile to move
    // through the queue. Cannot fix this problem with current DataStruct
    pthread_yield();
    pthread_mutex_lock(&dequeueLock);
    h_JobDescription = FrontResult(d_finishedJobs);
    if(h_JobDescription->JobID==ID)DequeueResult(d_finishedJobs);
    pthread_mutex_unlock(&dequeueLock);
    first = 0;
    printf("Current front:%d  MyID:%d\n", h_JobDescription->JobID, ID);
  }
  //printf("Recieved result on Job #%d\n", ID);
}

extern "C"
void gemtcCleanup(){
  int temp=1;
  cudaSafeMemcpy(&temp, d_kill, sizeof(int), hipMemcpyHostToDevice, 
                 stream_dataIn, "Writing the kill command to SuperKernel");

  DisposeQueue(d_newJobs);

  DisposeQueue(d_finishedJobs);

  hipStreamDestroy(stream_kernel);
  hipStreamDestroy(stream_dataIn);
  hipStreamDestroy(stream_dataOut);

  pthread_mutex_destroy(&memcpyLock);
  pthread_mutex_destroy(&enqueueLock);
  pthread_mutex_destroy(&dequeueLock);
  pthread_mutex_destroy(&memoryListLock);
}

extern "C"
void gemtcPush(int taskType, int threads, int ID, void *d_parameters){
  //Enqueue the given task to the device
  //Returns as soon as the task is in Device Memory
  JobPointer h_JobDescription = (JobPointer) malloc(sizeof(JobDescription));
  h_JobDescription->JobType = taskType;
  h_JobDescription->numThreads = threads;
  h_JobDescription->params = d_parameters;
  h_JobDescription->JobID = ID;

  pthread_mutex_lock(&enqueueLock);  //Start Critical Section

  EnqueueJob(h_JobDescription, d_newJobs);

  pthread_mutex_unlock(&enqueueLock); //End Critical Section
}

struct ResultPair{int ID; void *params;};
extern "C"
void *gemtcPoll(){
  //Returns a pair with the ID and param pointer of the first job in the queue
  //If the queue is empty, this returns a NULL
  JobPointer h_JobDescription = (JobPointer) malloc(sizeof(JobDescription));
  pthread_mutex_lock(&dequeueLock);  //Start Critical Section
  h_JobDescription = MaybeFandD(d_finishedJobs);//returns null if empty
  pthread_mutex_unlock(&dequeueLock); //End Critical Section
  if(h_JobDescription==NULL){
    free(h_JobDescription);
    return NULL;
  }
  ResultPair *ret = (ResultPair *) malloc(sizeof(ResultPair));

  ret->ID = h_JobDescription->JobID;
  ret->params = h_JobDescription->params;

  free(h_JobDescription);
  return (void *)ret;
}

extern "C"
void gemtcMemcpyHostToDevice(void *device, void *host, int size){
  cudaSafeMemcpy(device, host, size, 
                 hipMemcpyHostToDevice, stream_dataIn, 
                 "HostToDevice API call");
}

extern "C"
void gemtcMemcpyDeviceToHost(void *host, void *device, int size){
  cudaSafeMemcpy(host, device, size, 
                 hipMemcpyDeviceToHost, stream_dataOut, 
                 "DeviceToHost API call");
}

extern "C"
void *gemtcGPUMalloc(int size){
  //This is defined in malloc/gemtcMalloc.cu
  return gemtcMalloc(size);
}

extern "C"
void gemtcGPUFree(void *p){
  //This is defined in malloc/gemtcMalloc.cu
  gemtcFree(p);
}
