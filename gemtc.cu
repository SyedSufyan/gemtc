#include <stdio.h>
#include <hip/hip_runtime.h>
#include <pthread.h>

hipStream_t stream_dataIn, stream_dataOut, stream_kernel;

pthread_mutex_t memcpyLock;
pthread_mutex_t enqueueLock;
pthread_mutex_t dequeueLock;

int *d_kill;

#include "DataMovement.cu"
#include "malloc/GemtcMalloc.cu"
#include "Queues/QueueJobs.cu"
#include "SuperKernel.cu"

Queue d_newJobs, d_finishedJobs;
/////////////////////
//Utility Functions//
/////////////////////
void *moveToCuda(void *val, int size){
  void *ret = gemtcMalloc(size);
  cudaSafeMemcpy(ret, val, size, 
                 hipMemcpyHostToDevice, stream_dataIn, 
                 "in moveToCuda of run()");
  return ret;
}
void *moveFromCuda(void *val, int size){
  void *ret = malloc(size);
  cudaSafeMemcpy(ret, val, size, 
                 hipMemcpyDeviceToHost, stream_dataOut, 
                 "in moveFromCuda of run()");
  return ret;
}

/////////////////
//API Functions//
/////////////////
extern "C"
void gemtcSetup(int QueueSize){
  pthread_mutex_init(&memcpyLock, NULL);
  pthread_mutex_init(&enqueueLock, NULL);
  pthread_mutex_init(&dequeueLock, NULL);

  int warp_size = 32;

  int warps = 8;
  int blocks = 14;

  dim3 threads(warp_size*warps, 1, 1);
  dim3 grid(blocks, 1, 1);

  hipStreamCreate(&stream_kernel);
  hipStreamCreate(&stream_dataIn);
  hipStreamCreate(&stream_dataOut);
  
  d_newJobs = CreateQueue(QueueSize);
  d_finishedJobs = CreateQueue(QueueSize);

  int temp = 0;
  d_kill = (int *) moveToCuda((void *)&temp, sizeof(int));

  hipDeviceSynchronize();

//Launch the super kernel
  superKernel<<< grid, threads, 0, stream_kernel>>>
             (d_newJobs, d_finishedJobs, d_kill);
}


extern "C"
void gemtcBlockingRun(int Type, int Threads, int ID, void *d_params){
  JobPointer h_JobDescription = (JobPointer) malloc(sizeof(JobDescription));
  h_JobDescription->JobType = Type;
  h_JobDescription->numThreads = Threads;
  h_JobDescription->params = d_params;
  h_JobDescription->JobID = ID;

  pthread_mutex_lock(&enqueueLock);  //Start Critical Section

  EnqueueJob(h_JobDescription, d_newJobs);

  pthread_mutex_unlock(&enqueueLock); //End Critical Section
  //  printf("Finished enqueue #%d\n", MyID);

  int first = 1;
  while(h_JobDescription->JobID!=ID || first){
    pthread_yield();
    pthread_mutex_lock(&dequeueLock);
    h_JobDescription = FrontResult(d_finishedJobs);
    if(h_JobDescription->JobID==ID)DequeueResult(d_finishedJobs);
    pthread_mutex_unlock(&dequeueLock);
    first = 0;
    printf("Current front:%d  MyID:%d\n", h_JobDescription->JobID, ID);
  }
  //printf("Recieved result on Job #%d\n", ID);
}

extern "C"
void gemtcCleanup(){
  int temp=1;
  cudaSafeMemcpy(&temp, d_kill, sizeof(int), hipMemcpyHostToDevice, 
                 stream_dataIn, "Writing the kill command to SuperKernel");

  DisposeQueue(d_newJobs);

  DisposeQueue(d_finishedJobs);

  hipStreamDestroy(stream_kernel);
  hipStreamDestroy(stream_dataIn);
  hipStreamDestroy(stream_dataOut);

  pthread_mutex_destroy(&memcpyLock);
  pthread_mutex_destroy(&enqueueLock);
  pthread_mutex_destroy(&dequeueLock);
}

extern "C"
void gemtcPush(int taskType, int threads, int ID, void *d_parameters){
  JobPointer h_JobDescription = (JobPointer) malloc(sizeof(JobDescription));
  h_JobDescription->JobType = taskType;
  h_JobDescription->numThreads = threads;
  h_JobDescription->params = d_parameters;
  h_JobDescription->JobID = ID;

  pthread_mutex_lock(&enqueueLock);  //Start Critical Section

  printf("start enqueue\n");

  EnqueueJob(h_JobDescription, d_newJobs);

  printf("end enqueue\n");

  pthread_mutex_unlock(&enqueueLock); //End Critical Section
}

struct ResultPair{int ID; void *params;};
extern "C"
void *gemtcPoll(){
  //Returns a pair with the ID and param pointer of the first job in the queue
  //If the queue is empty, this returns a NULL
  JobPointer h_JobDescription = (JobPointer) malloc(sizeof(JobDescription));

  pthread_mutex_lock(&dequeueLock);  //Start Critical Section

  h_JobDescription = MaybeFandD(d_finishedJobs);//returns null if empty

  pthread_mutex_unlock(&dequeueLock); //End Critical Section

  if(h_JobDescription==NULL){
    free(h_JobDescription);
    return NULL;
  }
  ResultPair *ret = (ResultPair *) malloc(sizeof(ResultPair));

  ret->ID = h_JobDescription->JobID;
  ret->params = h_JobDescription->params;

  free(h_JobDescription);
  return (void *)ret;
}

extern "C"
void gemtcMemcpyHostToDevice(void *device, void *host, int size){
  cudaSafeMemcpy(device, host, size, 
                 hipMemcpyHostToDevice, stream_dataIn, 
                 "HostToDevice API call");
}

extern "C"
void gemtcMemcpyDeviceToHost(void *host, void *device, int size){
  cudaSafeMemcpy(host, device, size, 
                 hipMemcpyDeviceToHost, stream_dataOut, 
                 "DeviceToHost API call");
}

extern "C"
void *gemtcGPUMalloc(int size){
  return gemtcMalloc(size);
}

extern "C"
void gemtcGPUFree(void *p){
  gemtcFree(p);
}
