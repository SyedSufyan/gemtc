#include <stdio.h>
#include <hip/hip_runtime.h>
#include <pthread.h>
#include <sys/time.h>
#include <unistd.h>
struct timeval tp;

hipStream_t stream_dataIn, stream_dataOut, stream_kernel;

pthread_mutex_t memcpyLock;
pthread_mutex_t enqueueLock;
pthread_mutex_t dequeueLock;

int *d_kill;

#include "DataMovement.cu"
#include "malloc/GemtcMalloc.cu"
#include "Queues/QueueJobs.cu"
#include "SuperKernel.cu"

Queue d_newJobs, d_finishedJobs;

//Jobs going in to the GPU
JobDescription *inBuffer;
int inSize;
int inMax;
double timeStamp;
//Jobs coming out of the GPU
JobDescription *outBuffer;
int outSize;
int outMax;

/*
This file contains the functions that make up the API to gemtc
They are:
*** Initialize/Deconstruct ***
  gemtcSetup()
  gemtcCleanup()

*** EnQueue/DeQueue Tasks  ***
  -- gemtcBlockingRun()   Not up to date
  gemtcPush()
  gemtcPoll()

*** Memory Transfer Calls  ***
  gemtcMemcpyHostToDevice()
  gemtcMemcpyDeviceToHost()

****Memory Management Calls***
  gemtcGPUMalloc()
  gemtcGPUFree()
 */


/////////////////////
//Utility Functions//
/////////////////////
void *moveToCuda(void *val, int size){
  void *ret = gemtcMalloc(size);
  cudaSafeMemcpy(ret, val, size, 
                 hipMemcpyHostToDevice, stream_dataIn, 
                 "in moveToCuda of run()");
  return ret;
}
void *moveFromCuda(void *val, int size){
  void *ret = malloc(size);
  cudaSafeMemcpy(ret, val, size, 
                 hipMemcpyDeviceToHost, stream_dataOut, 
                 "in moveFromCuda of run()");
  return ret;
}
double getTime_usec() {
    gettimeofday(&tp, NULL);
    return static_cast<double>(tp.tv_sec) * 1E6
            + static_cast<double>(tp.tv_usec);
}
void *bufferFlush(void *junk){
  while(1){
    pthread_mutex_lock(&enqueueLock);  //Start Critical Section

    double curTime = getTime_usec();
    if(curTime-timeStamp > 100 && inSize!=0){
      EnqueueJobBatch(inBuffer, inSize, d_newJobs);
      inSize=0;
    }

    pthread_mutex_unlock(&enqueueLock); //End Critical Section
    pthread_yield();  //wait for awhile before polling again
  }
}

/////////////////
//API Functions//
/////////////////
extern "C"
void gemtcSetup(int QueueSize, int Overfill){
//QueueSize determines the size of both of the queues in GPU memory
//  that hold pointers to task descriptions for new and finished tasks
//Overfill is a flag:
//  0  means  launch enough warps to have a one-to-one mapping with 
//            16 Cuda Core Groupings. High Efficiency
//  1  means  launch the maximum number of warps per SM
//            Highest Throughput
//            Extra warps will be "Hyperthreaded"

  //initialize locks
  pthread_mutex_init(&memcpyLock, NULL);
  pthread_mutex_init(&enqueueLock, NULL);
  pthread_mutex_init(&dequeueLock, NULL);
  pthread_mutex_init(&memoryListLock, NULL);

  inMax = 100;
  inSize = 0;
  inBuffer = (JobDescription *) malloc(inMax*sizeof(JobDescription));
  timeStamp = 0;

  outMax = 100;
  outSize = 0;
  outBuffer = (JobDescription *) malloc(outMax*sizeof(JobDescription));

  hipDeviceProp_t devProp;
  hipGetDeviceProperties(&devProp, 0); //default to first GPU

  int warp_size = devProp.warpSize;  //Always 32
  int warps;
  int blocks = devProp.multiProcessorCount;
  if(Overfill){
    warps = devProp.maxThreadsPerBlock/32;
  }else{
    int coresPerSM = _ConvertSMVer2Cores(devProp.major, devProp.minor);
    warps = coresPerSM/16;  //A warp runs on 16 cores
  }

  printf("Workers:  %d\n", warps*blocks);

  dim3 threads(warp_size*warps, 1, 1);
  dim3 grid(blocks, 1, 1);

  //Init Streams for the SuperKernel and various memory copies
  hipStreamCreate(&stream_kernel);
  hipStreamCreate(&stream_dataIn);
  hipStreamCreate(&stream_dataOut);
  
  //Initialize Device Memory with Queues
  d_newJobs = CreateQueue(QueueSize);
  d_finishedJobs = CreateQueue(QueueSize);

  //Initialize kill flag in device Memory
  int temp = 0;
  d_kill = (int *) moveToCuda((void *)&temp, sizeof(int));

  hipDeviceSynchronize();

  //Launch thread to dump outBuffer
  pthread_t bufferFlusher;
  pthread_create(&bufferFlusher, NULL, bufferFlush, NULL);

//Launch the super kernel
  superKernel<<< grid, threads, 8192, stream_kernel>>>  //8192 = 8kBytes of shared Memory
             (d_newJobs, d_finishedJobs, d_kill);
}


//This function is currently out of date
extern "C"
void gemtcBlockingRun(int Type, int Threads, int ID, void *d_params){
  //This funcyion will enqueue the given task to the device
  //Then block until it returns
  //   This is busy blocking where it polls the GPU to see if it finished
  JobPointer h_JobDescription = (JobPointer) malloc(sizeof(JobDescription));
  h_JobDescription->JobType = Type;
  h_JobDescription->numThreads = Threads;
  h_JobDescription->params = d_params;
  h_JobDescription->JobID = ID;

  pthread_mutex_lock(&enqueueLock);  //Start Critical Section

  EnqueueJob(h_JobDescription, d_newJobs);

  pthread_mutex_unlock(&enqueueLock); //End Critical Section

  int first = 1;
  while(h_JobDescription->JobID!=ID || first){
    //Loop until our task is at the front of the result queue
    // Non-ideal because the task could finish but take awhile to move
    // through the queue. Cannot fix this problem with current DataStruct
    pthread_yield();
    pthread_mutex_lock(&dequeueLock);
    h_JobDescription = FrontResult(d_finishedJobs);
    if(h_JobDescription->JobID==ID)DequeueResult(d_finishedJobs);
    pthread_mutex_unlock(&dequeueLock);
    first = 0;
    printf("Current front:%d  MyID:%d\n", h_JobDescription->JobID, ID);
  }
  //printf("Recieved result on Job #%d\n", ID);
}

extern "C"
void gemtcCleanup(){
  int temp=1;
  cudaSafeMemcpy(d_kill, &temp, sizeof(int), hipMemcpyHostToDevice, 
                 stream_dataIn, "Writing the kill command to SuperKernel");

  //Wait for SuperKernel to die
  hipEvent_t Super_Kernel_Finished;
  hipEventCreate(&Super_Kernel_Finished);
  hipEventRecord(Super_Kernel_Finished, stream_kernel);
  hipEventSynchronize(Super_Kernel_Finished);

  hipError_t e = hipGetLastError();

  DisposeQueue(d_newJobs);

  DisposeQueue(d_finishedJobs);

  hipStreamDestroy(stream_kernel);
  hipStreamDestroy(stream_dataIn);
  hipStreamDestroy(stream_dataOut);

  pthread_mutex_destroy(&memcpyLock);
  pthread_mutex_destroy(&enqueueLock);
  pthread_mutex_destroy(&dequeueLock);
  pthread_mutex_destroy(&memoryListLock);
}

extern "C"
void gemtcPush(int taskType, int threads, int ID, void *d_parameters){
  pthread_mutex_lock(&enqueueLock);  //Start Critical Section

  inBuffer[inSize].JobType = taskType;
  inBuffer[inSize].numThreads = threads;
  inBuffer[inSize].params = d_parameters;
  inBuffer[inSize].JobID = ID;
  inSize++;

  timeStamp = getTime_usec();

  if(inSize==inMax){
    EnqueueJobBatch(inBuffer, inSize, d_newJobs);
    inSize=0;
  }
  pthread_mutex_unlock(&enqueueLock); //End Critical Section
}

extern "C"
void gemtcPoll(int *ID, void **params){
  pthread_mutex_lock(&dequeueLock);  //Start Critical Section

  if(outSize==0){
    outSize = FrontAndDequeueBatch(outBuffer, outMax, d_finishedJobs);//returns null if empty
  }
  if(outSize==0){
    *ID=-1;
    *params=NULL;
    pthread_mutex_unlock(&dequeueLock); //End Critical Section
    return;
  }else{
    outSize--;
    *ID = outBuffer[outSize].JobID;
    *params = outBuffer[outSize].params;
  }
  pthread_mutex_unlock(&dequeueLock); //End Critical Section
}

extern "C"
void gemtcMemcpyHostToDevice(void *device, void *host, int size){
  cudaSafeMemcpy(device, host, size, 
                 hipMemcpyHostToDevice, stream_dataIn, 
                 "HostToDevice API call");
}

extern "C"
void gemtcMemcpyDeviceToHost(void *host, void *device, int size){
  cudaSafeMemcpy(host, device, size, 
                 hipMemcpyDeviceToHost, stream_dataOut, 
                 "DeviceToHost API call");
}

extern "C"
void *gemtcGPUMalloc(int size){
  //This is defined in malloc/gemtcMalloc.cu
  return gemtcMalloc(size);
}

extern "C"
void gemtcGPUFree(void *p){
  //This is defined in malloc/gemtcMalloc.cu
  gemtcFree(p);
}
