#include "hip/hip_runtime.h"
/*
 * Benchmarking code for Imogen's ported kernel from "gpuImogen/gpuclass/cudaArrayAtomic.cu" 
 */
#include "../../../gemtc.cu"
#include<stdio.h>
#include<stdlib.h>
#include<time.h>

/*
 * main engine
 * Input data is generated within by random number genetation
 */
int main(int argc, char **argv){

    /*
     * NUM_TASKS is the total number of tasks which will be submitted to GeMTC
     * LOOP_SIZE is the total number of tasks submitted to GeMTC queue before watiting for polling for results
     * ARRAY_SIZE is the length of array
     */
    int NUM_TASKS, LOOP_SIZE, ARRAY_SIZE;

    /*
     * Timers for benchmarking
     */
    struct timespec start, end;
    double time_spent =0.0;

    /*
     * Report log for invalid number of arguments
     */
    if (argc > 3) {
        NUM_TASKS = atoi(argv[1]);
	LOOP_SIZE = atoi(argv[2]);
	ARRAY_SIZE = atoi(argv[3]);
    } else {
        printf("This test requires three parameters:\n");
        printf("int NUM_TASKS, int LOOP_SIZE, int ARRAY_SIZE\n");
        printf("where\n");
        printf("NUM_TASKS is the total number ArrayAtomic tasks to be sent to gemtc\n");
        printf("LOOP_SIZE is the number of tasks should be sent to gemtc before waiting for results\n");
        printf("ARRAY_SIZE is the side length of the matrix that is going to be squared\n");
        exit(1);
    }

    /*
     * Setup gemtc
     */
    gemtcSetup(25600, 0);

    /*
     * Local variables
     */
    int j;

    /*
     * (3 + ARRAY_SIZE) because 1st parameter will select sub-kernel and 2nd will send the threahold
     * 3rd parameter will be size of the arry, lastly the array itself
     */
    double *h_params = (double *) malloc(sizeof(double)*(3 + ARRAY_SIZE));
    memset(h_params, 0, sizeof(double)*(3+ARRAY_SIZE));

    /*
     * Select first sub-kernel
     */
    h_params[0] = 1;

    /*
     * Minimum threshold
     */
    h_params[1] = 0.053;

    /*
     * Array size
     */
    h_params[2] = ARRAY_SIZE;

    /*
     * Generate array of random floating point numbers
     */
    for (j = 3; j < ARRAY_SIZE + 3; j++) {
        h_params[j]= ((double) rand())/INT_MAX;
    }

    /*
     * Purposefully set 1 array parameters to be lessa then minimum
     */
    h_params[5] = .009;

    //printf("Minimum Threshold %f\n", h_params[1]);
    //printf("ORIGINAL ARRAY \n");
    //for(j=3; j<ARRAY_SIZE+3; j++) {
    //  printf("Element %f\n", h_params[j]);
    //}
    printf("\n");

    double *d_params = NULL;
    /*
     * Start benchmarking timer
     */
    clock_gettime(CLOCK_MONOTONIC_RAW, &start);

    for (j = 0; j < NUM_TASKS/LOOP_SIZE; j++) {
        int i;
        h_params[0] = 1;
        h_params[1] = 0.053;
        h_params[2] = ARRAY_SIZE;
        h_params[5] = .009;

        /*
	 * Submit number of jobs = LOOP_SIZE to gemtc
	 */
        for (i = 0; i < LOOP_SIZE; i++) {
            /*
	     * Allocate memory
	     */
            d_params = (double *) gemtcGPUMalloc(sizeof(double)*(3+ARRAY_SIZE));

	    if (!d_params) {
                printf("GeMTC memory allocation failure\n");
	        return 0;
           } else {
              //printf("Submitting j=%d, i=%d, taskID %d\n", j, i, (i+j*LOOP_SIZE));
           }

           /*
            * Copy package to gemtc
            */
           gemtcMemcpyHostToDevice(d_params, h_params, sizeof(double)*(3+ARRAY_SIZE));

           /*
            * 24 is the ArrayAtomic kernel
            */
           gemtcPush(24, 32, i+j*LOOP_SIZE, d_params);
           //gemtcGPUFree(d_params);
        }

        /*
         * Poll for results
         */
        for (i = 0; i < LOOP_SIZE; i++) {
            void *ret=NULL;
            int id;
          
            /*
             * Poll
             */
            while (ret == NULL) {
               gemtcPoll(&id, &ret);
            }
            //printf("Received job %d\n", id);
            /*
             * Copy the results back into host
             */
            gemtcMemcpyDeviceToHost(h_params, 
                                    (double*)ret, 
                                    sizeof(double)*(3+ARRAY_SIZE));
            /*
             * Free gemtc memory
             */
            gemtcGPUFree(ret);
            ret = NULL;
        } 
    }
    /*
     * Stop timer
     */
    clock_gettime(CLOCK_MONOTONIC_RAW, &end);
    /* 
     * Evaulate time taken for the computation
     */
    time_spent = (end.tv_sec - start.tv_sec) +  (end.tv_nsec - start.tv_nsec)/1e9;

    printf(" Time taken %f seconds\n", time_spent);
    printf("\n");
    //printf("NORMALIZED ARRAY \n");
    //for(j=3; j<ARRAY_SIZE+3; j++) {
    //  printf("Element %f\n", h_params[j]);
    //}
    printf("\n");

    /*
     * Cleanup
     */
    gemtcCleanup();
    free(h_params);
    return 0;
}
