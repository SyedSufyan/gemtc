#include "hip/hip_runtime.h"
/*
 * Benchmarking code for Imogen's ported cukern_TVDStep_hydro_uniform() kernel from "gpuImogen/gpuclass/cudaFluidTVD.cu" 
 */
#include "../../../gemtc.cu"
#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#include<math.h>

/*
 * main engine
 */
int main(int argc, char **argv){
    /*
     * NUM_TASKS is the total number of tasks which will be submitted to GeMTC
     * LOOP_SIZE is the total number of tasks submitted to GeMTC queue before watiting for polling for results
     */
    int NUM_TASKS, LOOP_SIZE;

    /*
     * Timers for benchmarking
     */
    struct timespec start, end;
    double time_spent = 0.0;
    /*
     * Data-set dimensions
     */
    int nx, ny, nz;

    /*
     * Report log for invalid number of arguments
     */
    if (argc > 2){
        NUM_TASKS = atoi(argv[1]);
        LOOP_SIZE = atoi(argv[2]);
    } else {
        printf("This test requires three parameters:\n");
        printf("int NUM_TASKS, int LOOP_SIZE, int ARRAY_SIZE\n");
        printf("where\n");
        printf("NUM_TASKS is the total number ArrayAtomic tasks to be sent to gemtc\n");
        printf("LOOP_SIZE is the number of tasks should be sent to gemtc before waiting for results\n");
        exit(1);
    }

    //printf("DEBUG LOG 1\n");
    /*
     * Setup gemtc
     */
    gemtcSetup(25600, 0);

    int i, j;
    int size, output_start, output_len;

    /*
     * Set size, output_start and output_len parameters
     * There is a sinle chunk of memory we allocate for all parameters to be passed
     * to and fro from GeMTC for this kernel. All the inputs and output equation
     * parameters are offset in this big chunk. 
     * While copying the result back we don't want to copy the whole chunk back 
     * instead just copy the output equation varaibles, Hence have to calculate 
     * offset for output parameters and their length.
     * There are 6 double variables, 11 cubical data variables and 1 variable of 2 dimention.
     */
    size = 6*sizeof(double) + (410*280*1)*11*sizeof(double) + (280*1)*sizeof(double);
    output_start = 3 + (410*280*1)*6;
    output_len = (410*280*1)*5*sizeof(double);

    //printf("DEBUG LOG 2\n");
    /*
     * Allocate host memory
     */ 
    double *h_params = (double *) malloc(size);
    memset(h_params, 0, size);

    /*
     * Set nx, ny, nz
     */
    nx = h_params[0] = 410.0;
    ny = h_params[1] = 280.0;
    nz = h_params[2] = 1.0;

    //printf("DEBUG LOG 3\n");

    /*
     * Set input variables
     */
    double* rho = (h_params + 3);
    double* E = rho + (nx*ny*nz);
    double* px = E + (nx*ny*nz);
    double* py = px + (nx*ny*nz);
    double* pz = py + (nx*ny*nz);
    double* P = pz + (nx*ny*nz);
    double* rho_out = P + (nx*ny*nz);
    double* E_out = rho_out + (nx*ny*nz);
    double* px_out = E_out + (nx*ny*nz);
    double* py_out = px_out + (nx*ny*nz);
    double* pz_out = py_out + (nx*ny*nz);
    double* Cfreeze = pz_out + (nx*ny*nz);
    double* lambda = Cfreeze + ny*nz;
    double* rhomin = lambda + 1;
    double* gamma = rhomin + 1;

    *lambda = 0.1;
    *gamma = 5/3;
    *rhomin = 1e-5;

    /*
     * Read data from files
     */
    FILE *f_rho = fopen("../data/rho.txt", "r");
    if(!f_rho) {
        printf("Unable to open ../data/rho.txt\n");
    }
    FILE *f_E = fopen("../data/E.txt", "r");
    if(!f_E) {
        printf("Unable to ../data/open E.txt\n");
    }
    FILE *f_px = fopen("../data/px.txt", "r");
    if(!f_px) {
        printf("Unable to ../data/open px.txt\n");
    }
    FILE *f_py = fopen("../data/py.txt", "r");
    if(!f_py) {
        printf("Unable to open ../data/py.txt\n");
    }
    FILE *f_pz = fopen("../data/pz.txt", "r");
    if(!f_pz) {
        printf("Unable to open ../data/pz.txt\n");
    }

    for(i = 0; i < 410; i++) {
        for(j = 0; j < 280; j++) {
            fscanf(f_rho, "%lf", &rho[i*280 + j]);
            fscanf(f_E, "%lf", &E[i*280 + j]);
            fscanf(f_px, "%lf", &px[i*280 + j]);
            fscanf(f_py, "%lf", &py[i*280 + j]);
            fscanf(f_pz, "%lf", &pz[i*280 + j]);
        }
        //printf("Integer %d\n", i);
    }
 
    /*
     * Close files
     */ 
    fclose(f_rho); 
    fclose(f_E); 
    fclose(f_px); 
    fclose(f_py); 
    fclose(f_pz); 
    
    /*
     * Always print bytes without it we can't analyse benchmark properly
     */
    printf("SiZE %d\n", size);

    /*
     * GeMTC data pointer
     */
    double *d_params = NULL;
    //double *d_params = (double *) gemtcGPUMalloc(size);
    /*
     * Start benchmarking timer
     */
    clock_gettime(CLOCK_MONOTONIC_RAW, &start);
    for (j = 0; j < NUM_TASKS/LOOP_SIZE; j++) {
        /*
         * memset the equation output parameters
         */
        memset((double*)h_params + output_start, 0, output_len);
        //printf(" j is %d\n",j);

        /*
	 * Submit number of jobs = LOOP_SIZE to gemtc
	 */
        for(i=0; i<LOOP_SIZE; i++){
            /*
	     * Allocate memory
	     */
            d_params = (double *) gemtcGPUMalloc(size);
            if (d_params == NULL) {
                printf("Unable to allocate memory\n");
            }

            /*
             * Copy package to gemtc
             */
            gemtcMemcpyHostToDevice(d_params, h_params, size);
            /*
             * 29 is the FluidTVD kernel
             */
            gemtcPush(29, 32, i+j*LOOP_SIZE, d_params);
            //printf("Pushing %d\n", i);
            //gemtcGPUFree(d_params);
        }
        //printf("DEBUG LOG 6\n");

        /*
         * Poll for results
         */
        for (i = 0; i < LOOP_SIZE; i++) {
            void *ret=NULL;
            int id;
            /*
             * Poll
             */
            while (ret == NULL){
            //printf("POLLING\n");
                gemtcPoll(&id, &ret);
            }
            /*
             * Copy the results back into host
             */
            gemtcMemcpyDeviceToHost(((double*)h_params + output_start), 
                                    ((double*)ret + output_start), 
                                    output_len);
            //printf("DEBUG LOG 7\n");
            /*
             * Free gemtc memory
             */
            gemtcGPUFree(ret);
            //printf("DEBUG LOG 8\n");
            ret = NULL;
        }
    }

    //printf("DEBUG LOG 9\n");
    /*
     * Stop timer
     */
    clock_gettime(CLOCK_MONOTONIC_RAW, &end);

    /* 
     * Evaulate time taken for the computation
     */
    time_spent = (end.tv_sec - start.tv_sec) +  (end.tv_nsec - start.tv_nsec)/1e9;

    printf(" Time taken %f seconds\n", time_spent);
    printf("\n");
    /*
     * Cleanup
     */
    gemtcCleanup();

    free(h_params);
    return 0;
}
