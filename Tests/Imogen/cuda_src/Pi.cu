#include "hip/hip_runtime.h"
/*
 * Calculating value of Pi using GeMTC kernel
 */
#include "../../../gemtc.cu"
#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#include<math.h>
#define SEED 35791246

int main(int argc, char **argv){
    /*
     * NUM_TASKS is the total number of tasks which will be submitted to GeMTC
     * LOOP_SIZE is the total number of tasks submitted to GeMTC queue before watiting for polling for results
     */
     int NUM_TASKS, LOOP_SIZE;
     int niter=0, i ,j;

    /*
     * Timers for benchmarking
     */
     struct timespec start, end;
     double time_spent = 0.0;

    /*
     * Report log for invalid number of arguments
     */
    if (argc > 2){
        NUM_TASKS = atoi(argv[1]);
        LOOP_SIZE = atoi(argv[2]);
        niter = atoi(argv[3]);
    } else {
         printf("This test requires three parameter:\n");
         printf("int NUM_TASKS, int LOOP_SIZE, int ITERATIONS\n");
         printf("int iterations\n");
         printf("where  NUM_TASKS is the total numer of task\n");
         printf("LOOP_SIZE is the number of tasks should be sent to gemtc before waiting for results\n");
         printf("ITERATIONS is the total numer of task\n");
         exit(1);
    }

    /*
     * Setup GeMTC
     */
    gemtcSetup(25600, 0);

    /*
     * device data pointer
     */
    double *d_params = NULL;

    /*
     * 2*niter random numbers will be generated and we will pass 
     * 2 double variables, 1 for how many iterations and 1 for getting back the
     * value of PI
     */
    int size = (2 + niter*2);

    /*
     * initialize seed for random number generation
     */
    srand(SEED);
  
    /*
     * Allocate host memory
     */ 
    double *h_params = (double*)malloc(sizeof(double)*size);

    /*
     * Geneate random float numbers
     */
    for(i=2; i<size; i++) {
        h_params[i] = (double)rand()/RAND_MAX;
    }

    /*
     * Set number of iterations
     */
    h_params[0] = niter;

    /*
     * Start timer
     */
    clock_gettime(CLOCK_MONOTONIC_RAW, &start);

    for (j = 0; j < NUM_TASKS/LOOP_SIZE; j++) {
        /*
	 * Submit number of jobs = LOOP_SIZE to gemtc
	 */
        for (i = 0; i < LOOP_SIZE; i++) {
            /*
	     * Allocate memory
	     */
            d_params = (double *) gemtcGPUMalloc(size*sizeof(double));
            /*
             * Copy package to gemtc
             */
             gemtcMemcpyHostToDevice(d_params, h_params, sizeof(double)*size);
             gemtcPush(30, 32, i+j*LOOP_SIZE, d_params);
        }
        /*
         * Poll for results
         */
        for(i=0; i<LOOP_SIZE; i++) {
            void *ret=NULL;
            int id;
          
            /*
             * Poll
             */
            while (ret == NULL) {
                gemtcPoll(&id, &ret);
            }
            /*
             * Copy the results back into host
             */
            gemtcMemcpyDeviceToHost(((double*)h_params + 1), 
                                    ((double*)ret + 1), 
                                    sizeof(double));
            /*
             * Free gemtc memory
             */
            gemtcGPUFree(ret);
        }
    }
    /*
     * Stop timer
     */
    clock_gettime(CLOCK_MONOTONIC_RAW, &end);
    /* 
     * Evaulate time taken for the computation
     */
    time_spent = (end.tv_sec - start.tv_sec) +  (end.tv_nsec - start.tv_nsec)/1e9;

    printf("Estimate of pi is %f \n", h_params[1]);
    printf("Time taken %f seconds\n", time_spent);
    /*
     * Cleanup
     */
    gemtcCleanup();
    free(h_params);
    return 0;
}
