#include "hip/hip_runtime.h"
/*
 * Benchmarking code for Imogen's ported cukern_FreezeSpeed_hydro() kernel from "gpuImogen/gpuclass/freezeAndPtot.cu" 
 */
#include "../../../gemtc.cu"
#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#include<math.h>

/*
 * main engine
 * Input data is generated according to "gpuImogen/gpuclass/unitTest.m"
 */
int main(int argc, char **argv){
    /*
     * NUM_TASKS is the total number of tasks which will be submitted to GeMTC
     * LOOP_SIZE is the total number of tasks submitted to GeMTC queue before watiting for polling for results
     * SIMULATION_TYPE is the data-set to choose for benchmarking
     */
    int NUM_TASKS, LOOP_SIZE, SIMULATION_TYPE;

    /*
     * Timers for benchmarking
     */
    struct timespec start, end;
    double time_spent = 0.0;

    /*
     * Data-set dimensions
     */
    int nx, ny, nz;

    /*
     * Report log for invalid number of arguments
     */
    if(argc>3){
        NUM_TASKS = atoi(argv[1]);
        LOOP_SIZE = atoi(argv[2]);
        SIMULATION_TYPE = atoi(argv[3]);
    } else {
        printf("This test requires three parameters:\n");
        printf("int NUM_TASKS, int LOOP_SIZE, int ARRAY_SIZE\n");
        printf("where\n");
        printf("NUM_TASKS is the total number ArrayAtomic tasks to be sent to gemtc\n");
        printf("LOOP_SIZE is the number of tasks should be sent to gemtc before waiting for results\n");
        printf("SIMULATION_TYPE 1 for 12*12*12 size simulation, 2 for 128*124*119 size simulation; 3 for 333*69*100 size simulation\n");
        exit(1);
    }

    //printf("DEBUG LOG 1\n");
    /*
     * Setup gemtc
     */
    gemtcSetup(25600, 0);

    int i, j, k;
    int size, output_start, output_len;

    /*
     * Set size, output_start and output_len parameters
     * There is a sinle chunk of memory we allocate for all parameters to be passed
     * to and fro from GeMTC for this kernel. All the inputs and output equation
     * parameters are offset in this big chunk. 
     * While copying the result back we don't want to copy the whole chunk back 
     * instead just copy the output equation varaibles, Hence have to calculate 
     * offset for output parameters and their length.
     * There are 5 double variables, 6 cubical data variables and 1 variable of 2 dimention.
     */
    switch(SIMULATION_TYPE) {
        case 1:
            size = 5*sizeof(double) + (12*12*12)*6*sizeof(double) + (12*12)*sizeof(double);
            output_start = 5 + (12*12*12)*5;
            output_len = (12*12*12)*sizeof(double) + (12*12)*sizeof(double);
            break;
        case 2:
            size = 5*sizeof(double) + (128*124*119)*6*sizeof(double)  + (124*119)*sizeof(double);
            output_start = 5 + (128*124*119)*5;
            output_len = (128*124*119)*sizeof(double) + (124*119)*sizeof(double);
            break;
        case 3:
            size = 5*sizeof(double) + (333*69*100)*6*sizeof(double) +  (69*100)*sizeof(double);
            output_start = 5 + (333*69*100)*5;
            output_len = (333*69*100)*sizeof(double) + (69*100)*sizeof(double);
            break;
        default:
            size = 5*sizeof(double) + (12*12*12)*6*sizeof(double) + (12*12)*sizeof(double);
            output_start = 5 + (12*12*12)*5;
            output_len = (12*12*12)*sizeof(double) + (12*12)*sizeof(double);
            break;
    }

    //printf("DEBUG LOG 2\n");
    /*
     * Allocate host memory
     */ 
    double *h_params = (double *) malloc(size);
    memset(h_params, 0, size);

    /*
     * Set nx, ny, nz
     */
    switch(SIMULATION_TYPE) {
        case 1:
            nx = h_params[0] = 12.0;
            ny = h_params[1] = 12.0;
            nz = h_params[2] = 12.0;
            break;
        case 2:
            nx = h_params[0] = 128.0;
            ny = h_params[1] = 124.0;
            nz = h_params[2] = 119.0;
            break;
        case 3:
            nx = h_params[0] = 333.0;
            ny = h_params[1] = 69.0;
            nz = h_params[2] = 100.0;
            break;
        default:
            nx = h_params[0] = 12.0;
            ny = h_params[1] = 12.0;
            nz = h_params[2] = 12.0;
            break;
    }
    //printf("DEBUG LOG 3\n");

    /*
     * ngrid implementation of Matlab
     */
    double* xpos = (double*) malloc(nx*ny*nz*sizeof(double));
    double* ypos = (double*) malloc(nx*ny*nz*sizeof(double));
    double* zpos = (double*) malloc(nx*ny*nz*sizeof(double));
    
    /*
     * Set equation parameter to point to appropriate offset.
     */
    double* rho = (h_params + 3);

    for (i = 0; i < nx; i++) {
        for (j = 0; j < ny; j++) {
            for (k = 0; k < nz; k++) {
                xpos[i*ny*nz + j*nz + k] = (i+1)*2*M_PI/(nx);
                ypos[i*ny*nz + j*nz + k] = (j+1)*2*M_PI/(ny);
                zpos[i*ny*nz + j*nz + k] = (k+1)*2*M_PI/(nz);
                rho[i*ny*nz + j*nz + k] = 1.0;
            }
        }
    }
    //printf("DEBUG LOG 4\n");

    /*
     * Set equation parameter to point to appropriate offset.
     */
    double *E = rho + (nx*ny*nz);
    double *px = E + (nx*ny*nz);
    double *py = px + (nx*ny*nz);
    double *pz = py + (nx*ny*nz);
    double *ptot = pz + (nx*ny*nz);

    /*
     * Set px, py, E, ptot
     */
    for (i = 0; i < nx; i++) {
        for (j = 0; j < ny; j++) {
            for (k = 0; k < nz; k++) {
                px[i*ny*nz + j*nz + k] = sin(xpos[i*ny*nz + j*nz + k]);
                py[i*ny*nz + j*nz + k] = 1 + sin(ypos[i*ny*nz + j*nz + k] + zpos[i*ny*nz + j*nz + k]);
                pz[i*ny*nz + j*nz + k] = cos(zpos[i*ny*nz + j*nz + k]);

                E[i*ny*nz + j*nz + k] = .5 * (px[i*ny*nz + j*nz + k]*px[i*ny*nz + j*nz + k] + py[i*ny*nz + j*nz + k]*py[i*ny*nz + j*nz + k] + pz[i*ny*nz + j*nz + k]*pz[i*ny*nz + j*nz + k])/rho[i*ny*nz + j*nz + k] + 2;

                ptot[i*ny*nz + j*nz + k] = (2/3)*(E[i*ny*nz + j*nz + k] - .5*(px[i*ny*nz + j*nz + k]*px[i*ny*nz + j*nz + k] + py[i*ny*nz + j*nz + k]*py[i*ny*nz + j*nz + k] + pz[i*ny*nz + j*nz + k]*pz[i*ny*nz + j*nz + k])/rho[i*ny*nz + j*nz + k]);
            }
        }
    }

    /*
     * Set values of remaining equation parameters
     */
    double* gamma = ptot + (nx*ny*nz);
    *gamma = 5/3;

    double* cs0 = gamma + 1;
    double var = 1e-5;
    *cs0 = sqrt((5/3)*pow(var, 2/3));

    //printf("SiZE %d\n", rhomin - h_params);
    /*
     * Always print bytes without it we can't analyse benchmark properly
     */
    printf("SiZE %d\n", size);

    /*
     * GeMTC data pointer
     */
    double *d_params = NULL;
    //double *d_params = (double *) gemtcGPUMalloc(size);

    /*
     * Start benchmarking timer
     */
    clock_gettime(CLOCK_MONOTONIC_RAW, &start);
    for (j = 0; j < NUM_TASKS/LOOP_SIZE; j++) {
        /*
         * memset the equation output parameters
         */
        memset((double*)h_params + output_start, 0, output_len);

        /*
	 * Submit number of jobs = LOOP_SIZE to gemtc
	 */
        for (i = 0; i < LOOP_SIZE; i++){
            /*
	     * Allocate memory
	     */
            d_params = (double *) gemtcGPUMalloc(size);
            if (d_params == NULL) {
                printf("Unable to allocate memory\n");
            }
            /*
             * Copy package to gemtc
             */
            gemtcMemcpyHostToDevice(d_params, h_params, size);

            /*
             * 28 is the freezeAndptot kernel
             */
            gemtcPush(28, 32, i+j*LOOP_SIZE, d_params);
            //gemtcGPUFree(d_params);
        }
        //printf("DEBUG LOG 6\n");

        /*
         * Poll for results
         */
        for (i = 0; i < LOOP_SIZE; i++) {
            void *ret=NULL;
            int id;
            /*
             * Poll
             */
            while (ret == NULL){
            //printf("POLLING\n");
                gemtcPoll(&id, &ret);
            }
            //gemtcMemcpyDeviceToHost(h_params + output_start, 
            //                        (double*)ret + output_start, 
            //                        output_len);
            //printf("DEBUG LOG 7\n");
            /*
             * Free gemtc memory
             */
            gemtcGPUFree(ret);
            //printf("DEBUG LOG 8\n");
            ret = NULL;
        }
        //printf("Done\n");
    }

    //printf("DEBUG LOG 9\n");
    /*
     * Stop timer
     */
    clock_gettime(CLOCK_MONOTONIC_RAW, &end);
    /* 
     * Evaulate time taken for the computation
     */
    time_spent = (end.tv_sec - start.tv_sec) +  (end.tv_nsec - start.tv_nsec)/1e9;

    printf(" Time taken %f seconds\n", time_spent);
    printf("\n");
    /*
     * Cleanup
     */
    gemtcCleanup();

    free(h_params);
    free(xpos);
    free(ypos);
    free(zpos);
    return 0;
}
