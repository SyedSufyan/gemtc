#include "hip/hip_runtime.h"
#include "../../gemtc.cu"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define MAX_WORKERS 32 
int pushJobs(int num_tasks, void *h_params, void *offset_pointer, int mem_needed, int microkernel);
void pullJobs(int kernel_calls); 
double cpu_time();

int main(int argc, char **argv){
  gemtcSetup(100000,0);

  const int np = 50; //Modify this variable.
  const int nd = 2; //This value should only be 2 or 3!
  const int step_num = 1; 
  const int seed = 123456789;
  const double mass = 1.0;
  const double dt = 0.0001;

  int a_size = np*nd;
  int a_mem = sizeof(double) * a_size; 

  double darray[a_size];
 
  int i; 
  for(i=0; i<a_size; i++){
    darray[i] = 0.0; 
  }
 
  //Setup the Table we will constantly reference.

  //Table | np | nd | mass | pos[] | vel[] | acc[] |  f[]  | pe[]  |  ke[]  |
  //Bytes | 4  | 4  |   8  | a_mem | a_mem | a_mem | a_mem | a_mem | a_mem  |

  int mem_needed = sizeof(int) * 2 + sizeof(double) + a_mem*6; 
  void *d_table = gemtcGPUMalloc(mem_needed); 
  void *h_table = malloc(mem_needed); 

  memcpy( h_table                 , &np   , sizeof(int));  
  memcpy( (((int*)h_table)+1)     , &nd   , sizeof(int)); 
  memcpy( (((double*)h_table)+1)  , &mass , sizeof(double));

  for(i=0; i<6; i++){
    memcpy( (((double*)h_table) + a_size*i + 2), darray, a_mem); 
  }
  //Copy Table onto Device Memory
  gemtcMemcpyHostToDevice(d_table, h_table, mem_needed);
  
  /////////////// Initialize ////////////////
  
  int init_mem_needed = sizeof(double) + sizeof(double)*nd + sizeof(int); 
  void *h_init_params = malloc(init_mem_needed);

  //Init Params  | &Table |  box[]  | seed |
  //Bytes        |   8    | 8 * nd  |  4   | 

  double box[nd];
  for(i=0; i<nd; i++){
    box[i] = 10.0;
  }

  memcpy( h_init_params                              , &d_table , sizeof(void*));
  memcpy( ((double*)h_init_params)+1                 ,  box     , nd*sizeof(double));
  memcpy( (int*)(((double*)h_init_params)+1+nd)      ,  &seed   , sizeof(int));
   
  void *d_init_params = gemtcGPUMalloc(init_mem_needed);
  gemtcMemcpyHostToDevice(d_init_params, h_init_params, init_mem_needed);

  /*The Init Kernel cannot be parallelized if we want to have same
    results as the .cpp version. As a result, we only call it on 1
    Microkernel */

  gemtcPush(17, 32, 1000, d_init_params); 
  pullJobs(1);   

  /////////////// Compute/Update Loop /////////////////
  printf("\nComputing inital forces and energies.\n");
  
  double e0, ctime1, ctime2;
  int j; 
  int print_step = step_num / 10;
  if(print_step == 0){ print_step++;}; 

  printf("Step\tP Energy\tK Energy\t(P+K-E0)/E0\n");
  for(j=0; j<=step_num; j++){

    //Compute Params  | &Table | offset | 
    //Bytes           |   8    |   4    | 

    int comp_mem_needed = sizeof(void*) + sizeof(int); 
    void *h_comp_params = malloc(comp_mem_needed);

    memcpy(h_comp_params, &d_table, sizeof(void*));

    void *comp_offset_pointer = ((double*)h_comp_params) + 1; 
   
    //Push the Jobs 
    int k_calls = pushJobs(np, h_comp_params, comp_offset_pointer, comp_mem_needed, 16);
    pullJobs(k_calls);   
    
    void *comp_table = malloc(mem_needed);

    //Get the Values from the Data Table. 
    gemtcMemcpyDeviceToHost(comp_table, d_table, mem_needed);

    double *pe = ((double*)comp_table) + 2 + 4 * a_size;
    double *ke = pe + a_size;

    for(i=0; i<a_size; i++){
      printf("%d: %f %f\n", i, pe[i], ke[i]);
    }
    
    double psum = 0.0;
    double ksum = 0.0; 
    
    for(i=0; i < a_size; i++){
      psum += pe[i];
      ksum += ke[i];
    }
   
    if(j == 0){
      e0 = psum + ksum; 
      printf("%d\t%.2f\t\t%.4f\t\t%f\n", j, psum, ksum, (psum+ksum-e0)/e0);
      ctime1 = cpu_time();
      continue;
    }

    if( j % print_step == 0){
      printf("%d\t%.2f\t\t%f\t\t%f\n", j, psum, ksum, (psum+ksum-e0)/e0);  
    }

    ////////////////UPDATE/////////////////

    //Update Params | &Table |  dt | offset |
    //Bytes         |   8    |  8  |   4    |

    int upda_mem_needed = sizeof(void*) + sizeof(double) + sizeof(int);

    void *h_upda_params = malloc(upda_mem_needed); 
    memcpy(h_upda_params               , &d_table, sizeof(void*));
    memcpy(((double*)h_upda_params) + 1,   &dt   , sizeof(double));
    
    void *upda_offset_pointer = ((double*)h_upda_params) + 2; 

    k_calls = pushJobs(np, h_upda_params, upda_offset_pointer, upda_mem_needed, 18);
    pullJobs(k_calls);
  }

  ctime2 = cpu_time();
  printf("Elapsed cpu time for main computation: %.2f\n", ctime2-ctime1);
  
  gemtcCleanup(); 
  return 0; 
}

int pushJobs(int num_tasks, void *h_params, void *offset_pointer, int mem_needed, int microkernel){
  int kernel_calls = num_tasks / MAX_WORKERS; 
  int i;

  for(i=0; i<= kernel_calls; i++){
    int offset = i * MAX_WORKERS; 
    int threads = (offset + MAX_WORKERS <= num_tasks) ? MAX_WORKERS : num_tasks-offset;  
    
    if(threads > 0){
      //Allocate device memory for params. 
      void *d_params = gemtcGPUMalloc(mem_needed);
      //Copy the offset into parameters.
      memcpy(offset_pointer, &offset , sizeof(int));
      //Copy params to device. 
      gemtcMemcpyHostToDevice(d_params, h_params, mem_needed); 
      //Push Job 
      printf("gemtcPush(%d, %d, %d, d_params);\n", microkernel, threads, i*1000); 
      gemtcPush(microkernel, threads, i*1000, d_params); 
    }
  }

  return kernel_calls; 
}

void pullJobs(int kernel_calls){
  int i; 
  for(i=0; i<kernel_calls; i++){ //Pulls for jobs. 
    void *ret = NULL;
    int id;

    while(ret==NULL){
      gemtcPoll(&id, &ret);
    } 
  }
}

double cpu_time(){
  double value = (double)clock() / (double)CLOCKS_PER_SEC;
  return value;
}
