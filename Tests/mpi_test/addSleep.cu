#include<hip/hip_runtime.h>
#include<stdio.h>

__global__ void addSleep(int *v, int *r){
  float v1 =(float) *v;
  int ret =0;
  while(ret <v1){
    ret = ret+1;
  }
  *r=ret;
}

void setupGemtc(int v){
  int * d_v, *d_r;
  hipMalloc(&d_v, sizeof(int));
  hipMalloc(&d_r, sizeof(int));
  hipMemcpy(d_v, &v, sizeof(int), hipMemcpyHostToDevice);

  dim3 threads(32, 1);
  dim3 grid(1, 1);

  addSleep<<<grid, threads, 0>>>(d_v, d_r);
  
  int r;
  hipMemcpy(&r, d_r, sizeof(int), hipMemcpyDeviceToHost);
  printf("%d\n", r);
}

