#include "../../gemtc.cu"
#include<stdio.h>
#include<stdlib.h>

int main(int argc, char **argv){

  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, 0);
  printf("  Global memory:  %d mb\n", (int) props.totalGlobalMem);
  printf("  Shared memory:  %d kb/block\n ",(int) props.sharedMemPerBlock);

  int NUM_TASKS, SLEEP_TIME;

  if(argc>2){
    NUM_TASKS = atoi(argv[1]);
    SLEEP_TIME = atoi(argv[2]);
  }else{
    printf("This test requires two parameters:\n");
    printf("   int NUM_TASKS, SLEEP_TIME\n");
    printf("where  NUM_TASKS is the total numer of tasks to be run, must be a multiple of 1000\n");
    printf("       SLEEP_TIME is the parameter that will be passed to each AddSleep, in microseconds\n");
    exit(1);
  }

  gemtcSetup(25600);

  //We will Push 1000 tasks
  // Then Poll until we have 1000 results
  // Untilwe have run all the tasks
  int j;
  for(j=0; j<NUM_TASKS/1000; j++){
    int i;
    for(i=0; i<1000; i++){
      int *d_sleepTime = (int *) gemtcGPUMalloc(sizeof(int));

      gemtcMemcpyHostToDevice(d_sleepTime, &SLEEP_TIME, sizeof(int));
      gemtcPush(0, 32, i+j*1000, d_sleepTime);
    }

    for(i=0; i<1000; i++){
      void *ret=NULL;
      int id;      
      while(ret==NULL){
        gemtcPoll(&id, &ret);
      }

      int h_sleepTime;
      gemtcMemcpyDeviceToHost(&h_sleepTime, ret, sizeof(int));
      //printf("Recieved task %d\n", id);
      gemtcGPUFree(ret);
      ret = NULL;
    }
  }

  gemtcCleanup();

  return 0;
}
