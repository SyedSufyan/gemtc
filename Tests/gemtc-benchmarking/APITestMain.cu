#include "../../gemtc.cu"
#include<stdio.h>
#include<stdlib.h>

int main(int argc, char **argv){

  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, 0);
  printf("  Global memory:  %d mb\n", (int) props.totalGlobalMem);
  printf("  Shared memory:  %d kb/block\n ",(int) props.sharedMemPerBlock);

  gemtcSetup(25600);

  int NUM_TASKS = 20000; //Must be a multiple of 1000
  int SLEEP_TIME = 1000000;

  if(argc>2){
    NUM_TASKS = atoi(argv[1]);
    SLEEP_TIME = atoi(argv[2]);
  }

  //We will Push 1000 tasks
  // Then Poll until we have 1000 results
  // Untilwe have run all the tasks
  int j;
  for(j=0; j<NUM_TASKS/1000; j++){
    int i;
    for(i=0; i<1000; i++){
      int *d_sleepTime = (int *) gemtcGPUMalloc(sizeof(int));

      gemtcMemcpyHostToDevice(d_sleepTime, &SLEEP_TIME, sizeof(int));
      gemtcPush(0, 32, i+j*1000, d_sleepTime);
    }

    ResultPair *ret=NULL;
    for(i=0; i<1000; i++){
      while(ret==NULL){
        ret = (ResultPair *)gemtcPoll();
      }

      int h_sleepTime;
      gemtcMemcpyDeviceToHost(&h_sleepTime, ret->params, sizeof(int));
      printf("Recieved task %d\n", ret->ID);
      gemtcGPUFree(ret->params);
      ret = NULL;
    }
  }

  gemtcCleanup();

  return 0;
}
