#include "../../gemtc.cu"
#include<stdio.h>
#include<stdlib.h>

int main(int argc, char **argv){

  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, 0);
 
  gemtcSetup(25600, 0);
  char* message = "Ben's Message"; //13 chars and a '\0'
  void* d_memory = gemtcGPUMalloc(14*sizeof(char));

  gemtcMemcpyHostToDevice(d_memory, &message, 14*sizeof(char));

  gemtcPush(15, 1, 12000, d_memory); 

  void *ret=NULL;
  int id;

  while(ret==NULL){
    gemtcPoll(&id, &ret);
  }

  char* h_ret_message = (char *) malloc(14*sizeof(char));
  gemtcMemcpyDeviceToHost(h_ret_message, ret, 14*sizeof(char));
  printf("Received task %d\n", id);
  printf("message = %s\n", h_ret_message);

  gemtcGPUFree(ret);
  ret = NULL;

  gemtcCleanup();

  free(h_ret_message);

  return 0;
}
