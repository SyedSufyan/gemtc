#include "../../gemtc.cu"
#include<stdio.h>
#include<stdlib.h>

int main(int argc, char **argv){

  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, 0);
 
  gemtcSetup(25600);
  char* message = "I am not the right message!.";
  void* d_memory = gemtcGPUMalloc(sizeof(message));

  gemtcMemcpyHostToDevice(d_memory, &message, sizeof(message));
  gemtcPush(15, 1, 12000, d_memory); 

  void *ret=NULL;
  int id;

  while(ret==NULL){
    gemtcPoll(&id, &ret);
  }
  
  char* h_ret_message;
  gemtcMemcpyDeviceToHost(&h_ret_message, ret, sizeof(char)*20);
  printf("Received task %d\n", id);
  printf("message = %s\n", h_ret_message);

  gemtcGPUFree(ret);
  ret = NULL;

  gemtcCleanup();

  return 0;
}
