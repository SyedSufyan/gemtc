#include "hip/hip_runtime.h"
#include "allocations.cu"
#include <time.h>
void testSleep()
{
    int sleepTime = 5;
    void* ret = run(0, 32, &sleepTime, sizeof(int));
    free(ret);
}


void testArrayAvg()
{
    void* param; void* ret;
    int N = 32 << 1;
    int size = sizeof(int)*(N*3+1);
    param = allocateArray(N, size);
    ret = run(12, 32, param, size);
    float* ret1 = (float*)ret;
    ++ret1;
    ret1=ret1+N; 
    printf("N:%d, Size: %d, Average: %f\n",N,size, ret1[0]);
    free(ret);free(param);
}


void testArrayMax()
{
    void* param; void* ret;
    int N = 32 << 10;
    int size = sizeof(int)*(N*3+1);
    param = allocateArray(N, size);
    ret = run(12, 32, param, size);
    float* ret1 = (float*)ret;
    ++ret1;
    ret1=ret1+N; 
    printf("N:%d, Size: %d, Max: %f\n",N,size, ret1[0]);
    free(ret);free(param);
}

void testArrayMin()
{
    void* param; void* ret;
    int N = 32 << 10;
    int size = sizeof(int)*(N*3+1);
    param = allocateArray(N, size);
    ret = run(11, 32, param, size);
    float* ret1 = (float*)ret;
    ++ret1;
    ret1=ret1+N; 
    printf("N:%d, Size: %d, Min: %f\n",N,size, ret1[0]);
    free(ret);free(param);
}

void testAdd()
{
    int testSize = 10;
    int N = 16;
    N = 65535*8;
    clock_t begin, end;
    double flop = N;
    void* param; void* ret;
    int size = sizeof(int)*(N*3+1);
    param = makeVectorAddArgs(N, size);
    //printf("testAdd: %d,%d\n", N, size);
    begin = clock();
    for (int idx = 1; idx <= testSize; ++idx)
    {
        ret = run(1, 32, param, size);
        free(ret);
    }
    end = clock();
    //printf("Start: %ld, End: %ld\n", begin, end);
    double time = (double)(end - begin)/CLOCKS_PER_SEC;
    time = time/testSize;
    flop = (N/time)*1.0e-6;
    printf("testAdd: N:%d,Size:%.10g MB, time:%.5g s, MFLOP:%.10g\n", 
            N, (double)size/1000000, time, flop);
#if 0 
    for (int idx = 0; idx < N; ++idx)
    {
       int v = A[idx] + B[idx];
       if ( v != C[idx])
       printf("v=%d\n",C[idx]-v); 
    }
#endif
}

void testVectorProduct()
{
    int N = 1024 << 4;
    int size = sizeof(float)*(32*N+1);
    void* param = makeVectorArgsAsFloat(N, size);
    void* ret = run(3, 32, param, size);
#if 1 
    float* ret1 = (float*)ret;
    float* A = ret1+1;
    float* B = A+N;
    float* C = B+N;
    printf("testVectorProduct:N%d,size: %d, Result: %f\n", N, size,C[0]);
#endif
    free(ret);free(param);
}

void testMatrixSquare()
{
    int ROW = (32 << 3) >> 1;
    int size = 0;
    void* param = makeMatrixTranspose(ROW, size);
    void* ret = run(2, 32, param,size);
    printf("testMatrixSquare:N%d,size: %d \n", ROW, size);
#if 0 
    float* ret1 = (float*)ret;
    float* A = ret1+1;
    float* B = A + ROW * ROW;
    float* result = (float*)malloc(ROW*ROW*sizeof(float));
    for (int idx = 0; idx < ROW; ++idx)
    {
        for (int jdx = 0; jdx < ROW; ++jdx)
        {
           float sum = 0;
           for (int kdx = 0; kdx < ROW; ++kdx)
           {
              float a = A[idx*ROW+kdx];
              float b = A[kdx*ROW+jdx];
              sum += a * b;
           }
           result[idx*ROW+jdx] = sum;
        }
    }
    for (int idx = 0; idx < ROW; ++idx)
    {
        for (int jdx = 0; jdx < ROW; ++jdx)
        {
           int kdx = idx*ROW+jdx;
           printf(" %f ", result[kdx] - B[kdx]);
        }
        printf("\n");
    }
    free(result);
#endif
    free(ret);free(param);
}
    
void testMatrixMultiply()
{
    clock_t start, end;
    int ROW = 320;//32*20;
    int size = 0;
    int numTests = 3;
    void* param = makeMatrixMult(ROW, size);
    start = clock();
    for (int idx = 1; idx <= numTests; ++idx)
    {
       //int ROW = 32 << idx;
       ROW = 300;//32*20;
       void* ret = run(4, 32, param,size);
       printf("idx:%d\n",idx);
       free(ret);
    }
    end = clock();
    free(param);
    double time = (double)(end-start)/CLOCKS_PER_SEC;
    time = time/numTests;
    int flops = 2*ROW*ROW*ROW;
    double flop = flops/time;
    flop = flop/1000000;
    printf("Elements: %d, Memory: %d, time: %.5g, gflop: %.5g\n", 
            ROW, size, time, flop); 
}

void testMatrixTranspose()
{
    int ROW = 32 << 1; int COLUMN = 32; int number = 1;
    int size = (number*ROW*COLUMN);
    void* param = makeMatrixTranspose(ROW, size);
    void* ret = run(5, 32, param,size);
    printf("testMatrixTranspose, Elements: %d, Memory: %d\n", ROW, size); 
#if 0 
    float* ret1 = (float*)ret;
    float* A = ret1+1;
    float* B = A + ROW * ROW;
    for (int idx = 0; idx < ROW; ++idx)
    {
        for (int jdx = 0; jdx < ROW; ++jdx)
        {
           int kdx = idx*ROW+jdx;
           printf(" %f ", A[kdx]);
        }
        printf("\n");
    }
    printf ("\n---------\n");
    for (int idx = 0; idx < ROW; ++idx)
    {
        for (int jdx = 0; jdx < ROW; ++jdx)
        {
           int kdx = idx*ROW+jdx;
           printf(" %f ", B[kdx]);
        }
        printf("\n");
    }
#endif
    free(ret);free(param);
}

void testMatrixVector()
{
    int ROW = 512; int size = 0;
    void* param = makeMatrixVectorArgs(ROW, size);
    void* ret = run(6, 32, param,size);
    printf("testMatrixVector, Elements: %d, Memory: %d\n", ROW, size); 
    free(ret);free(param);
}

void testMatrixInverse()
{
    int ROW = 3; int size = 0;
    void* param = makeMatrixInverse(ROW, size);
    void* ret = run(7, 32, param,size);
    printf("testMatrixInverse, Elements: %d, Memory: %d\n", ROW, size); 
    free(ret);free(param);
}

void testStencil()
{
   int N  =  128;
   float dt = 0.00001f;
   float time = 0.4f;
   int step = ceil(time/dt);
   int size = 0;
   void* param = allocateStencil( N,size);
   step = 2;
   for (int t=0; t<step; t++)
   {    
        void* ret = run(8, 32, param,size);
        free(ret);
        ret = run(9, 32, param,size);
        free(ret);
        printf("Processing time step: %d\n", t);
   }
   free(param);
}

void testBlackScholes()
{
   int N = 100 << 5;
   int size = 0;
   void *param = allocateBlackScholes(N, size);
   void* ret = run(10,32,param,size);
    printf("testBlackScholes, Elements: %d, Memory: %d\n", N, size); 
   free(ret);
}
