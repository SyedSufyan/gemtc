#include "hip/hip_runtime.h"
#include "allocations.cu"

void testSleep()
{
    int sleepTime = 5;
    void* ret = run(0, 32, &sleepTime, sizeof(int));
    free(ret);
}


void testArrayAvg()
{
    void* param; void* ret;
    int N = 32 << 1;
    int size = sizeof(int)*(N*3+1);
    param = allocateArray(N, size);
    ret = run(12, 32, param, size);
    float* ret1 = (float*)ret;
    ++ret1;
    ret1=ret1+N; 
    printf("N:%d, Size: %d, Average: %f\n",N,size, ret1[0]);
    free(ret);free(param);
}


void testArrayMax()
{
    void* param; void* ret;
    int N = 32 << 10;
    int size = sizeof(int)*(N*3+1);
    param = allocateArray(N, size);
    ret = run(12, 32, param, size);
    float* ret1 = (float*)ret;
    ++ret1;
    ret1=ret1+N; 
    printf("N:%d, Size: %d, Max: %f\n",N,size, ret1[0]);
    free(ret);free(param);
}

void testArrayMin()
{
    void* param; void* ret;
    int N = 32 << 10;
    int size = sizeof(int)*(N*3+1);
    param = allocateArray(N, size);
    ret = run(11, 32, param, size);
    float* ret1 = (float*)ret;
    ++ret1;
    ret1=ret1+N; 
    printf("N:%d, Size: %d, Min: %f\n",N,size, ret1[0]);
    free(ret);free(param);
}

void testAdd()
{
    void* param; void* ret;
    // runs a task on the gpu
    int N = 1024 << 4;
    //N = 32;
    int size = sizeof(int)*(N*3+1);
    param = makeVectorAddArgs(N, size);
    //printf("testAdd: %d,%d\n", N, size);
    ret = run(1, 32, param, size);
    int* ret1 = (int*)ret;
    int* ret2 = (int*)param;
    int* A = ret1+1;
    int* B = A+N;
    int* C = B+N;
    printf("testAdd: N:%d,Size:%d\n", N, size);
#if 0 
    for (int idx = 0; idx < N; ++idx)
    {
       int v = A[idx] + B[idx];
       if ( v != C[idx])
       printf("v=%d\n",C[idx]-v); 
    }
#endif
    free(ret);free(param);
}

void testVectorProduct()
{
    int N = 1024 << 4;
    int size = sizeof(float)*(32*N+1);
    void* param = makeVectorArgsAsFloat(N, size);
    void* ret = run(3, 32, param, size);
#if 1 
    float* ret1 = (float*)ret;
    float* A = ret1+1;
    float* B = A+N;
    float* C = B+N;
    printf("testVectorProduct:N%d,size: %d, Result: %f\n", N, size,C[0]);
#endif
    free(ret);free(param);
}

void testMatrixSquare()
{
    int ROW = (32 << 3) >> 1;
    int size = 0;
    void* param = makeMatrixTranspose(ROW, size);
    void* ret = run(2, 32, param,size);
    printf("testMatrixSquare:N%d,size: %d \n", ROW, size);
#if 0 
    float* ret1 = (float*)ret;
    float* A = ret1+1;
    float* B = A + ROW * ROW;
    float* result = (float*)malloc(ROW*ROW*sizeof(float));
    for (int idx = 0; idx < ROW; ++idx)
    {
        for (int jdx = 0; jdx < ROW; ++jdx)
        {
           float sum = 0;
           for (int kdx = 0; kdx < ROW; ++kdx)
           {
              float a = A[idx*ROW+kdx];
              float b = A[kdx*ROW+jdx];
              sum += a * b;
           }
           result[idx*ROW+jdx] = sum;
        }
    }
    for (int idx = 0; idx < ROW; ++idx)
    {
        for (int jdx = 0; jdx < ROW; ++jdx)
        {
           int kdx = idx*ROW+jdx;
           printf(" %f ", result[kdx] - B[kdx]);
        }
        printf("\n");
    }
    free(result);
#endif
    free(ret);free(param);
}
    
void testMatrixMultiply()
{
    int ROW = 32 << 1;
    int size = 0;
    void* param = makeMatrixMult(ROW, size);
    void* ret = run(4, 32, param,size);
    printf("testMatrixMultiply, Elements: %d, Memory: %d\n", ROW, size); 
    free(ret);free(param);
}

void testMatrixTranspose()
{
    int ROW = 32 << 1; int COLUMN = 32; int number = 1;
    int size = (number*ROW*COLUMN);
    void* param = makeMatrixTranspose(ROW, size);
    void* ret = run(5, 32, param,size);
    printf("testMatrixTranspose, Elements: %d, Memory: %d\n", ROW, size); 
#if 0 
    float* ret1 = (float*)ret;
    float* A = ret1+1;
    float* B = A + ROW * ROW;
    for (int idx = 0; idx < ROW; ++idx)
    {
        for (int jdx = 0; jdx < ROW; ++jdx)
        {
           int kdx = idx*ROW+jdx;
           printf(" %f ", A[kdx]);
        }
        printf("\n");
    }
    printf ("\n---------\n");
    for (int idx = 0; idx < ROW; ++idx)
    {
        for (int jdx = 0; jdx < ROW; ++jdx)
        {
           int kdx = idx*ROW+jdx;
           printf(" %f ", B[kdx]);
        }
        printf("\n");
    }
#endif
    free(ret);free(param);
}

void testMatrixVector()
{
    int ROW = 512; int size = 0;
    void* param = makeMatrixVectorArgs(ROW, size);
    void* ret = run(6, 32, param,size);
    printf("testMatrixVector, Elements: %d, Memory: %d\n", ROW, size); 
    free(ret);free(param);
}

void testMatrixInverse()
{
    int ROW = 3; int size = 0;
    void* param = makeMatrixInverse(ROW, size);
    void* ret = run(7, 32, param,size);
    printf("testMatrixInverse, Elements: %d, Memory: %d\n", ROW, size); 
    free(ret);free(param);
}

void testStencil()
{
   int N  =  128;
   float dt = 0.00001f;
   float time = 0.4f;
   int step = ceil(time/dt);
   int size = 0;
   void* param = allocateStencil( N,size);
   step = 2;
   for (int t=0; t<step; t++)
   {    
        void* ret = run(8, 32, param,size);
        free(ret);
        ret = run(9, 32, param,size);
        free(ret);
        printf("Processing time step: %d\n", t);
   }
   free(param);
}

void testBlackScholes()
{
   int N = 100 << 5;
   int size = 0;
   void *param = allocateBlackScholes(N, size);
   void* ret = run(10,32,param,size);
    printf("testBlackScholes, Elements: %d, Memory: %d\n", N, size); 
   free(ret);
}
