#include "../../gemtc.cu"
#include<stdio.h>
#include<stdlib.h>

int main(int argc, char **argv){

  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, 0);
  
  //We have 1024 MB 
  //printf("\tGlobal memory: %d mb\n", (int) (props.totalGlobalMem/(1024*1024)));

  gemtcSetup(25600);
  int NUM_NODES = 0;
  int *d_num_nodes = (int *)gemtcGPUMalloc(sizeof(int));

  gemtcMemcpyHostToDevice(d_num_nodes, &NUM_NODES, sizeof(int));
  gemtcPush(15, 1, 12000, d_num_nodes); 

  void *ret=NULL;
  int id;

  while(ret==NULL){
    gemtcPoll(&id, &ret);
  }
  
  int h_num_nodes;
  gemtcMemcpyDeviceToHost(&h_num_nodes, ret, sizeof(int));
  printf("Received task %d\n", id);
  printf("ret_val = %d\n", h_num_nodes);

  gemtcGPUFree(ret);
  ret = NULL;

  gemtcCleanup();

  return 0;
}
