#include <hip/hip_runtime.h>
#include <stdio.h>

void saxpy (float* X, float* Y, float* Z, int n);
float avg (float* arr, int n);

__global__
void saxpyKernel(float *x, float *y, float *z, float a, int n) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (id < n)
    z[id] = a*x[id] + y[id];
}

int main () {
  int N = 1<<20;
  int size = N*sizeof(float);
  // Host input and output vectors
  float *h_x, *h_y, *h_z;
  
  // Allocate host memory for vecs
  h_x = (float*)malloc(size);
  h_y = (float*)malloc(size);
  h_z = (float*)malloc(size);
  
  int i;
  for (i = 0; i < N; i++) {
    h_x[i] = 1.0;
    h_y[i] = 2.0;
  }

  // Perform SAXPY on 1M elements
  saxpy(h_x, h_y, h_z, N);

  printf("AVG = %f\n", avg(h_z, N));
  
  // free host memory
  free(h_x);
  free(h_y);
  free(h_z);

  return 0;
}

void saxpy (float* X, float* Y, float* Z, int n) {
  //Device input and output vectors  
  float *d_x, *d_y, *d_z;
  int size = n*sizeof(float);

  // Allocate device memory
  hipMalloc((void**)&d_x, size);
  hipMalloc((void**)&d_y, size);
  hipError_t z_err = hipMalloc((void**)&d_z, size);
  if (z_err != hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString(z_err), __FILE__, __LINE__);}

  // Copy X and Y vectors to device
  hipMemcpy(d_x, X, size, hipMemcpyHostToDevice);
  hipMemcpy(d_y, Y, size, hipMemcpyHostToDevice);

  // number of threads per block
  int blockSize = 1024;
  // number of blocks
  //int gridSize = (int)ceil((float)n/blockSize);
  int gridSize = n/blockSize;

  saxpyKernel<<<gridSize, blockSize>>>(d_x, d_y, d_z, 2.0, n);

  // Copy z from device to host
  hipMemcpy(Z, d_z, size, hipMemcpyDeviceToHost);

  // free device memory
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);
  
}

float avg (float* arr, int n) {
  int i;
  float total = 0;
  for (i = 0; i < n; i++) {
    total += arr[i];
  }
  return total / n;
}
